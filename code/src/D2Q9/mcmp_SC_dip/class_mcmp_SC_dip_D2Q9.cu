
# include "class_mcmp_SC_dip_D2Q9.cuh"
# include "../../IO/GetPot"
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

class_mcmp_SC_dip_D2Q9::class_mcmp_SC_dip_D2Q9()
{
	Q = 9;
	GetPot inputParams("input.dat");	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = 1;
	if (nVoxels != Nx*Ny*Nz) cout << "nVoxels does not match Nx, Ny, Nz!" << endl;
	numIolets = inputParams("Lattice/numIolets",0);
	nParts = inputParams("Particles/nParts",0);
	nu = inputParams("LBM/nu",0.1666666);
	gAB = inputParams("LBM/gAB",6.0);
	gAS = inputParams("LBM/gAS",4.5);
	gBS = inputParams("LBM/gBS",4.5); 
	omega = inputParams("LBM/omega",0.0);
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

class_mcmp_SC_dip_D2Q9::~class_mcmp_SC_dip_D2Q9()
{
		
}



// --------------------------------------------------------
// Allocate arrays:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::allocate()
{
	// allocate array memory (host):
    uH = (float*)malloc(nVoxels*sizeof(float));
	vH = (float*)malloc(nVoxels*sizeof(float));
    rAH = (float*)malloc(nVoxels*sizeof(float));
	rBH = (float*)malloc(nVoxels*sizeof(float));
	xH = (int*)malloc(nVoxels*sizeof(int));
	yH = (int*)malloc(nVoxels*sizeof(int));
	nListH = (int*)malloc(nVoxels*Q*sizeof(int));
	voxelTypeH = (int*)malloc(nVoxels*sizeof(int));
	streamIndexH = (int*)malloc(nVoxels*Q*sizeof(int));	
	ioletsH = (iolet2D*)malloc(numIolets*sizeof(iolet2D));
	ptH = (particle2D_dip*)malloc(nParts*sizeof(particle2D_dip));
    pfxH = (float*)malloc(nVoxels*sizeof(float));
	pfyH = (float*)malloc(nVoxels*sizeof(float));
			
	// allocate array memory (device):
	hipMalloc((void **) &u, nVoxels*sizeof(float));
	hipMalloc((void **) &v, nVoxels*sizeof(float));
	hipMalloc((void **) &rA, nVoxels*sizeof(float));
	hipMalloc((void **) &rB, nVoxels*sizeof(float));
	hipMalloc((void **) &rS, nVoxels*sizeof(float));
	hipMalloc((void **) &f1A, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2A, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &f1B, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2B, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &FxA, nVoxels*sizeof(float));
	hipMalloc((void **) &FyA, nVoxels*sizeof(float));
	hipMalloc((void **) &FxB, nVoxels*sizeof(float));
	hipMalloc((void **) &FyB, nVoxels*sizeof(float));
	hipMalloc((void **) &x, nVoxels*sizeof(int));	
	hipMalloc((void **) &y, nVoxels*sizeof(int));	
	hipMalloc((void **) &nList, nVoxels*Q*sizeof(int));	
	hipMalloc((void **) &voxelType, nVoxels*sizeof(int));
	hipMalloc((void **) &pIDgrid, nVoxels*sizeof(int));
	hipMalloc((void **) &streamIndex, nVoxels*Q*sizeof(int));	
	hipMalloc((void **) &iolets, numIolets*sizeof(iolet2D));
	hipMalloc((void **) &pt, nParts*sizeof(particle2D_dip));	
	hipMalloc((void **) &pfx, nVoxels*sizeof(float));
	hipMalloc((void **) &pfy, nVoxels*sizeof(float));
	
	
}



// --------------------------------------------------------
// Deallocate arrays:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::deallocate()
{
	// free array memory (host):
	free(uH);
	free(vH);
	free(rAH);
	free(rBH);
	free(xH);
	free(yH);
	free(nListH);
	free(voxelTypeH);
	free(streamIndexH);	
	free(ioletsH);	
	free(ptH);
		
	// free array memory (device):
	hipFree(u);
	hipFree(v);	
	hipFree(rA);
	hipFree(rB);
	hipFree(rS);
	hipFree(f1A);
	hipFree(f2A);	
	hipFree(f1B);
	hipFree(f2B);
	hipFree(FxA);
	hipFree(FxB);
	hipFree(FyA);
	hipFree(FyB);
	hipFree(x);
	hipFree(y);
	hipFree(nList);
	hipFree(voxelType);
	hipFree(streamIndex);
	hipFree(iolets);
	hipFree(pt);
}



// --------------------------------------------------------
// Copy arrays from host to device:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::memcopy_host_to_device()
{
    hipMemcpy(u, uH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(v, vH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rA, rAH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rB, rBH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(x, xH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(y, yH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(nList, nListH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(voxelType, voxelTypeH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(streamIndex, streamIndexH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(iolets, ioletsH, sizeof(iolet2D)*numIolets, hipMemcpyHostToDevice);
	hipMemcpy(pt, ptH, sizeof(particle2D_dip)*nParts, hipMemcpyHostToDevice);
}



// --------------------------------------------------------
// Copy arrays from device to host:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::memcopy_device_to_host()
{
    hipMemcpy(uH, u, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(vH, v, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rAH, rA, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rBH, rB, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(ptH, pt, sizeof(particle2D_dip)*nParts, hipMemcpyDeviceToHost);
    hipMemcpy(pfxH, pfx, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(pfyH, pfy, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
}



// --------------------------------------------------------
// Copy arrays from device to host:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::memcopy_device_to_host_particles()
{
    hipMemcpy(ptH, pt, sizeof(particle2D_dip)*nParts, hipMemcpyDeviceToHost);
}



// --------------------------------------------------------
// Initialize lattice as a "box":
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::create_lattice_box()
{
	GetPot inputParams("input.dat");	
	int flowDir = inputParams("Lattice/flowDir",0);
	int xLBC = inputParams("Lattice/xLBC",0);
	int xUBC = inputParams("Lattice/xUBC",0);
	int yLBC = inputParams("Lattice/yLBC",0);
	int yUBC = inputParams("Lattice/yUBC",0);			
	build_box_lattice_D2Q9(nVoxels,flowDir,Nx,Ny,
	                       xLBC,xUBC,yLBC,yUBC,
	                       voxelTypeH,nListH);
}



// --------------------------------------------------------
// Initialize lattice as a "box" with periodic BC's:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::create_lattice_box_periodic()
{
	build_box_lattice_D2Q9(nVoxels,Nx,Ny,voxelTypeH,nListH);
}



// --------------------------------------------------------
// Initialize lattice from "file":
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::create_lattice_file()
{
	
}



// --------------------------------------------------------
// Build the streamIndex[] array for PUSH streaming:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::stream_index_push()
{
	stream_index_push_D2Q9(nVoxels,nListH,streamIndexH);
}



// --------------------------------------------------------
// Build the streamIndex[] array for PULL streaming:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::stream_index_pull()
{
	stream_index_pull_D2Q9(nVoxels,nListH,streamIndexH);
}



// --------------------------------------------------------
// Read information about iolet:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::read_iolet_info(int i, const char* name) 
{
	char namemod[20];
	GetPot inputParams("input.dat");
	if (i >= 0 and i < numIolets) {
		strcpy(namemod, name);
		strcat(namemod, "/type");
		ioletsH[i].type = inputParams(namemod,1);
		strcpy(namemod, name);
		strcat(namemod, "/uBC");
		ioletsH[i].uBC  = inputParams(namemod,0.0);
		strcpy(namemod, name);
		strcat(namemod, "/vBC");
		ioletsH[i].vBC  = inputParams(namemod,0.0);
		strcpy(namemod, name);
		strcat(namemod, "/rBC");
		ioletsH[i].rBC  = inputParams(namemod,1.0);
		strcpy(namemod, name);
		strcat(namemod, "/pBC");
		ioletsH[i].pBC  = inputParams(namemod,0.0);
	}
	else {
		cout << "iolet index is not correct" << endl;
	}
}



// --------------------------------------------------------
// Swap the populations 1 and 2 for both A and B:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::swap_populations()
{
	float* tempA = f1A;
	float* tempB = f1B;
	f1A = f2A;
	f1B = f2B;
	f2A = tempA;
	f2B = tempB;
}



// --------------------------------------------------------
// Setters for host arrays:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::setU(int i, float val)
{
	uH[i] = val;
}

void class_mcmp_SC_dip_D2Q9::setV(int i, float val)
{
	vH[i] = val;
}

void class_mcmp_SC_dip_D2Q9::setX(int i, int val)
{
	xH[i] = val;
}

void class_mcmp_SC_dip_D2Q9::setY(int i, int val)
{
	yH[i] = val;
}

void class_mcmp_SC_dip_D2Q9::setRA(int i, float val)
{
	rAH[i] = val;
}

void class_mcmp_SC_dip_D2Q9::setRB(int i, float val)
{
	rBH[i] = val;
}

void class_mcmp_SC_dip_D2Q9::setVoxelType(int i, int val)
{
	voxelTypeH[i] = val;
}

void class_mcmp_SC_dip_D2Q9::setPrx(int i, float val)
{
	ptH[i].r.x = val;
}

void class_mcmp_SC_dip_D2Q9::setPry(int i, float val)
{
	ptH[i].r.y = val;
}

void class_mcmp_SC_dip_D2Q9::setPvx(int i, float val)
{
	ptH[i].v.x = val;
}

void class_mcmp_SC_dip_D2Q9::setPvy(int i, float val)
{
	ptH[i].v.y = val;
}

void class_mcmp_SC_dip_D2Q9::setPrInner(int i, float val)
{
	ptH[i].rInner = val;
}

void class_mcmp_SC_dip_D2Q9::setPrOuter(int i, float val)
{
	ptH[i].rOuter = val;
}



// --------------------------------------------------------
// Getters for host arrays:
// --------------------------------------------------------

float class_mcmp_SC_dip_D2Q9::getU(int i)
{
	return uH[i];
}

float class_mcmp_SC_dip_D2Q9::getV(int i)
{
	return vH[i];
}

float class_mcmp_SC_dip_D2Q9::getRA(int i)
{
	return rAH[i];
}

float class_mcmp_SC_dip_D2Q9::getRB(int i)
{
	return rBH[i];
}

float class_mcmp_SC_dip_D2Q9::getPrx(int i)
{
	return ptH[i].r.x;
}

float class_mcmp_SC_dip_D2Q9::getPry(int i)
{
	return ptH[i].r.y;
}

float class_mcmp_SC_dip_D2Q9::getPfx(int i)
{
	return ptH[i].f.x;
}

float class_mcmp_SC_dip_D2Q9::getPfy(int i)
{
	return ptH[i].f.y;
}

float class_mcmp_SC_dip_D2Q9::getPrInner(int i)
{
	return ptH[i].rInner;
}

float class_mcmp_SC_dip_D2Q9::getPrOuter(int i)
{
	return ptH[i].rOuter;
}
 


// --------------------------------------------------------
// Calls to Kernels:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::initial_equilibrium_dip(int nBlocks, int nThreads)
{
	mcmp_initial_equilibrium_dip_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,u,v,nVoxels);	
}

void class_mcmp_SC_dip_D2Q9::compute_density_dip(int nBlocks, int nThreads)
{
	mcmp_compute_density_dip_D2Q9
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,nVoxels);
}

void class_mcmp_SC_dip_D2Q9::map_particles_to_lattice_dip(int nBlocks, int nThreads)
{
	mcmp_map_particles_to_lattice_dip_D2Q9
	<<<nBlocks,nThreads>>> (rS,pt,x,y,pIDgrid,nVoxels,nParts);
} 

void class_mcmp_SC_dip_D2Q9::compute_SC_forces_dip(int nBlocks, int nThreads)
{
	mcmp_compute_SC_forces_dip_D2Q9 
	<<<nBlocks,nThreads>>> (rA,rB,rS,FxA,FxB,FyA,FyB,pfx,pfy,pt,nList,pIDgrid,gAB,gAS,gBS,omega,nVoxels);	
}

void class_mcmp_SC_dip_D2Q9::compute_velocity_dip(int nBlocks, int nThreads)
{
	mcmp_compute_velocity_dip_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,rS,FxA,FxB,FyA,FyB,u,v,pt,pIDgrid,nVoxels);
}

void class_mcmp_SC_dip_D2Q9::compute_velocity_dip_2(int nBlocks, int nThreads)
{
	mcmp_compute_velocity_dip_2_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,rS,FxA,FxB,FyA,FyB,u,v,pt,pIDgrid,nVoxels);
}

void class_mcmp_SC_dip_D2Q9::set_boundary_velocity_dip(float uBC, float vBC, int nBlocks, int nThreads)
{
	mcmp_set_boundary_velocity_dip_D2Q9 
	<<<nBlocks,nThreads>>> (uBC,vBC,rA,rB,FxA,FxB,FyA,FyB,u,v,y,Ny,nVoxels);
}

void class_mcmp_SC_dip_D2Q9::collide_stream_dip(int nBlocks, int nThreads)
{
	mcmp_collide_stream_dip_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,f2A,f2B,rA,rB,u,v,FxA,FxB,FyA,FyB,streamIndex,nu,nVoxels);
}

void class_mcmp_SC_dip_D2Q9::move_particles_dip(int nBlocks, int nThreads)
{
	mcmp_move_particles_dip_D2Q9
	<<<nBlocks,nThreads>>> (pt,nParts);
}

void class_mcmp_SC_dip_D2Q9::fix_particle_velocity_dip(float pvel, int nBlocks, int nThreads)
{
	mcmp_fix_particle_velocity_dip_D2Q9
	<<<nBlocks,nThreads>>> (pt,pvel,nParts);
}

void class_mcmp_SC_dip_D2Q9::zero_particle_forces_dip(int nBlocks, int nThreads)
{
	mcmp_zero_particle_forces_dip_D2Q9
	<<<nBlocks,nThreads>>> (pt,nParts);
}



// --------------------------------------------------------
// Wrtie output:
// --------------------------------------------------------

void class_mcmp_SC_dip_D2Q9::write_output(std::string tagname, int step)
{
	//write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,rAH,rBH,uH,vH);
	write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,rAH,rBH,pfxH,pfyH);
	//write_vtk_structured_grid_2D("rA",step,Nx,Ny,Nz,rAH,uH,vH);
	//write_vtk_structured_grid_2D("rB",step,Nx,Ny,Nz,rBH,uH,vH);
}









