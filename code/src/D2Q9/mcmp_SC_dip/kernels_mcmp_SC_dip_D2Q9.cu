#include "hip/hip_runtime.h"

# include "kernels_mcmp_SC_dip_D2Q9.cuh"
# include "../mcmp_SC/mcmp_pseudopotential.cuh"
# include <stdio.h>



// --------------------------------------------------------
// Zero particle forces:
// --------------------------------------------------------

__global__ void mcmp_zero_particle_forces_dip_D2Q9(particle2D_dip* pt,
							                       int nParts)
{
	// define particle:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nParts) {		
		pt[i].f = make_float2(0.0);
		pt[i].mass = 0.0;  // this gets added up when re-mapping particles on lattice
	}
}



// --------------------------------------------------------
// Update particle velocities and positions:
// --------------------------------------------------------

__global__ void mcmp_move_particles_dip_D2Q9(particle2D_dip* pt,
   								             int nParts)
{
	// define particle:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nParts) {		
		float2 a = pt[i].f/pt[i].mass;
		pt[i].r += pt[i].v + 0.5*a;  // assume dt = 1
		pt[i].v += a;
	}
}



// --------------------------------------------------------
// Fix particle velocity:
// --------------------------------------------------------

__global__ void mcmp_fix_particle_velocity_dip_D2Q9(particle2D_dip* pt,
                                                    float pvel,
   								                    int nParts)
{
	// define particle:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nParts) {	
		//printf("%f \n",pt[0].f.x);	
		pt[i].f = make_float2(0.0);
		if (i == 0) {
			pt[i].v.x = -pvel;
			pt[i].v.y = 0.00;
		}
		if (i == 1) {
			pt[i].v.x = pvel;
			pt[i].v.y = 0.00;
		}		
	}
}



// --------------------------------------------------------
// D2Q9 kernel to update the particle fields on the lattice: 
// --------------------------------------------------------

__global__ void mcmp_map_particles_to_lattice_dip_D2Q9(float* rS,			                  
                                                       particle2D_dip* pt,
													   int* x,
													   int* y,
													   int* pIDgrid,
										               int nVoxels,
													   int nParts)
{
	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
				
		// --------------------------------------------------	
		// default values:
		// --------------------------------------------------
				
		rS[i] = 0.0;
		pIDgrid[i] = -1;
		
		// --------------------------------------------------	
		// loop over particles:
		// --------------------------------------------------
		
		for (int j=0; j<nParts; j++) {
			
			// ---------------------------	
			// distance to particle c.o.m:
			// ---------------------------
			
			float dx = float(x[i]) - pt[j].r.x;
			float dy = float(y[i]) - pt[j].r.y;
			float r = sqrt(dx*dx + dy*dy);
						
			// ---------------------------	
			// assign values:
			// ---------------------------
			
			float rI = pt[j].rInner;
			float rO = pt[j].rOuter;			
			if (r <= rO) {
				if (r < rI) {
					rS[i] = 1.0;
				}
				else {
					float rr = r - rI;
					rS[i] = 1.0 - rr/(rO-rI);					
				}
				pIDgrid[i] = j;	
				atomicAdd(&pt[j].mass,rS[i]);		
			}			
		}
	}
}



// --------------------------------------------------------
// D2Q9 set velocity on the y=0 and y=Ny-1 boundaries: 
// --------------------------------------------------------

__global__ void mcmp_set_boundary_velocity_dip_D2Q9(float uBC,
                                                    float vBC,
	                                                float* rA,
										            float* rB,
										            float* FxA,
										            float* FxB,
										            float* FyA,
										            float* FyB,
										            float* u,
										            float* v,
													int* y,											        
											        int Ny,
										            int nVoxels) 
{
	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		if (y[i] == 0 || y[i] == Ny-1) {
			float rTotal = rA[i] + rB[i];
			float fxBC = (uBC - u[i])*2.0*rTotal;
			float fyBC = (vBC - v[i])*2.0*rTotal;
			u[i] += 0.5*fxBC/rTotal;
			v[i] += 0.5*fyBC/rTotal;
			FxA[i] += fxBC*(rA[i]/rTotal);
			FxB[i] += fxBC*(rB[i]/rTotal);
			FyA[i] += fyBC*(rA[i]/rTotal);
			FyB[i] += fyBC*(rB[i]/rTotal);
		}		
	}
}



// --------------------------------------------------------
// D2Q9 initialize kernel: 
// --------------------------------------------------------

__global__ void mcmp_initial_equilibrium_dip_D2Q9(float* fA,
                                                  float* fB,
										          float* rA,
											      float* rB,
										          float* u,
										          float* v,
										          int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	// initialize populations to equilibrium values:
	if (i < nVoxels) {	
		
		int offst = 9*i;
		const float w0 = 4.0/9.0;
		const float ws = 1.0/9.0;
		const float wd = 1.0/36.0;
		const float omusq = 1.0 - 1.5*(u[i]*u[i] + v[i]*v[i]);
		
		// dir 0
		float feq = w0*omusq;
		fA[offst+0] = feq*rA[i];
		fB[offst+0] = feq*rB[i];
		
		// dir 1
		float evel = u[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+1] = feq*rA[i];
		fB[offst+1] = feq*rB[i];
		
		// dir 2
		evel = v[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+2] = feq*rA[i];
		fB[offst+2] = feq*rB[i];
		
		// dir 3
		evel = -u[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+3] = feq*rA[i];
		fB[offst+3] = feq*rB[i];
		
		// dir 4
		evel = -v[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+4] = feq*rA[i];
		fB[offst+4] = feq*rB[i];
		
		// dir 5
		evel = u[i] + v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+5] = feq*rA[i];
		fB[offst+5] = feq*rB[i];
		
		// dir 6
		evel = -u[i] + v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+6] = feq*rA[i];
		fB[offst+6] = feq*rB[i];
		
		// dir 7
		evel = -u[i] - v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+7] = feq*rA[i];
		fB[offst+7] = feq*rB[i];
		
		// dir 8
		evel = u[i] - v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+8] = feq*rA[i];
		fB[offst+8] = feq*rB[i];				
	}
}



// --------------------------------------------------------
// D2Q9 compute velocity (barycentric) for the system.
// Here, the fluid velocity is calculated as normal, but
// it is amended to match the particle velocity.
// --------------------------------------------------------

__global__ void mcmp_compute_velocity_dip_D2Q9(float* fA,
                                               float* fB,
										       float* rA,
										       float* rB,
											   float* rS,
										       float* FxA,
										       float* FxB,
										       float* FyA,
										       float* FyB,
										       float* u,
										       float* v,											   
											   particle2D_dip* pt,											   
											   int* pIDgrid,
										       int nVoxels) 
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		// barycentric velocity		
		int offst = i*9;			
		float uA = fA[offst+1] + fA[offst+5] + fA[offst+8] - (fA[offst+3] + fA[offst+6] + fA[offst+7]) + 0.5*FxA[i];
		float uB = fB[offst+1] + fB[offst+5] + fB[offst+8] - (fB[offst+3] + fB[offst+6] + fB[offst+7]) + 0.5*FxB[i];
		float vA = fA[offst+2] + fA[offst+5] + fA[offst+6] - (fA[offst+4] + fA[offst+7] + fA[offst+8]) + 0.5*FyA[i];
		float vB = fB[offst+2] + fB[offst+5] + fB[offst+6] - (fB[offst+4] + fB[offst+7] + fB[offst+8]) + 0.5*FyB[i];
		float rTotal = rA[i] + rB[i];
		u[i] = (uA + uB)/rTotal;
		v[i] = (vA + vB)/rTotal;
		// modification due to particles
		int pID = pIDgrid[i];
		if (pID > -1) {
			float partvx = pt[pID].v.x;
			float partvy = pt[pID].v.y;
			float partfx = (partvx - u[i])*2.0*rTotal*rS[i];
			float partfy = (partvy - v[i])*2.0*rTotal*rS[i];
			// ammend fluid velocity
			u[i] += 0.5*partfx/rTotal;
			v[i] += 0.5*partfy/rTotal;
			// ammend fluid forces
			FxA[i] += partfx*(rA[i]/rTotal); 
			FxB[i] += partfx*(rB[i]/rTotal);
			FyA[i] += partfy*(rA[i]/rTotal);
			FyB[i] += partfy*(rB[i]/rTotal);
			// ammend particle forces  (AtomicAdd!)
			atomicAdd(&pt[pID].f.x, -partfx);
			atomicAdd(&pt[pID].f.y, -partfy);			
		}							
	}
}



// --------------------------------------------------------
// D2Q9 compute velocity (barycentric) for the system.
// Here, the fluid velocity is calculated by incorporating
// the particle velocity in the weighted sum.   
// -------------------------------------------------------- 

__global__ void mcmp_compute_velocity_dip_2_D2Q9(float* fA,
                                                 float* fB,
										         float* rA,
										         float* rB,
											     float* rS,
										         float* FxA,
										         float* FxB,
										         float* FyA,
										         float* FyB,
										         float* u,
										         float* v,											   
											     particle2D_dip* pt,											   
											     int* pIDgrid,
										         int nVoxels) 
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		// barycentric velocity		
		int offst = i*9;			
		float uA = fA[offst+1] + fA[offst+5] + fA[offst+8] - (fA[offst+3] + fA[offst+6] + fA[offst+7]) + 0.5*FxA[i];
		float uB = fB[offst+1] + fB[offst+5] + fB[offst+8] - (fB[offst+3] + fB[offst+6] + fB[offst+7]) + 0.5*FxB[i];
		float vA = fA[offst+2] + fA[offst+5] + fA[offst+6] - (fA[offst+4] + fA[offst+7] + fA[offst+8]) + 0.5*FyA[i];
		float vB = fB[offst+2] + fB[offst+5] + fB[offst+6] - (fB[offst+4] + fB[offst+7] + fB[offst+8]) + 0.5*FyB[i];
		float rTotal = rA[i] + rB[i] + rS[i];
		// include contribution from particles:
		float rSVx = 0.0;
		float rSVy = 0.0;
		int pID = pIDgrid[i]; 
		if (pID >= 0) {
			rSVx = rS[i]*pt[pID].v.x;
			rSVy = rS[i]*pt[pID].v.y;
		}
		u[i] = (uA + uB + rSVx)/rTotal;
		v[i] = (vA + vB + rSVy)/rTotal;		
		// add force to particles:
		if (pID > -1) {
			float pFx = 2.0*rS[i]*(u[i] - pt[pID].v.x);
			float pFy = 2.0*rS[i]*(v[i] - pt[pID].v.y);
			atomicAdd(&pt[pID].f.x, pFx);
			atomicAdd(&pt[pID].f.y, pFy);
		}
	}
}



// --------------------------------------------------------
// D2Q9 compute density for each component: 
// --------------------------------------------------------

__global__ void mcmp_compute_density_dip_D2Q9(float* fA,
                                        	  float* fB,
										      float* rA,
										      float* rB,
										      int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		int offst = i*9;			
		rA[i] = fA[offst] + fA[offst+1] + fA[offst+2] + fA[offst+3] + fA[offst+4] + fA[offst+5] + fA[offst+6] +
		        fA[offst+7] + fA[offst+8];
		rB[i] = fB[offst] + fB[offst+1] + fB[offst+2] + fB[offst+3] + fB[offst+4] + fB[offst+5] + fB[offst+6] +
		        fB[offst+7] + fB[offst+8];
	}
}



// --------------------------------------------------------
// D2Q9 compute Shan-Chen forces for the components
// using pseudo-potential, psi = rho_0(1-exp(-rho/rho_o))
// --------------------------------------------------------

__global__ void mcmp_compute_SC_forces_dip_D2Q9(float* rA,
										        float* rB,
												float* rS,
										        float* FxA,
										        float* FxB,
										        float* FyA,
										        float* FyB,
												float* pfx,
												float* pfy,
												particle2D_dip* pt,
											    int* nList,
												int* pIDgrid,												
											    float gAB,	
												float gAS,
												float gBS,
												float omega,										    
										        int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
				
		// index for nList[]
		int offst = i*9;
		
		// values of neighbor psi
		float p0A = psi(rA[i]);				
		float p1A = psi(rA[nList[offst+1]]);
		float p2A = psi(rA[nList[offst+2]]);
		float p3A = psi(rA[nList[offst+3]]);
		float p4A = psi(rA[nList[offst+4]]);
		float p5A = psi(rA[nList[offst+5]]);
		float p6A = psi(rA[nList[offst+6]]);
		float p7A = psi(rA[nList[offst+7]]);
		float p8A = psi(rA[nList[offst+8]]);
		
		float p0B = psi(rB[i]);				
		float p1B = psi(rB[nList[offst+1]]);
		float p2B = psi(rB[nList[offst+2]]);
		float p3B = psi(rB[nList[offst+3]]);
		float p4B = psi(rB[nList[offst+4]]);
		float p5B = psi(rB[nList[offst+5]]);
		float p6B = psi(rB[nList[offst+6]]);
		float p7B = psi(rB[nList[offst+7]]);
		float p8B = psi(rB[nList[offst+8]]);
		
		float r0S = rS[i];
		float r1S = rS[nList[offst+1]];
		float r2S = rS[nList[offst+2]];
		float r3S = rS[nList[offst+3]];
		float r4S = rS[nList[offst+4]];
		float r5S = rS[nList[offst+5]];
		float r6S = rS[nList[offst+6]];
		float r7S = rS[nList[offst+7]];
		float r8S = rS[nList[offst+8]];
		
		float p0SA = r0S + omega*r0S*(1.0-r0S);		
		float p1SA = r1S + omega*r1S*(1.0-r1S);
		float p2SA = r2S + omega*r2S*(1.0-r2S);
		float p3SA = r3S + omega*r3S*(1.0-r3S);
		float p4SA = r4S + omega*r4S*(1.0-r4S);
		float p5SA = r5S + omega*r5S*(1.0-r5S);
		float p6SA = r6S + omega*r6S*(1.0-r6S);
		float p7SA = r7S + omega*r7S*(1.0-r7S);
		float p8SA = r8S + omega*r8S*(1.0-r8S);
		
		float p0SB = r0S - omega*r0S*(1.0-r0S);
		float p1SB = r1S - omega*r1S*(1.0-r1S);
		float p2SB = r2S - omega*r2S*(1.0-r2S);
		float p3SB = r3S - omega*r3S*(1.0-r3S);
		float p4SB = r4S - omega*r4S*(1.0-r4S);
		float p5SB = r5S - omega*r5S*(1.0-r5S);
		float p6SB = r6S - omega*r6S*(1.0-r6S);
		float p7SB = r7S - omega*r7S*(1.0-r7S);
		float p8SB = r8S - omega*r8S*(1.0-r8S);
		
		// sum neighbor psi values times wi times ei
		float ws = 1.0/9.0;
		float wd = 1.0/36.0;		
		float sumNbrPsiAx = ws*p1A + wd*p5A + wd*p8A - (ws*p3A + wd*p6A + wd*p7A);
		float sumNbrPsiAy = ws*p2A + wd*p5A + wd*p6A - (ws*p4A + wd*p7A + wd*p8A);
		float sumNbrPsiBx = ws*p1B + wd*p5B + wd*p8B - (ws*p3B + wd*p6B + wd*p7B);
		float sumNbrPsiBy = ws*p2B + wd*p5B + wd*p6B - (ws*p4B + wd*p7B + wd*p8B);
		float sumNbrPsiSAx = ws*p1SA + wd*p5SA + wd*p8SA - (ws*p3SA + wd*p6SA + wd*p7SA);
		float sumNbrPsiSBx = ws*p1SB + wd*p5SB + wd*p8SB - (ws*p3SB + wd*p6SB + wd*p7SB);
		float sumNbrPsiSAy = ws*p2SA + wd*p5SA + wd*p6SA - (ws*p4SA + wd*p7SA + wd*p8SA);
		float sumNbrPsiSBy = ws*p2SB + wd*p5SB + wd*p6SB - (ws*p4SB + wd*p7SB + wd*p8SB);
		
		// fluid forces
		FxA[i] = -p0A*(gAB*sumNbrPsiBx + gAS*sumNbrPsiSAx);
		FxB[i] = -p0B*(gAB*sumNbrPsiAx + gBS*sumNbrPsiSBx);
		FyA[i] = -p0A*(gAB*sumNbrPsiBy + gAS*sumNbrPsiSAy);
		FyB[i] = -p0B*(gAB*sumNbrPsiAy + gBS*sumNbrPsiSBy);
		
		// particle forces
		pfx[i] = 0.0;
		pfy[i] = 0.0;
		int pID = pIDgrid[i];
		if (pID > -1) {
			float FxS = -(p0SA*gAS*sumNbrPsiAx + p0SB*gBS*sumNbrPsiBx);
			float FyS = -(p0SA*gAS*sumNbrPsiAy + p0SB*gBS*sumNbrPsiBy);
			atomicAdd(&pt[pID].f.x, FxS);
			atomicAdd(&pt[pID].f.y, FyS);
			pfx[i] = FxS;
			pfy[i] = FyS;
		}
								
	}
}



// --------------------------------------------------------
// D2Q9 update kernel:
// --------------------------------------------------------

__global__ void mcmp_collide_stream_dip_D2Q9(float* f1A,
                                         	 float* f1B,
										 	 float* f2A,
										 	 float* f2B,
										 	 float* rA,
										 	 float* rB,
										 	 float* u,
										 	 float* v,
										 	 float* FxA,
										 	 float* FxB,
										 	 float* FyA,
										 	 float* FyB,											 
										 	 int* streamIndex,											 
										 	 float nu,
										 	 int nVoxels)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
				
		// --------------------------------------------------	
		// COLLISION & STREAMING - standard BGK operator with
		//                         a PUSH propagator.  This step
		//                         includes the Guo forcing
		//                         scheme applied to the Shan-Chen
		//                         MCMP model according to Kruger et al.
		// --------------------------------------------------
				
		// useful constants
		int offst = 9*i;
		const float w0 = 4.0/9.0;
		const float ws = 1.0/9.0;
		const float wd = 1.0/36.0;		
		const float omega = 2.0/(6.0*nu + 1.0);   // 1/tau
		const float omomega = 1.0 - omega;        // 1 - 1/tau
		const float omomega2 = 1.0 - 0.5*omega;   // 1 - 1/(2tau)
		const float omusq = 1.0 - 1.5*(u[i]*u[i] + v[i]*v[i]);
		const float ux = u[i];
		const float vy = v[i];
										
		// direction 0
		float evel = 0.0;       // e dot velocity
		float emiu = 0.0-ux;    // e minus u
		float emiv = 0.0-vy;    // e minus v
		float feq = w0*omusq;
		float frcA = w0*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv) );
		float frcB = w0*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv) );		
		f2A[streamIndex[offst+0]] = omomega*f1A[offst+0] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+0]] = omomega*f1B[offst+0] + omega*feq*rB[i] + omomega2*frcB;
				
		// direction 1
		evel = ux;
		emiu = 1.0-ux;
		emiv = 0.0-vy;
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = ws*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv) );
		frcB = ws*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv) );
		f2A[streamIndex[offst+1]] = omomega*f1A[offst+1] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+1]] = omomega*f1B[offst+1] + omega*feq*rB[i] + omomega2*frcB;
		
		// direction 2
		evel = vy; 
		emiu = 0.0-ux;
		emiv = 1.0-vy;
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = ws*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv + 9.0*evel) );
		frcB = ws*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv + 9.0*evel) );
		f2A[streamIndex[offst+2]] = omomega*f1A[offst+2] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+2]] = omomega*f1B[offst+2] + omega*feq*rB[i] + omomega2*frcB;
				
		// direction 3
		evel = -ux;
		emiu = -1.0-ux;
		emiv =  0.0-vy;
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = ws*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv) );
		frcB = ws*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv) );		
		f2A[streamIndex[offst+3]] = omomega*f1A[offst+3] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+3]] = omomega*f1B[offst+3] + omega*feq*rB[i] + omomega2*frcB;
		
		// direction 4
		evel = -vy;
		emiu =  0.0-ux;
		emiv = -1.0-vy;
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = ws*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv - 9.0*evel) );
		frcB = ws*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv - 9.0*evel) );
		f2A[streamIndex[offst+4]] = omomega*f1A[offst+4] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+4]] = omomega*f1B[offst+4] + omega*feq*rB[i] + omomega2*frcB;
		
		// direction 5
		evel = ux + vy;
		emiu = 1.0-ux;
		emiv = 1.0-vy;
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = wd*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv + 9.0*evel) );
		frcB = wd*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv + 9.0*evel) );
		f2A[streamIndex[offst+5]] = omomega*f1A[offst+5] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+5]] = omomega*f1B[offst+5] + omega*feq*rB[i] + omomega2*frcB;
		
		// direction 6
		evel = -ux + vy;
		emiu = -1.0-ux;
		emiv =  1.0-vy;
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = wd*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv + 9.0*evel) );
		frcB = wd*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv + 9.0*evel) );		
		f2A[streamIndex[offst+6]] = omomega*f1A[offst+6] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+6]] = omomega*f1B[offst+6] + omega*feq*rB[i] + omomega2*frcB;
		
		// direction 7
		evel = -ux - vy;
		emiu = -1.0-ux;
		emiv = -1.0-vy;
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = wd*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv - 9.0*evel) );
		frcB = wd*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv - 9.0*evel) );		
		f2A[streamIndex[offst+7]] = omomega*f1A[offst+7] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+7]] = omomega*f1B[offst+7] + omega*feq*rB[i] + omomega2*frcB;
		
		// direction 8
		evel = ux - vy;
		emiu =  1.0-ux;
		emiv = -1.0-vy;
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		frcA = wd*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv - 9.0*evel) );
		frcB = wd*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv - 9.0*evel) );		
		f2A[streamIndex[offst+8]] = omomega*f1A[offst+8] + omega*feq*rA[i] + omomega2*frcA;
		f2B[streamIndex[offst+8]] = omomega*f1B[offst+8] + omega*feq*rB[i] + omomega2*frcB;		
		
	}
}



