
# include "particles2D.cuh"
# include "../../IO/GetPot"
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

particles2D::particles2D()
{
	GetPot inputParams("input.dat");	
	nVoxels = inputParams("Lattice/nVoxels",0);
	nParts = inputParams("Particles/nParts",0);	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

particles2D::~particles2D()
{
		
}



// --------------------------------------------------------
// Allocate arrays:
// --------------------------------------------------------

void particles2D::allocate()
{
	// allocate array memory (host):
    xH = (float*)malloc(nParts*sizeof(float));
	yH = (float*)malloc(nParts*sizeof(float));
	radH = (float*)malloc(nParts*sizeof(float));
				
	// allocate array memory (device):
	hipMalloc((void **) &x, nParts*sizeof(float));
	hipMalloc((void **) &y, nParts*sizeof(float));
	hipMalloc((void **) &vx, nParts*sizeof(float));
	hipMalloc((void **) &vy, nParts*sizeof(float));
	hipMalloc((void **) &fx, nParts*sizeof(float));
	hipMalloc((void **) &fy, nParts*sizeof(float));
	hipMalloc((void **) &rad, nParts*sizeof(float));
	hipMalloc((void **) &pIDgrid, nVoxels*sizeof(int));	
}



// --------------------------------------------------------
// Deallocate arrays:
// --------------------------------------------------------

void particles2D::deallocate()
{
	// free array memory (host):
	free(xH);
	free(yH);
	free(radH);
				
	// free array memory (device):
	hipFree(x);
	hipFree(y);
	hipFree(vx);
	hipFree(vy);
	hipFree(fx);
	hipFree(fy);	
	hipFree(rad);
	hipFree(pIDgrid);
}



// --------------------------------------------------------
// Copy arrays from host to device:
// --------------------------------------------------------

void particles2D::memcopy_host_to_device()
{
    hipMemcpy(x, xH, sizeof(float)*nParts, hipMemcpyHostToDevice);
	hipMemcpy(y, yH, sizeof(float)*nParts, hipMemcpyHostToDevice);
	hipMemcpy(rad, radH, sizeof(float)*nParts, hipMemcpyHostToDevice);
}



// --------------------------------------------------------
// Copy arrays from device to host:
// --------------------------------------------------------

void particles2D::memcopy_device_to_host()
{
    hipMemcpy(xH, x, sizeof(float)*nParts, hipMemcpyDeviceToHost);
	hipMemcpy(yH, y, sizeof(float)*nParts, hipMemcpyDeviceToHost);
}



// --------------------------------------------------------
// Wrtie output:
// --------------------------------------------------------

void particles2D::write_output(std::string tagname, int step)
{
	
}








