#include "hip/hip_runtime.h"

# include "mcmp_compute_density_psm_D2Q9.cuh"
# include <stdio.h>

// --------------------------------------------------------
// D2Q9 compute density for each component: 
// --------------------------------------------------------

__global__ void mcmp_compute_density_psm_D2Q9(float* fA,
                                        	  float* fB,
										      float* rA,
										      float* rB,
										      int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		int offst = i*9;			
		rA[i] = fA[offst] + fA[offst+1] + fA[offst+2] + fA[offst+3] + fA[offst+4] + fA[offst+5] + fA[offst+6] +
		        fA[offst+7] + fA[offst+8];
		rB[i] = fB[offst] + fB[offst+1] + fB[offst+2] + fB[offst+3] + fB[offst+4] + fB[offst+5] + fB[offst+6] +
		        fB[offst+7] + fB[offst+8];
	}
}