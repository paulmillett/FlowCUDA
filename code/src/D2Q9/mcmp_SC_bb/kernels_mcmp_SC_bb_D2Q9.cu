#include "hip/hip_runtime.h"

# include "kernels_mcmp_SC_bb_D2Q9.cuh"
# include "../mcmp_SC/mcmp_pseudopotential.cuh"
# include <stdio.h>



// --------------------------------------------------------
// Zero particle forces:
// --------------------------------------------------------

__global__ void mcmp_zero_particle_forces_bb_D2Q9(particle2D_bb* pt,
							                      int nParts)
{
	// define particle:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nParts) {		
		pt[i].f = make_float2(0.0);
	}
}



// --------------------------------------------------------
// Update particle velocities and positions:
// --------------------------------------------------------

__global__ void mcmp_move_particles_bb_D2Q9(particle2D_bb* pt,
   								            int nParts)
{
	// define particle:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nParts) {	
		printf("particle force-x = %f \n",pt[i].f.x); 
		float2 a = pt[i].f/pt[i].mass;
		pt[i].r += pt[i].v + 0.5*a;  // assume dt = 1
		pt[i].v += a;
	}
}



// --------------------------------------------------------
// Fix particle velocity:
// --------------------------------------------------------

__global__ void mcmp_fix_particle_velocity_bb_D2Q9(particle2D_bb* pt,
                                                   float pvel,
   								                   int nParts)
{
	// define particle:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nParts) {	
		pt[i].f = make_float2(0.0);
		if (i == 0) {
			pt[i].v.x = -pvel;
			pt[i].v.y = 0.00;
		}
		if (i == 1) {
			pt[i].v.x = pvel;
			pt[i].v.y = 0.00;
		}		
	}
}



// --------------------------------------------------------
// Calculate particle-particle forces:
// --------------------------------------------------------

__global__ void mcmp_particle_particle_forces_bb_D2Q9(particle2D_bb* pt,
                                                      float K,
													  float halo,
   								                      int nParts)
{
	// define particle:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nParts) {	
		for (int j=0; j<nParts; j++) {
			if (i==j) continue;
			float2 rij = pt[i].r - pt[j].r;
			float rr = length(rij);
			// Hertz contact force:
			float twoRadii = pt[i].rad + pt[j].rad + halo;
			if (rr < twoRadii) {
				float fmag = 2.5*K*pow(twoRadii - rr,1.5);
				pt[i].f += fmag*(rij/rr);
			}
		}		
	}
}



// --------------------------------------------------------
// D2Q9 initialize populations to equilibrium values: 
// --------------------------------------------------------

__global__ void mcmp_initial_equilibrium_bb_D2Q9(float* fA,
                                                 float* fB,
										         float* rA,
											     float* rB,
										         float* u,
										         float* v,
										         int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {			
		int offst = 9*i;
		equilibrium_populations_bb_D2Q9(fA,fB,rA[i],rB[i],u[i],v[i],offst);		
	}
}



// --------------------------------------------------------
// D2Q9 equilibirium populations: 
// --------------------------------------------------------

__device__ void equilibrium_populations_bb_D2Q9(float* fA,
                                                float* fB,
										        float rA,
											    float rB,
										        float u,
										        float v,
												int offst)
{
	const float w0 = 4.0/9.0;
	const float ws = 1.0/9.0;
	const float wd = 1.0/36.0;
	const float omusq = 1.0 - 1.5*(u*u + v*v);	
	// dir 0
	float feq = w0*omusq;
	fA[offst+0] = feq*rA;
	fB[offst+0] = feq*rB;	
	// dir 1
	float evel = u;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+1] = feq*rA;
	fB[offst+1] = feq*rB;	
	// dir 2
	evel = v;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+2] = feq*rA;
	fB[offst+2] = feq*rB;	
	// dir 3
	evel = -u;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+3] = feq*rA;
	fB[offst+3] = feq*rB;	
	// dir 4
	evel = -v;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+4] = feq*rA;
	fB[offst+4] = feq*rB;	
	// dir 5
	evel = u + v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+5] = feq*rA;
	fB[offst+5] = feq*rB;	
	// dir 6
	evel = -u + v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+6] = feq*rA;
	fB[offst+6] = feq*rB;	
	// dir 7
	evel = -u - v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+7] = feq*rA;
	fB[offst+7] = feq*rB;	
	// dir 8
	evel = u - v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	fA[offst+8] = feq*rA;
	fB[offst+8] = feq*rB;
}



// --------------------------------------------------------
// Map particles to grid by updating s[] and pIDgrid[]:
// --------------------------------------------------------

__global__ void mcmp_map_particles_on_lattice_bb_D2Q9(particle2D_bb* pt,
                                                      int* x,
						    		                  int* y,
												      int* s,
													  int* sprev,									   
									                  int* pIDgrid,													   
									                  int nVoxels,
									                  int nParts)
{
	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nVoxels) {
		// set previous "s" value:
		sprev[i] = s[i];
		// default values:		
		s[i] = 0;
		pIDgrid[i] = -1;
		// loop over particles:
		for (int j=0; j<nParts; j++) {
			float dx = float(x[i]) - pt[j].r.x;
			float dy = float(y[i]) - pt[j].r.y; 
			float rr = sqrt(dx*dx + dy*dy);
			if (rr <= pt[j].rad) {
				s[i] = 1;
				pIDgrid[i] = j;					
			}		
		}							
	}
}



// --------------------------------------------------------
// D2Q9 kernel to cover/uncover voxels as particles move: 
// --------------------------------------------------------

__global__ void mcmp_cover_uncover_bb_D2Q9(int* s,
                                           int* sprev,
										   int* nList,
										   float* u,
										   float* v,
										   float* rA,
										   float* rB,
										   float* fA,
										   float* fB,
										   int nVoxels)
{
	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nVoxels) {
		if (sprev[i] == 1 && s[i] == 1) stay_covered_D2Q9(i,fA,fB);		
		if (sprev[i] == 0 && s[i] == 1) cover_voxel_D2Q9(i,s,sprev,nList,u,v,rA,rB,fA,fB);
		if (sprev[i] == 1 && s[i] == 0) uncover_voxel_D2Q9(i,s,sprev,nList,u,v,rA,rB,fA,fB);			
	}
}



// --------------------------------------------------------
// D2Q9 kernel to keep solid site populations zero: 
// --------------------------------------------------------
	
__device__ void stay_covered_D2Q9(int i,
                                  float* fA,
								  float* fB)
{
	int offst = 9*i;		
	for (int n=0; n<9; n++) {
		fA[offst+n] = 0.0;
		fB[offst+n] = 0.0;
	}
}



// --------------------------------------------------------
// D2Q9 kernel to cover lattice site: 
// --------------------------------------------------------

__device__ void cover_voxel_D2Q9(int i,
                                 int* s,
								 int* sprev,
								 int* nList,
								 float* u,
								 float* v,
								 float* rA,
								 float* rB,
								 float* fA,
								 float* fB)
{
	
	// --------------------------------------------	
	// sum up all the neighbors that are fluid:
	// --------------------------------------------
	
	int offst = 9*i;
	
	/*
	int nfn = 0;	
	for (int n=1; n<9; n++) {
		int nabor = nList[offst+n];
		if (sprev[nabor] == 0 && s[nabor] == 0) nfn++;
	}
	
	// --------------------------------------------	
	// determine the density to distribute to each
	// neighbor:
	// --------------------------------------------
	
	float rAdist = 0.0;
	float rBdist = 0.0;
	if (nfn > 0) {
		rAdist = rA[i]/float(nfn);
		rBdist = rB[i]/float(nfn);
	}
	
	// --------------------------------------------	
	// add current voxel's density to neighboring
	// fluid voxel densities:
	// --------------------------------------------
		
	for (int n=1; n<9; n++) {
		int nabor = nList[offst+n];
		if (sprev[nabor] == 0 && s[nabor] == 0) {
			add_density_to_populations_D2Q9(nabor,rAdist,rBdist,u[nabor],v[nabor],fA,fB);
		}
	}
	*/
	
	// --------------------------------------------	
	// zero the populations for this voxel:
	// --------------------------------------------
	
	for (int n=0; n<9; n++) {
		fA[offst+n] = 0.0;
		fB[offst+n] = 0.0;
	}
		
}



// --------------------------------------------------------
// D2Q9 kernel to cover lattice site: 
// --------------------------------------------------------

__device__ void uncover_voxel_D2Q9(int i,
                                   int* s,
								   int* sprev,
								   int* nList,
								   float* u,
								   float* v,
								   float* rA,
								   float* rB,
								   float* fA,
								   float* fB)
{
	
	// --------------------------------------------	
	// sum up all the neighbors that are fluid:
	// --------------------------------------------
	
	int nfn = 0;
	int offst = 9*i;
	float avenbrRA = 0.0;
	float avenbrRB = 0.0;
	for (int n=1; n<9; n++) {
		int nabor = nList[offst+n];
		if (sprev[nabor] == 0 && s[nabor] == 0) {
			nfn++;
			avenbrRA += rA[nabor];
			avenbrRB += rB[nabor];
		}
	}
	avenbrRA /= float(nfn);
	avenbrRB /= float(nfn);
	
	// --------------------------------------------	
	// assign the equilibrium populations:
	// --------------------------------------------
	
	equilibrium_populations_bb_D2Q9(fA,fB,avenbrRA,avenbrRB,u[i],v[i],offst);
	
	// --------------------------------------------	
	// reduce neighboring fluid densities to
	// conserve mass:
	// --------------------------------------------
	
	/*
	float rAdist = -avenbrRA/float(nfn);
	float rBdist = -avenbrRB/float(nfn);
	for (int n=1; n<9; n++) {
		int nabor = nList[offst+n];
		if (sprev[nabor] == 0 && s[nabor] == 0) {
			add_density_to_populations_D2Q9(nabor,rAdist,rBdist,u[nabor],v[nabor],fA,fB);
		}
	}
	*/
	
}



// --------------------------------------------------------
// D2Q9 kernel to add density to populations: 
// --------------------------------------------------------

__device__ void add_density_to_populations_D2Q9(int i,
                                                float rAdist,
												float rBdist,
												float u,
												float v,
												float* fA,
												float* fB)
{
	const int offst = 9*i;
	const float w0 = 4.0/9.0;
	const float ws = 1.0/9.0;
	const float wd = 1.0/36.0;	
	const float omusq = 1.0 - 1.5*(u*u + v*v);
	// dir 0
	float feq = w0*omusq;
	atomicAdd(&fA[offst+0], feq*rAdist);
	atomicAdd(&fB[offst+0], feq*rBdist);
	// dir 1
	float evel = u;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+1], feq*rAdist);
	atomicAdd(&fB[offst+1], feq*rBdist);
	// dir 2
	evel = v;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+2], feq*rAdist);
	atomicAdd(&fB[offst+2], feq*rBdist);
	// dir 3
	evel = -u;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+3], feq*rAdist);
	atomicAdd(&fB[offst+3], feq*rBdist);
	// dir 4
	evel = -v;
	feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+4], feq*rAdist);
	atomicAdd(&fB[offst+4], feq*rBdist);
	// dir 5
	evel = u + v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+5], feq*rAdist);
	atomicAdd(&fB[offst+5], feq*rBdist);
	// dir 6
	evel = -u + v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+6], feq*rAdist);
	atomicAdd(&fB[offst+6], feq*rBdist);
	// dir 7
	evel = -u - v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+7], feq*rAdist);
	atomicAdd(&fB[offst+7], feq*rBdist);
	// dir 8
	evel = u - v;
	feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
	atomicAdd(&fA[offst+8], feq*rAdist);	
	atomicAdd(&fB[offst+8], feq*rBdist);
}



// --------------------------------------------------------
// D2Q9 kernel to update the particle fields on the lattice: 
// --------------------------------------------------------

__global__ void mcmp_update_particles_on_lattice_D2Q9(float* fA,
                                                      float* fB,
										              float* rA,
											          float* rB,
										              float* u,
										              float* v,
													  particle2D_bb* pt,
													  int* x,
													  int* y,
													  int* s,
													  int* pIDgrid,
													  int* nList,													  
										              int nVoxels,
													  int nParts)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {	
		
		int offst = 9*i;
		
		// --------------------------------------------	
		// get the current state of voxel:
		// --------------------------------------------
		
		int s0 = s[i];
		
		// --------------------------------------------	
		// get the new state of voxel by seeing if any
		// particles overlay it:
		// --------------------------------------------
				
		int s1 = 0;
		int pID = -1;
		float partvx = 0.0;
		float partvy = 0.0;
		for (int p=0; p<nParts; p++) {
			float dx = float(x[i]) - pt[p].r.x;
			float dy = float(y[i]) - pt[p].r.y;
			float rp = sqrt(dx*dx + dy*dy);
			if (rp <= pt[p].rad) {
				s1 = 1;
				pID = p;
				partvx = pt[p].v.x;
				partvy = pt[p].v.y;
			}		
		}
		s[i] = s1;
		
		// --------------------------------------------	
		// decide course of action:
		// --------------------------------------------
		
		// fluid site STAYS fluid site
		if (s0 == 0 && s1 == 0) {
			pIDgrid[i] = -1;  // this is redundant, but that's OK
		}
		
		// particle site STAYS particle site
		else if (s0 == 1 && s1 == 1) {			
			u[i] = partvx;
			v[i] = partvy;
			pIDgrid[i] = pID;
			// zero all the populations			
			for (int n=0; n<9; n++) {
				fA[offst+n] = 0.0;
				fB[offst+n] = 0.0;
			}
		}
		
		// fluid site becomes particle site (COVERING)
		else if (s0 == 0 && s1 == 1) {						
			// update velocity to particle's velocity
			u[i] = partvx;
			v[i] = partvy;
			pIDgrid[i] = pID;
			// zero all the populations			
			for (int n=0; n<9; n++) {
				fA[offst+n] = 0.0;
				fB[offst+n] = 0.0;
			}
		}
		
		// particle site becomes fluid site (UNCOVERING)
		else if (s0 == 1 && s1 == 0) {				
			// assign voxel velocity with particle velocity
			int pID = pIDgrid[i];
			u[i] = pt[pID].v.x; 
			v[i] = pt[pID].v.y;
			pIDgrid[i] = -1;
			// get average density of surrounding fluid sites:
			int num_fluid_nabors = 0;
			float aver_rA_nabors = 0.0;
			float aver_rB_nabors = 0.0;			
			for (int n=1; n<9; n++) {  // do not include self, n=0
				int nID = nList[offst+n];
				if (s[nID] == 0) {
					num_fluid_nabors++;
					aver_rA_nabors += rA[nID];
					aver_rB_nabors += rB[nID];
				}
			}			
			if (num_fluid_nabors > 0) {
				rA[i] = aver_rA_nabors/num_fluid_nabors;
				rB[i] = aver_rB_nabors/num_fluid_nabors;
			}
			// set populations to the equilibrium for the given
			// velocity and density:
			equilibrium_populations_bb_D2Q9(fA,fB,rA[i],rB[i],u[i],v[i],offst);
			
		}				
	}
}



// --------------------------------------------------------
// D2Q9 compute velocity (barycentric) for the system: 
// --------------------------------------------------------

__global__ void mcmp_compute_velocity_bb_D2Q9(float* fA,
                                              float* fB,
										      float* rA,
										      float* rB,
										      float* FxA,
										      float* FxB,
										      float* FyA,
										      float* FyB,
										      float* u,
										      float* v,
											  int* s,
											  int* pIDgrid,
											  particle2D_bb* pt,
										      int nVoxels) 
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		
		// --------------------------------------------------	
		// if this is a fluid site:
		// --------------------------------------------------
		
		if (s[i] == 0) {
			int offst = i*9;			
			float uA = fA[offst+1] + fA[offst+5] + fA[offst+8] - (fA[offst+3] + fA[offst+6] + fA[offst+7]) + 0.5*FxA[i];
			float uB = fB[offst+1] + fB[offst+5] + fB[offst+8] - (fB[offst+3] + fB[offst+6] + fB[offst+7]) + 0.5*FxB[i];
			float vA = fA[offst+2] + fA[offst+5] + fA[offst+6] - (fA[offst+4] + fA[offst+7] + fA[offst+8]) + 0.5*FyA[i];
			float vB = fB[offst+2] + fB[offst+5] + fB[offst+6] - (fB[offst+4] + fB[offst+7] + fB[offst+8]) + 0.5*FyB[i];
			float rTotal = rA[i] + rB[i];
			u[i] = (uA + uB)/rTotal;
			v[i] = (vA + vB)/rTotal;	
		}
		
		// --------------------------------------------------	
		// if this is a solid site:
		// --------------------------------------------------
		
		else if (s[i] == 1) {
			int pID = pIDgrid[i];
			u[i] = pt[pID].v.x;
			v[i] = pt[pID].v.y;	
		}
					
	}
}



// --------------------------------------------------------
// D2Q9 set velocity on the y=0 and y=Ny-1 boundaries: 
// --------------------------------------------------------

__global__ void mcmp_set_boundary_velocity_bb_D2Q9(float uBC,
                                                   float vBC,
	                                               float* rA,
										           float* rB,
										           float* FxA,
										           float* FxB,
										           float* FyA,
										           float* FyB,
										           float* u,
										           float* v,
												   int* y,											        
											       int Ny,
										           int nVoxels) 
{
	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		if (y[i] == 0 || y[i] == Ny-1) {
			float rTotal = rA[i] + rB[i];
			float fxBC = (uBC - u[i])*2.0*rTotal;
			float fyBC = (vBC - v[i])*2.0*rTotal;
			u[i] += 0.5*fxBC/rTotal;
			v[i] += 0.5*fyBC/rTotal;
			FxA[i] += fxBC*(rA[i]/rTotal);
			FxB[i] += fxBC*(rB[i]/rTotal);
			FyA[i] += fyBC*(rA[i]/rTotal);
			FyB[i] += fyBC*(rB[i]/rTotal);
		}		
	}
}



// --------------------------------------------------------
// D2Q9 set shear velocity on the y=0 and y=Ny-1 boundaries: 
// --------------------------------------------------------

__global__ void mcmp_set_boundary_shear_velocity_bb_D2Q9(float uBot,
                                                         float uTop,
	                                                     float* rA,
										                 float* rB,
										                 float* FxA,
										                 float* FxB,
										                 float* FyA,
										                 float* FyB,
										                 float* u,
										                 float* v,
												         int* y,											        
											             int Ny,
										                 int nVoxels) 
{
	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		if (y[i] == 0) {
			float rTotal = rA[i] + rB[i];
			float fxBC = (uBot - u[i])*2.0*rTotal;
			float fyBC = (0.0  - v[i])*2.0*rTotal;
			u[i] += 0.5*fxBC/rTotal;
			v[i] += 0.5*fyBC/rTotal;
			FxA[i] += fxBC*(rA[i]/rTotal);
			FxB[i] += fxBC*(rB[i]/rTotal);
			FyA[i] += fyBC*(rA[i]/rTotal);
			FyB[i] += fyBC*(rB[i]/rTotal);
		} 
		if (y[i] == Ny-1) {
			float rTotal = rA[i] + rB[i];
			float fxBC = (uTop - u[i])*2.0*rTotal;
			float fyBC = (0.0  - v[i])*2.0*rTotal;
			u[i] += 0.5*fxBC/rTotal;
			v[i] += 0.5*fyBC/rTotal;
			FxA[i] += fxBC*(rA[i]/rTotal);
			FxB[i] += fxBC*(rB[i]/rTotal);
			FyA[i] += fyBC*(rA[i]/rTotal);
			FyB[i] += fyBC*(rB[i]/rTotal);
		}		
	}
}



// --------------------------------------------------------
// D2Q9 compute density for each component: 
// --------------------------------------------------------

__global__ void mcmp_compute_density_bb_D2Q9(float* fA,
                                        	 float* fB,
										     float* rA,
										     float* rB,											 
										     int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		int offst = i*9;			
		rA[i] = fA[offst] + fA[offst+1] + fA[offst+2] + fA[offst+3] + fA[offst+4] + fA[offst+5] + fA[offst+6] +
		        fA[offst+7] + fA[offst+8];
		rB[i] = fB[offst] + fB[offst+1] + fB[offst+2] + fB[offst+3] + fB[offst+4] + fB[offst+5] + fB[offst+6] +
		        fB[offst+7] + fB[offst+8];		
	}
}



// --------------------------------------------------------
// D2Q9 compute virtual density for each component: 
// --------------------------------------------------------

__global__ void mcmp_compute_virtual_density_bb_D2Q9(float* rAvirt,
                                        	         float* rBvirt,
										             float* rA,
										             float* rB,
													 int* s,
													 int* nList,
													 float omega,
										             int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		
		int offst = i*9;
		
		// fluid node:
		if (s[i] == 0) {
			rAvirt[i] = 0.0;
			rBvirt[i] = 0.0;
		}
		
		// solid node:
		if (s[i] == 1) {
			const float ws = 1.0/9.0;
			const float wd = 1.0/36.0;			
			float r1A = rA[nList[offst+1]];
			float r2A = rA[nList[offst+2]];
			float r3A = rA[nList[offst+3]];
			float r4A = rA[nList[offst+4]];
			float r5A = rA[nList[offst+5]];
			float r6A = rA[nList[offst+6]];
			float r7A = rA[nList[offst+7]];
			float r8A = rA[nList[offst+8]];	
			float r1B = rB[nList[offst+1]];
			float r2B = rB[nList[offst+2]];
			float r3B = rB[nList[offst+3]];
			float r4B = rB[nList[offst+4]];
			float r5B = rB[nList[offst+5]];
			float r6B = rB[nList[offst+6]];
			float r7B = rB[nList[offst+7]];
			float r8B = rB[nList[offst+8]];				
			float s1 = ws*(1 - s[nList[offst+1]]);
			float s2 = ws*(1 - s[nList[offst+2]]);
			float s3 = ws*(1 - s[nList[offst+3]]);
			float s4 = ws*(1 - s[nList[offst+4]]);
			float s5 = wd*(1 - s[nList[offst+5]]);
			float s6 = wd*(1 - s[nList[offst+6]]);
			float s7 = wd*(1 - s[nList[offst+7]]);
			float s8 = wd*(1 - s[nList[offst+8]]);			
			float sumRA = s1*r1A + s2*r2A + s3*r3A + s4*r4A + 
				          s5*r5A + s6*r6A + s7*r7A + s8*r8A;
			float sumRB = s1*r1B + s2*r2B + s3*r3B + s4*r4B + 
				          s5*r5B + s6*r6B + s7*r7B + s8*r8B;
			float sumWS = s1+s2+s3+s4+s5+s6+s7+s8;		
			if (sumWS > 0.0) {
				rAvirt[i] = sumRA/sumWS*(1.0+omega);
				rBvirt[i] = sumRB/sumWS*(1.0-omega);	
			}	
			else {
				rAvirt[i] = 0.0;
				rBvirt[i] = 0.0;	
			}
					
		}		
	}
}



// --------------------------------------------------------
// D2Q9 compute Shan-Chen forces for the components
// using pseudo-potential, psi = rho_0(1-exp(-rho/rho_o))
// --------------------------------------------------------

__global__ void mcmp_compute_SC_forces_bb_D2Q9(float* rA,
										       float* rB,											   
										       float* FxA,
										       float* FxB,
										       float* FyA,
										       float* FyB,
											   particle2D_bb* pt,
											   int* pIDgrid,
											   int* s,
											   int* nList,
											   float gAB,
											   float gAS,
											   float gBS,
										       int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
							
		int offst = i*9;
		
		float r0A = psi(rA[i]);	
		float r1A = psi(rA[nList[offst+1]]);
		float r2A = psi(rA[nList[offst+2]]);
		float r3A = psi(rA[nList[offst+3]]);
		float r4A = psi(rA[nList[offst+4]]);
		float r5A = psi(rA[nList[offst+5]]);
		float r6A = psi(rA[nList[offst+6]]);
		float r7A = psi(rA[nList[offst+7]]);
		float r8A = psi(rA[nList[offst+8]]);
	
		float r0B = psi(rB[i]);		
		float r1B = psi(rB[nList[offst+1]]);
		float r2B = psi(rB[nList[offst+2]]);
		float r3B = psi(rB[nList[offst+3]]);
		float r4B = psi(rB[nList[offst+4]]);
		float r5B = psi(rB[nList[offst+5]]);
		float r6B = psi(rB[nList[offst+6]]);
		float r7B = psi(rB[nList[offst+7]]);
		float r8B = psi(rB[nList[offst+8]]);
		
		float s1 = float(s[nList[offst+1]]);
		float s2 = float(s[nList[offst+2]]);
		float s3 = float(s[nList[offst+3]]);
		float s4 = float(s[nList[offst+4]]);
		float s5 = float(s[nList[offst+5]]);
		float s6 = float(s[nList[offst+6]]);
		float s7 = float(s[nList[offst+7]]);
		float s8 = float(s[nList[offst+8]]);
	
		const float ws = 1.0/9.0;
		const float wd = 1.0/36.0;		
		float sumNbrRhoAx = ws*r1A + wd*r5A + wd*r8A - (ws*r3A + wd*r6A + wd*r7A);
		float sumNbrRhoAy = ws*r2A + wd*r5A + wd*r6A - (ws*r4A + wd*r7A + wd*r8A);
		float sumNbrRhoBx = ws*r1B + wd*r5B + wd*r8B - (ws*r3B + wd*r6B + wd*r7B);
		float sumNbrRhoBy = ws*r2B + wd*r5B + wd*r6B - (ws*r4B + wd*r7B + wd*r8B);
		float sumNbrSx = ws*s1 + wd*s5 + wd*s8 - (ws*s3 + wd*s6 + wd*s7);
		float sumNbrSy = ws*s2 + wd*s5 + wd*s6 - (ws*s4 + wd*s7 + wd*s8);
		
		// --------------------------------------------	
		// if this is a fluid site:
		// --------------------------------------------
		
		if (s[i] == 0) {				
			FxA[i] = -r0A*(gAB*sumNbrRhoBx + gAS*sumNbrSx);
			FxB[i] = -r0B*(gAB*sumNbrRhoAx + gBS*sumNbrSx);
			FyA[i] = -r0A*(gAB*sumNbrRhoBy + gAS*sumNbrSy);
			FyB[i] = -r0B*(gAB*sumNbrRhoAy + gBS*sumNbrSy);						
		}
		
		// --------------------------------------------	
		// if this is a solid site:
		// --------------------------------------------
		
		if (s[i] == 1) {			
			float fxFS = -(gAS*sumNbrRhoAx + gBS*sumNbrRhoBx);
			float fyFS = -(gAS*sumNbrRhoAy + gBS*sumNbrRhoBy);
			int pID = pIDgrid[i];
			atomicAdd(&pt[pID].f.x, fxFS);
			atomicAdd(&pt[pID].f.y, fyFS);							
		}
		
	}
}



// --------------------------------------------------------
// D2Q9 compute Shan-Chen forces for the components
// using pseudo-potential, psi = rho_0(1-exp(-rho/rho_o))
//
// Note: here we use the virtual fluid as described in
//       Jansen & Harting, PRE, 83, 046707 (2011).
//
// --------------------------------------------------------

__global__ void mcmp_compute_SC_forces_bb_2_D2Q9(float* rA,
										         float* rB,	
												 float* rAvirt,
												 float* rBvirt,										   
										         float* FxA,
										         float* FxB,
										         float* FyA,
										         float* FyB,
											     particle2D_bb* pt,
											     int* pIDgrid,
											     int* s,
											     int* nList,
											     float gAB,											     
										         int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
							
		int offst = i*9;
		
		float r0A = psi(rA[i]);	
		float r1A = psi(rA[nList[offst+1]]);
		float r2A = psi(rA[nList[offst+2]]);
		float r3A = psi(rA[nList[offst+3]]);
		float r4A = psi(rA[nList[offst+4]]);
		float r5A = psi(rA[nList[offst+5]]);
		float r6A = psi(rA[nList[offst+6]]);
		float r7A = psi(rA[nList[offst+7]]);
		float r8A = psi(rA[nList[offst+8]]);
	
		float r0B = psi(rB[i]);		
		float r1B = psi(rB[nList[offst+1]]);
		float r2B = psi(rB[nList[offst+2]]);
		float r3B = psi(rB[nList[offst+3]]);
		float r4B = psi(rB[nList[offst+4]]);
		float r5B = psi(rB[nList[offst+5]]);
		float r6B = psi(rB[nList[offst+6]]);
		float r7B = psi(rB[nList[offst+7]]);
		float r8B = psi(rB[nList[offst+8]]);
		
		float r0AV = psi(rAvirt[i]);	
		float r1AV = psi(rAvirt[nList[offst+1]]);
		float r2AV = psi(rAvirt[nList[offst+2]]);
		float r3AV = psi(rAvirt[nList[offst+3]]);
		float r4AV = psi(rAvirt[nList[offst+4]]);
		float r5AV = psi(rAvirt[nList[offst+5]]);
		float r6AV = psi(rAvirt[nList[offst+6]]);
		float r7AV = psi(rAvirt[nList[offst+7]]);
		float r8AV = psi(rAvirt[nList[offst+8]]);
	
		float r0BV = psi(rBvirt[i]);		
		float r1BV = psi(rBvirt[nList[offst+1]]);
		float r2BV = psi(rBvirt[nList[offst+2]]);
		float r3BV = psi(rBvirt[nList[offst+3]]);
		float r4BV = psi(rBvirt[nList[offst+4]]);
		float r5BV = psi(rBvirt[nList[offst+5]]);
		float r6BV = psi(rBvirt[nList[offst+6]]);
		float r7BV = psi(rBvirt[nList[offst+7]]);
		float r8BV = psi(rBvirt[nList[offst+8]]);
	
		const float ws = 1.0/9.0;
		const float wd = 1.0/36.0;		
		float sumNbrRhoAx = ws*r1A + wd*r5A + wd*r8A - (ws*r3A + wd*r6A + wd*r7A);
		float sumNbrRhoAy = ws*r2A + wd*r5A + wd*r6A - (ws*r4A + wd*r7A + wd*r8A);
		float sumNbrRhoBx = ws*r1B + wd*r5B + wd*r8B - (ws*r3B + wd*r6B + wd*r7B);
		float sumNbrRhoBy = ws*r2B + wd*r5B + wd*r6B - (ws*r4B + wd*r7B + wd*r8B);		
		
		// --------------------------------------------	
		// if this is a fluid site:
		// --------------------------------------------
		
		if (s[i] == 0) {	
			float sumNbrRhoAVx = ws*r1AV + wd*r5AV + wd*r8AV - (ws*r3AV + wd*r6AV + wd*r7AV);
			float sumNbrRhoAVy = ws*r2AV + wd*r5AV + wd*r6AV - (ws*r4AV + wd*r7AV + wd*r8AV);
			float sumNbrRhoBVx = ws*r1BV + wd*r5BV + wd*r8BV - (ws*r3BV + wd*r6BV + wd*r7BV);
			float sumNbrRhoBVy = ws*r2BV + wd*r5BV + wd*r6BV - (ws*r4BV + wd*r7BV + wd*r8BV);				
			FxA[i] = -r0A*gAB*(sumNbrRhoBx + sumNbrRhoBVx);
			FxB[i] = -r0B*gAB*(sumNbrRhoAx + sumNbrRhoAVx);
			FyA[i] = -r0A*gAB*(sumNbrRhoBy + sumNbrRhoBVy);
			FyB[i] = -r0B*gAB*(sumNbrRhoAy + sumNbrRhoAVy);						
		}
		
		// --------------------------------------------	
		// if this is a solid site:
		// --------------------------------------------
		
		if (s[i] == 1) {
			float fxAV = -r0AV*gAB*(sumNbrRhoBx);
			float fxBV = -r0BV*gAB*(sumNbrRhoAx);
			float fyAV = -r0AV*gAB*(sumNbrRhoBy);
			float fyBV = -r0BV*gAB*(sumNbrRhoAy);		
			int pID = pIDgrid[i];
			atomicAdd(&pt[pID].f.x, fxAV + fxBV);
			atomicAdd(&pt[pID].f.y, fyAV + fyBV);							
		}
		
	}
}



// --------------------------------------------------------
// D2Q9 update kernel:
// --------------------------------------------------------

__global__ void mcmp_collide_stream_bb_D2Q9(float* f1A,
                                         	float* f1B,
										 	float* f2A,
										 	float* f2B,
										 	float* rA,
										 	float* rB,
										 	float* u,
										 	float* v,
										 	float* FxA,
										 	float* FxB,
										 	float* FyA,
										 	float* FyB,
											int* s,
										 	int* streamIndex,
										 	float nu,
										 	int nVoxels)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
		
		// --------------------------------------------------	
		// Only do collide & stream if "i" is a fluid node:
		// --------------------------------------------------
		
		if (s[i] == 0) {
			
			// --------------------------------------------------	
			// COLLISION & STREAMING - standard BGK operator with
			//                         a PUSH propagator.  This step
			//                         includes the Guo forcing
			//                         scheme applied to the Shan-Chen
			//                         MCMP model according to Kruger et al.
			// --------------------------------------------------
				
			// useful constants
			int offst = 9*i;
			const float w0 = 4.0/9.0;
			const float ws = 1.0/9.0;
			const float wd = 1.0/36.0;		
			const float omega = 2.0/(6.0*nu + 1.0);   // 1/tau
			const float omomega = 1.0 - omega;        // 1 - 1/tau
			const float omomega2 = 1.0 - 0.5*omega;   // 1 - 1/(2tau)
			const float omusq = 1.0 - 1.5*(u[i]*u[i] + v[i]*v[i]);
			const float ux = u[i];
			const float vy = v[i];
										
			// direction 0
			float evel = 0.0;       // e dot velocity
			float emiu = 0.0-ux;    // e minus u
			float emiv = 0.0-vy;    // e minus v
			float feq = w0*omusq;
			float frcA = w0*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv) );
			float frcB = w0*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv) );		
			f2A[streamIndex[offst+0]] = omomega*f1A[offst+0] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+0]] = omomega*f1B[offst+0] + omega*feq*rB[i] + omomega2*frcB;
				
			// direction 1
			evel = ux;
			emiu = 1.0-ux;
			emiv = 0.0-vy;
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = ws*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv) );
			frcB = ws*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv) );
			f2A[streamIndex[offst+1]] = omomega*f1A[offst+1] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+1]] = omomega*f1B[offst+1] + omega*feq*rB[i] + omomega2*frcB;
		
			// direction 2
			evel = vy; 
			emiu = 0.0-ux;
			emiv = 1.0-vy;
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = ws*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv + 9.0*evel) );
			frcB = ws*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv + 9.0*evel) );
			f2A[streamIndex[offst+2]] = omomega*f1A[offst+2] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+2]] = omomega*f1B[offst+2] + omega*feq*rB[i] + omomega2*frcB;
				
			// direction 3
			evel = -ux;
			emiu = -1.0-ux;
			emiv =  0.0-vy;
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = ws*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv) );
			frcB = ws*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv) );		
			f2A[streamIndex[offst+3]] = omomega*f1A[offst+3] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+3]] = omomega*f1B[offst+3] + omega*feq*rB[i] + omomega2*frcB;
		
			// direction 4
			evel = -vy;
			emiu =  0.0-ux;
			emiv = -1.0-vy;
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = ws*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv - 9.0*evel) );
			frcB = ws*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv - 9.0*evel) );
			f2A[streamIndex[offst+4]] = omomega*f1A[offst+4] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+4]] = omomega*f1B[offst+4] + omega*feq*rB[i] + omomega2*frcB;
		
			// direction 5
			evel = ux + vy;
			emiu = 1.0-ux;
			emiv = 1.0-vy;
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = wd*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv + 9.0*evel) );
			frcB = wd*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv + 9.0*evel) );
			f2A[streamIndex[offst+5]] = omomega*f1A[offst+5] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+5]] = omomega*f1B[offst+5] + omega*feq*rB[i] + omomega2*frcB;
		
			// direction 6
			evel = -ux + vy;
			emiu = -1.0-ux;
			emiv =  1.0-vy;
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = wd*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv + 9.0*evel) );
			frcB = wd*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv + 9.0*evel) );		
			f2A[streamIndex[offst+6]] = omomega*f1A[offst+6] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+6]] = omomega*f1B[offst+6] + omega*feq*rB[i] + omomega2*frcB;
		
			// direction 7
			evel = -ux - vy;
			emiu = -1.0-ux;
			emiv = -1.0-vy;
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = wd*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv - 9.0*evel) );
			frcB = wd*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv - 9.0*evel) );		
			f2A[streamIndex[offst+7]] = omomega*f1A[offst+7] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+7]] = omomega*f1B[offst+7] + omega*feq*rB[i] + omomega2*frcB;
		
			// direction 8
			evel = ux - vy;
			emiu =  1.0-ux;
			emiv = -1.0-vy;
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			frcA = wd*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv - 9.0*evel) );
			frcB = wd*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv - 9.0*evel) );		
			f2A[streamIndex[offst+8]] = omomega*f1A[offst+8] + omega*feq*rA[i] + omomega2*frcA;
			f2B[streamIndex[offst+8]] = omomega*f1B[offst+8] + omega*feq*rB[i] + omomega2*frcB;			
			
		}					
	}
}



// --------------------------------------------------------
// D2Q9 implement bounce-back conditions:
// --------------------------------------------------------

__global__ void mcmp_bounce_back_D2Q9(float* f2A, 
									  float* f2B,
									  int* s,
									  int* nList,									  
									  int* streamIndex,
									  int nVoxels)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
								
		// --------------------------------------------------	
		// If the current voxel is solid, then bounce-back
		// the populations just received via streaming 
		// back to the neighboring voxel:
		// --------------------------------------------------
		
		if (s[i] == 1) {
			
			int offst = 9*i;
						
			// dir 1 bounce-back to nabor 3 as dir 3:
			if (s[nList[offst+3]] == 0) {
				f2A[streamIndex[offst+3]] = f2A[offst+1];
				f2B[streamIndex[offst+3]] = f2B[offst+1];
				f2A[offst+1] = 0.0;
				f2B[offst+1] = 0.0;
			}
			
			// dir 2 bounce-back to nabor 4 as dir 4:
			if (s[nList[offst+4]] == 0) {
				f2A[streamIndex[offst+4]] = f2A[offst+2];
				f2B[streamIndex[offst+4]] = f2B[offst+2];
				f2A[offst+2] = 0.0;
				f2B[offst+2] = 0.0;
			}
			
			// dir 3 bounce-back to nabor 1 as dir 1:
			if (s[nList[offst+1]] == 0) {
				f2A[streamIndex[offst+1]] = f2A[offst+3];
				f2B[streamIndex[offst+1]] = f2B[offst+3];
				f2A[offst+3] = 0.0;
				f2B[offst+3] = 0.0;
			}
			
			// dir 4 bounce-back to nabor 2 as dir 2:
			if (s[nList[offst+2]] == 0) {
				f2A[streamIndex[offst+2]] = f2A[offst+4];
				f2B[streamIndex[offst+2]] = f2B[offst+4];
				f2A[offst+4] = 0.0;
				f2B[offst+4] = 0.0;
			}
			
			// dir 5 bounce-back to nabor 7 as dir 7:
			if (s[nList[offst+7]] == 0) {
				f2A[streamIndex[offst+7]] = f2A[offst+5];
				f2B[streamIndex[offst+7]] = f2B[offst+5];
				f2A[offst+5] = 0.0;
				f2B[offst+5] = 0.0;
			}
			
			// dir 6 bounce-back to nabor 8 as dir 8:
			if (s[nList[offst+8]] == 0) {
				f2A[streamIndex[offst+8]] = f2A[offst+6];
				f2B[streamIndex[offst+8]] = f2B[offst+6];
				f2A[offst+6] = 0.0;
				f2B[offst+6] = 0.0;
			}
			
			// dir 7 bounce-back to nabor 5 as dir 5:
			if (s[nList[offst+5]] == 0) {
				f2A[streamIndex[offst+5]] = f2A[offst+7];
				f2B[streamIndex[offst+5]] = f2B[offst+7];
				f2A[offst+7] = 0.0;
				f2B[offst+7] = 0.0;
			}
			
			// dir 8 bounce-back to nabor 6 as dir 6:
			if (s[nList[offst+6]] == 0) {
				f2A[streamIndex[offst+6]] = f2A[offst+8];
				f2B[streamIndex[offst+6]] = f2B[offst+8];
				f2A[offst+8] = 0.0;
				f2B[offst+8] = 0.0;
			}
			
		}	
	}		
}



// --------------------------------------------------------
// D2Q9 implement bounce-back conditions for moving
// solids:
// --------------------------------------------------------

__global__ void mcmp_bounce_back_moving_D2Q9(float* f2A, 
									         float* f2B,
											 float* rA,
											 float* rB,
											 float* u,
											 float* v,
											 particle2D_bb* pt,
											 int* pIDgrid,
									         int* s,
									         int* nList,									  
									         int* streamIndex,
									         int nVoxels)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
								
		// --------------------------------------------------	
		// If the current voxel is solid, then bounce-back
		// the populations just received via streaming 
		// back to the neighboring voxel:
		// --------------------------------------------------
		
		if (s[i] == 1) {
						
			int offst = 9*i;
			int pID = pIDgrid[i];
			const float ws = 1.0/9.0;
			const float wd = 1.0/36.0;
			float meF2S  = 0.0;  // momentum exchange fluid to solid
			float meF2Sx = 0.0;  // momentum exchange fluid to solid (x)
			float meF2Sy = 0.0;  // momentum exchange fluid to solid (y)
						
			// dir 1 bounce-back to nabor 3 as dir 3:
			if (s[nList[offst+3]] == 0) {
				// bounce-back
				float evel = u[i];
				f2A[streamIndex[offst+3]] = f2A[offst+1] - 6.0*ws*rA[nList[offst+3]]*evel;
				f2B[streamIndex[offst+3]] = f2B[offst+1] - 6.0*ws*rB[nList[offst+3]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+1] - 6.0*ws*rA[nList[offst+3]]*evel + 
					    2.0*f2B[offst+1] - 6.0*ws*rB[nList[offst+3]]*evel;
				meF2Sx = meF2S;
				meF2Sy = 0.0;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);
				// zero populations inside particle				
				f2A[offst+1] = 0.0;
				f2B[offst+1] = 0.0;				
			}
			
			// dir 2 bounce-back to nabor 4 as dir 4:
			if (s[nList[offst+4]] == 0) {
				// bounce-back
				float evel = v[i];
				f2A[streamIndex[offst+4]] = f2A[offst+2] - 6.0*ws*rA[nList[offst+4]]*evel;
				f2B[streamIndex[offst+4]] = f2B[offst+2] - 6.0*ws*rB[nList[offst+4]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+2] - 6.0*ws*rA[nList[offst+4]]*evel + 
					    2.0*f2B[offst+2] - 6.0*ws*rB[nList[offst+4]]*evel;
				meF2Sx = 0.0;
				meF2Sy = meF2S;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);
				// zero populations inside particle	
				f2A[offst+2] = 0.0;
				f2B[offst+2] = 0.0;
			}
			
			// dir 3 bounce-back to nabor 1 as dir 1:
			if (s[nList[offst+1]] == 0) {
				// bounce-back
				float evel = -u[i];
				f2A[streamIndex[offst+1]] = f2A[offst+3] - 6.0*ws*rA[nList[offst+1]]*evel;
				f2B[streamIndex[offst+1]] = f2B[offst+3] - 6.0*ws*rB[nList[offst+1]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+3] - 6.0*ws*rA[nList[offst+1]]*evel + 
					    2.0*f2B[offst+3] - 6.0*ws*rB[nList[offst+1]]*evel;
				meF2Sx = -meF2S;
				meF2Sy = 0.0;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);	
				// zero populations inside particle
				f2A[offst+3] = 0.0;
				f2B[offst+3] = 0.0;
			}
			
			// dir 4 bounce-back to nabor 2 as dir 2:
			if (s[nList[offst+2]] == 0) {
				// bounce-back
				float evel = -v[i];
				f2A[streamIndex[offst+2]] = f2A[offst+4] - 6.0*ws*rA[nList[offst+2]]*evel;
				f2B[streamIndex[offst+2]] = f2B[offst+4] - 6.0*ws*rB[nList[offst+2]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+4] - 6.0*ws*rA[nList[offst+2]]*evel + 
					    2.0*f2B[offst+4] - 6.0*ws*rB[nList[offst+2]]*evel;
				meF2Sx = 0.0;  
				meF2Sy = -meF2S;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);	
				// zero populations inside particle
				f2A[offst+4] = 0.0;
				f2B[offst+4] = 0.0;
			}
			
			// dir 5 bounce-back to nabor 7 as dir 7:
			if (s[nList[offst+7]] == 0) {
				// bounce-back
				float evel = u[i] + v[i];
				f2A[streamIndex[offst+7]] = f2A[offst+5] - 6.0*wd*rA[nList[offst+7]]*evel;
				f2B[streamIndex[offst+7]] = f2B[offst+5] - 6.0*wd*rB[nList[offst+7]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+5] - 6.0*wd*rA[nList[offst+7]]*evel + 
					    2.0*f2B[offst+5] - 6.0*wd*rB[nList[offst+7]]*evel;  
				meF2Sx = meF2S;  
				meF2Sy = meF2S;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);	
				// zero populations inside particle
				f2A[offst+5] = 0.0;
				f2B[offst+5] = 0.0;
			}
			
			// dir 6 bounce-back to nabor 8 as dir 8:
			if (s[nList[offst+8]] == 0) {
				// bounce-back
				float evel = -u[i] + v[i];
				f2A[streamIndex[offst+8]] = f2A[offst+6] - 6.0*wd*rA[nList[offst+8]]*evel;
				f2B[streamIndex[offst+8]] = f2B[offst+6] - 6.0*wd*rB[nList[offst+8]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+6] - 6.0*wd*rA[nList[offst+8]]*evel + 
					    2.0*f2B[offst+6] - 6.0*wd*rB[nList[offst+8]]*evel;  
				meF2Sx = -meF2S;  
				meF2Sy = meF2S;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);	
				// zero populations inside particle
				f2A[offst+6] = 0.0;
				f2B[offst+6] = 0.0;
			}
			
			// dir 7 bounce-back to nabor 5 as dir 5:
			if (s[nList[offst+5]] == 0) {
				// bounce-back
				float evel = -u[i] - v[i];
				f2A[streamIndex[offst+5]] = f2A[offst+7] - 6.0*wd*rA[nList[offst+5]]*evel;
				f2B[streamIndex[offst+5]] = f2B[offst+7] - 6.0*wd*rB[nList[offst+5]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+7] - 6.0*wd*rA[nList[offst+5]]*evel + 
					    2.0*f2B[offst+7] - 6.0*wd*rB[nList[offst+5]]*evel;  
				meF2Sx = -meF2S;  
				meF2Sy = -meF2S;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);
				// zero populations inside particle	
				f2A[offst+7] = 0.0;
				f2B[offst+7] = 0.0;
			}
			
			// dir 8 bounce-back to nabor 6 as dir 6:
			if (s[nList[offst+6]] == 0) {
				// bounce-back
				float evel = u[i] - v[i];
				f2A[streamIndex[offst+6]] = f2A[offst+8] - 6.0*wd*rA[nList[offst+6]]*evel;
				f2B[streamIndex[offst+6]] = f2B[offst+8] - 6.0*wd*rB[nList[offst+6]]*evel;
				// momentum exchange to particle
				meF2S = 2.0*f2A[offst+8] - 6.0*wd*rA[nList[offst+6]]*evel + 
					    2.0*f2B[offst+8] - 6.0*wd*rB[nList[offst+6]]*evel;  
				meF2Sx = meF2S;  
				meF2Sy = -meF2S;
				atomicAdd(&pt[pID].f.x, meF2Sx);
				atomicAdd(&pt[pID].f.y, meF2Sy);
				// zero populations inside particle	
				f2A[offst+8] = 0.0;
				f2B[offst+8] = 0.0;
			}
			
		}	
	}		
}



