#include "hip/hip_runtime.h"

# include "kernels_mcmp_SC_bb_D2Q9.cuh"
# include "../mcmp_SC/mcmp_pseudopotential.cuh"
# include <stdio.h>



// --------------------------------------------------------
// D2Q9 initialize kernel: 
// --------------------------------------------------------

__global__ void mcmp_initial_equilibrium_bb_D2Q9(float* fA,
                                                 float* fB,
										         float* rA,
											     float* rB,
										         float* u,
										         float* v,
										         int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	// initialize populations to equilibrium values:
	if (i < nVoxels) {	
		
		int offst = 9*i;
		const float w0 = 4.0/9.0;
		const float ws = 1.0/9.0;
		const float wd = 1.0/36.0;
		const float omusq = 1.0 - 1.5*(u[i]*u[i] + v[i]*v[i]);
		
		// dir 0
		float feq = w0*omusq;
		fA[offst+0] = feq*rA[i];
		fB[offst+0] = feq*rB[i];
		
		// dir 1
		float evel = u[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+1] = feq*rA[i];
		fB[offst+1] = feq*rB[i];
		
		// dir 2
		evel = v[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+2] = feq*rA[i];
		fB[offst+2] = feq*rB[i];
		
		// dir 3
		evel = -u[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+3] = feq*rA[i];
		fB[offst+3] = feq*rB[i];
		
		// dir 4
		evel = -v[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+4] = feq*rA[i];
		fB[offst+4] = feq*rB[i];
		
		// dir 5
		evel = u[i] + v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+5] = feq*rA[i];
		fB[offst+5] = feq*rB[i];
		
		// dir 6
		evel = -u[i] + v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+6] = feq*rA[i];
		fB[offst+6] = feq*rB[i];
		
		// dir 7
		evel = -u[i] - v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+7] = feq*rA[i];
		fB[offst+7] = feq*rB[i];
		
		// dir 8
		evel = u[i] - v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+8] = feq*rA[i];
		fB[offst+8] = feq*rB[i];
				
	}
}



// --------------------------------------------------------
// Map particles to grid by updating rS[] and pID[] arrays:
// --------------------------------------------------------

__global__ void mcmp_initial_particles_on_lattice_D2Q9(float* prx,
                                                       float* pry,
					  								   float* prad,
                                                       int* x,
										               int* y,
													   int* s,									   
										               int* pIDgrid,										   
										               int nVoxels,
										               int nParts)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
		
		// --------------------------------------------------	
		// default values:
		// --------------------------------------------------
				
		s[i] = 0;
		pIDgrid[i] = -1;
		
		// --------------------------------------------------	
		// loop over particles:
		// --------------------------------------------------
		
		for (int j=0; j<nParts; j++) {
			
			// ---------------------------	
			// distance to particle c.o.m:
			// ---------------------------
			
			float dx = float(x[i]) - prx[j];
			float dy = float(y[i]) - pry[j];
			float rr = sqrt(dx*dx + dy*dy);
			
			// ---------------------------	
			// assign values:
			// ---------------------------
			
			if (rr <= prad[j]) {
				s[i] = 1;
				pIDgrid[i] = j;	
			}		
		}							
	}
}



// --------------------------------------------------------
// D2Q9 kernel to update the particle fields on the lattice: 
// --------------------------------------------------------

__global__ void mcmp_update_particles_on_lattice_D2Q9(float* fA,
                                                      float* fB,
										              float* rA,
											          float* rB,
										              float* u,
										              float* v,
													  float* prx,
													  float* pry,
													  float* pvx,
													  float* pvy,
													  float* prad,
													  int* x,
													  int* y,
													  int* s,
													  int* pIDgrid,
													  int* nList,													  
										              int nVoxels,
													  int nParts)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {	
		
		int offst = 9*i;
		
		// --------------------------------------------	
		// get the current state of voxel:
		// --------------------------------------------
		
		int s0 = s[i];
		
		// --------------------------------------------	
		// get the new state of voxel by seeing if any
		// particles overlay it:
		// --------------------------------------------
				
		int s1 = 0;
		int pID = 0;
		float partvx = 0.0;
		float partvy = 0.0;
		for (int p=0; p<nParts; p++) {
			float dx = float(x[i]) - prx[p];
			float dy = float(y[i]) - pry[p];
			float rp = sqrt(dx*dx + dy*dy);
			if (rp <= prad[p]) {
				s1 = 1;
				pID = p;
				partvx = pvx[p];
				partvy = pvy[p];
			}		
		}
		s[i] = s1;
		
		// --------------------------------------------	
		// decide course of action:
		// --------------------------------------------
		
		// fluid site STAYS fluid site
		if (s0 == 0 && s1 == 0) {
			pIDgrid[i] = -1;  // this is redundant, but that's OK
		}
		
		// particle site STAYS particle site
		else if (s0 == 1 && s1 == 1) {			
			u[i] = partvx;
			v[i] = partvy;
			pIDgrid[i] = pID;
			// zero all the populations			
			for (int n=0; n<9; n++) {
				fA[offst+n] = 0.0;
				fB[offst+n] = 0.0;
			}
		}
		
		// fluid site becomes particle site (COVERING)
		else if (s0 == 0 && s1 == 1) {						
			// update velocity to particle's velocity
			u[i] = partvx;
			v[i] = partvy;
			pIDgrid[i] = pID;
			// zero all the populations			
			for (int n=0; n<9; n++) {
				fA[offst+n] = 0.0;
				fB[offst+n] = 0.0;
			}
		}
		
		// particle site becomes fluid site (UNCOVERING)
		else if (s0 == 1 && s1 == 0) {				
			// assign voxel velocity with particle velocity
			u[i] = pvx[pIDgrid[i]]; 
			v[i] = pvy[pIDgrid[i]];
			pIDgrid[i] = -1;
			// get average density of surrounding fluid sites:
			int num_fluid_nabors = 0;
			float aver_rA_nabors = 0.0;
			float aver_rB_nabors = 0.0;			
			for (int n=1; n<9; n++) {  // do not include self, n=0
				int nID = nList[offst+n];
				if (s[nID] == 0) {
					num_fluid_nabors++;
					aver_rA_nabors += rA[nID];
					aver_rB_nabors += rB[nID];
				}
			}			
			if (num_fluid_nabors > 0) {
				rA[i] = aver_rA_nabors/num_fluid_nabors;
				rB[i] = aver_rB_nabors/num_fluid_nabors;
			}
			// set populations to the equilibrium for the given
			// velocity and density:
			const float w0 = 4.0/9.0;
			const float ws = 1.0/9.0;
			const float wd = 1.0/36.0;
			const float omusq = 1.0 - 1.5*(u[i]*u[i] + v[i]*v[i]);		
			// dir 0
			float feq = w0*omusq;
			fA[offst+0] = feq*rA[i];
			fB[offst+0] = feq*rB[i];		
			// dir 1
			float evel = u[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+1] = feq*rA[i];
			fB[offst+1] = feq*rB[i];		
			// dir 2
			evel = v[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+2] = feq*rA[i];
			fB[offst+2] = feq*rB[i];		
			// dir 3
			evel = -u[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+3] = feq*rA[i];
			fB[offst+3] = feq*rB[i];		
			// dir 4
			evel = -v[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+4] = feq*rA[i];
			fB[offst+4] = feq*rB[i];		
			// dir 5
			evel = u[i] + v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+5] = feq*rA[i];
			fB[offst+5] = feq*rB[i];		
			// dir 6
			evel = -u[i] + v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+6] = feq*rA[i];
			fB[offst+6] = feq*rB[i];		
			// dir 7
			evel = -u[i] - v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+7] = feq*rA[i];
			fB[offst+7] = feq*rB[i];		
			// dir 8
			evel = u[i] - v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			fA[offst+8] = feq*rA[i];
			fB[offst+8] = feq*rB[i];
		}				
	}
}



// --------------------------------------------------------
// D2Q9 compute velocity (barycentric) for the system: 
// --------------------------------------------------------

__global__ void mcmp_compute_velocity_bb_D2Q9(float* fA,
                                              float* fB,
										      float* rA,
										      float* rB,
										      float* FxA,
										      float* FxB,
										      float* FyA,
										      float* FyB,
										      float* u,
										      float* v,
											  int* s,
										      int nVoxels) 
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		
		// --------------------------------------------------	
		// if this is a fluid site:
		// --------------------------------------------------
		
		if (s[i] == 0) {
			int offst = i*9;			
			float uA = fA[offst+1] + fA[offst+5] + fA[offst+8] - (fA[offst+3] + fA[offst+6] + fA[offst+7]) + 0.5*FxA[i];
			float uB = fB[offst+1] + fB[offst+5] + fB[offst+8] - (fB[offst+3] + fB[offst+6] + fB[offst+7]) + 0.5*FxB[i];
			float vA = fA[offst+2] + fA[offst+5] + fA[offst+6] - (fA[offst+4] + fA[offst+7] + fA[offst+8]) + 0.5*FyA[i];
			float vB = fB[offst+2] + fB[offst+5] + fB[offst+6] - (fB[offst+4] + fB[offst+7] + fB[offst+8]) + 0.5*FyB[i];
			float rTotal = rA[i] + rB[i];
			u[i] = (uA + uB)/rTotal;
			v[i] = (vA + vB)/rTotal;	
		}
		
		// --------------------------------------------------	
		// if this is a solid site:
		// --------------------------------------------------
		
		else if (s[i] == 1) {
			//u[i] = 0.0;  // later, fill in with particle velocity
			//v[i] = 0.0;  
		}
					
	}
}



// --------------------------------------------------------
// D2Q9 compute density for each component: 
// --------------------------------------------------------

__global__ void mcmp_compute_density_bb_D2Q9(float* fA,
                                        	 float* fB,
										     float* rA,
										     float* rB,
										     int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		int offst = i*9;			
		rA[i] = fA[offst] + fA[offst+1] + fA[offst+2] + fA[offst+3] + fA[offst+4] + fA[offst+5] + fA[offst+6] +
		        fA[offst+7] + fA[offst+8];
		rB[i] = fB[offst] + fB[offst+1] + fB[offst+2] + fB[offst+3] + fB[offst+4] + fB[offst+5] + fB[offst+6] +
		        fB[offst+7] + fB[offst+8];
	}
}



// --------------------------------------------------------
// D2Q9 compute Shan-Chen forces for the components
// using pseudo-potential, psi = rho_0(1-exp(-rho/rho_o))
// --------------------------------------------------------

__global__ void mcmp_compute_SC_forces_bb_D2Q9(float* rA,
										       float* rB,
										       float* FxA,
										       float* FxB,
										       float* FyA,
										       float* FyB,
											   int* s,
											   int* nList,
											   float gAB,
											   float gAS,
											   float gBS,
										       int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		
		// --------------------------------------------	
		// if this is a fluid site:
		// --------------------------------------------
		
		if (s[i] == 0) {
			
			int offst = i*9;
			
			float r0A = psi(rA[i]);	 			
			float r1A = psi(rA[nList[offst+1]]);
			float r2A = psi(rA[nList[offst+2]]);
			float r3A = psi(rA[nList[offst+3]]);
			float r4A = psi(rA[nList[offst+4]]);
			float r5A = psi(rA[nList[offst+5]]);
			float r6A = psi(rA[nList[offst+6]]);
			float r7A = psi(rA[nList[offst+7]]);
			float r8A = psi(rA[nList[offst+8]]);
		
			float r0B = psi(rB[i]);		
			float r1B = psi(rB[nList[offst+1]]);
			float r2B = psi(rB[nList[offst+2]]);
			float r3B = psi(rB[nList[offst+3]]);
			float r4B = psi(rB[nList[offst+4]]);
			float r5B = psi(rB[nList[offst+5]]);
			float r6B = psi(rB[nList[offst+6]]);
			float r7B = psi(rB[nList[offst+7]]);
			float r8B = psi(rB[nList[offst+8]]);
			
			float s1 = float(s[nList[offst+1]]);
			float s2 = float(s[nList[offst+2]]);
			float s3 = float(s[nList[offst+3]]);
			float s4 = float(s[nList[offst+4]]);
			float s5 = float(s[nList[offst+5]]);
			float s6 = float(s[nList[offst+6]]);
			float s7 = float(s[nList[offst+7]]);
			float s8 = float(s[nList[offst+8]]);
		
			float ws = 1.0/9.0;
			float wd = 1.0/36.0;		
			float sumNbrRhoAx = ws*r1A + wd*r5A + wd*r8A - (ws*r3A + wd*r6A + wd*r7A);
			float sumNbrRhoAy = ws*r2A + wd*r5A + wd*r6A - (ws*r4A + wd*r7A + wd*r8A);
			float sumNbrRhoBx = ws*r1B + wd*r5B + wd*r8B - (ws*r3B + wd*r6B + wd*r7B);
			float sumNbrRhoBy = ws*r2B + wd*r5B + wd*r6B - (ws*r4B + wd*r7B + wd*r8B);
			float sumNbrSx = ws*s1 + wd*s5 + wd*s8 - (ws*s3 + wd*s6 + wd*s7);
			float sumNbrSy = ws*s2 + wd*s5 + wd*s6 - (ws*s4 + wd*s7 + wd*s8);
			
			FxA[i] = -r0A*(gAB*sumNbrRhoBx + gAS*sumNbrSx);
			FxB[i] = -r0B*(gAB*sumNbrRhoAx + gBS*sumNbrSx);
			FyA[i] = -r0A*(gAB*sumNbrRhoBy + gAS*sumNbrSy);
			FyB[i] = -r0B*(gAB*sumNbrRhoAy + gBS*sumNbrSy);
						
		}		
	}
}



// --------------------------------------------------------
// D2Q9 update kernel:
// --------------------------------------------------------

__global__ void mcmp_collide_stream_bb_D2Q9(float* f1A,
                                         	float* f1B,
										 	float* f2A,
										 	float* f2B,
										 	float* rA,
										 	float* rB,
										 	float* u,
										 	float* v,
										 	float* FxA,
										 	float* FxB,
										 	float* FyA,
										 	float* FyB,
											int* s,
										 	int* streamIndex,
										 	float nu,
										 	int nVoxels)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
		
		// --------------------------------------------------	
		// Only do collide & stream if "i" is a fluid node:
		// --------------------------------------------------
		
		if (s[i] == 0) {
			
			// --------------------------------------------------	
			// FORCING - this step includes the Guo forcing
			//           scheme applied to the Shan-Chen
			//           MCMP model according to Kruger et al.
			// --------------------------------------------------
		
			float w0 = 4.0/9.0;
			float ws = 1.0/9.0;
			float wd = 1.0/36.0;
		
			float evel = 0.0;       // e dot velocity
			float emiu = 0.0-u[i];  // e minus u
			float emiv = 0.0-v[i];  // e minus v
			float frc0A = w0*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv) );
			float frc0B = w0*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv) );
		
			evel = u[i];
			emiu = 1.0-u[i];
			emiv = 0.0-v[i];
			float frc1A = ws*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv) );
			float frc1B = ws*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv) );
		
			evel = v[i]; 
			emiu = 0.0-u[i];
			emiv = 1.0-v[i];
			float frc2A = ws*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv + 9.0*evel) );
			float frc2B = ws*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv + 9.0*evel) );
		
			evel = -u[i];
			emiu = -1.0-u[i];
			emiv =  0.0-v[i];
			float frc3A = ws*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv) );
			float frc3B = ws*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv) );
		
			evel = -v[i];
			emiu =  0.0-u[i];
			emiv = -1.0-v[i];
			float frc4A = ws*( FxA[i]*(3.0*emiu) + FyA[i]*(3.0*emiv - 9.0*evel) );
			float frc4B = ws*( FxB[i]*(3.0*emiu) + FyB[i]*(3.0*emiv - 9.0*evel) );
		
			evel = u[i] + v[i];
			emiu = 1.0-u[i];
			emiv = 1.0-v[i];
			float frc5A = wd*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv + 9.0*evel) );
			float frc5B = wd*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv + 9.0*evel) );
		
			evel = -u[i] + v[i];
			emiu = -1.0-u[i];
			emiv =  1.0-v[i];
			float frc6A = wd*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv + 9.0*evel) );
			float frc6B = wd*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv + 9.0*evel) );
				
			evel = -u[i] - v[i];
			emiu = -1.0-u[i];
			emiv = -1.0-v[i];
			float frc7A = wd*( FxA[i]*(3.0*emiu - 9.0*evel) + FyA[i]*(3.0*emiv - 9.0*evel) );
			float frc7B = wd*( FxB[i]*(3.0*emiu - 9.0*evel) + FyB[i]*(3.0*emiv - 9.0*evel) );
		
			evel = u[i] - v[i];
			emiu =  1.0-u[i];
			emiv = -1.0-v[i];
			float frc8A = wd*( FxA[i]*(3.0*emiu + 9.0*evel) + FyA[i]*(3.0*emiv - 9.0*evel) );
			float frc8B = wd*( FxB[i]*(3.0*emiu + 9.0*evel) + FyB[i]*(3.0*emiv - 9.0*evel) );
				
			// --------------------------------------------------	
			// COLLISION & STREAMING - standard BGK operator with
			//                         a PUSH propagator.
			// --------------------------------------------------
		
			int offst = 9*i;
			const float omega = 2.0/(6.0*nu + 1.0);   // 1/tau
			const float omomega = 1.0 - omega;        // 1 - 1/tau
			const float omomega2 = 1.0 - 0.5*omega;   // 1 - 1/(2tau)
			const float omusq = 1.0 - 1.5*(u[i]*u[i] + v[i]*v[i]);
		
			// dir 0
			float feq = w0*omusq;
			f2A[streamIndex[offst+0]] = omomega*f1A[offst+0] + omega*feq*rA[i] + omomega2*frc0A;
			f2B[streamIndex[offst+0]] = omomega*f1B[offst+0] + omega*feq*rB[i] + omomega2*frc0B;
		
			// dir 1
			evel = u[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+1]] = omomega*f1A[offst+1] + omega*feq*rA[i] + omomega2*frc1A;
			f2B[streamIndex[offst+1]] = omomega*f1B[offst+1] + omega*feq*rB[i] + omomega2*frc1B;
		
			// dir 2
			evel = v[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+2]] = omomega*f1A[offst+2] + omega*feq*rA[i] + omomega2*frc2A;
			f2B[streamIndex[offst+2]] = omomega*f1B[offst+2] + omega*feq*rB[i] + omomega2*frc2B;
		
			// dir 3
			evel = -u[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+3]] = omomega*f1A[offst+3] + omega*feq*rA[i] + omomega2*frc3A;
			f2B[streamIndex[offst+3]] = omomega*f1B[offst+3] + omega*feq*rB[i] + omomega2*frc3B;
		
			// dir 4
			evel = -v[i];
			feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+4]] = omomega*f1A[offst+4] + omega*feq*rA[i] + omomega2*frc4A;
			f2B[streamIndex[offst+4]] = omomega*f1B[offst+4] + omega*feq*rB[i] + omomega2*frc4B;
		
			// dir 5
			evel = u[i] + v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+5]] = omomega*f1A[offst+5] + omega*feq*rA[i] + omomega2*frc5A;
			f2B[streamIndex[offst+5]] = omomega*f1B[offst+5] + omega*feq*rB[i] + omomega2*frc5B;
		
			// dir 6
			evel = -u[i] + v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+6]] = omomega*f1A[offst+6] + omega*feq*rA[i] + omomega2*frc6A;
			f2B[streamIndex[offst+6]] = omomega*f1B[offst+6] + omega*feq*rB[i] + omomega2*frc6B;
		
			// dir 7
			evel = -u[i] - v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+7]] = omomega*f1A[offst+7] + omega*feq*rA[i] + omomega2*frc7A;
			f2B[streamIndex[offst+7]] = omomega*f1B[offst+7] + omega*feq*rB[i] + omomega2*frc7B;
		
			// dir 8
			evel = u[i] - v[i];
			feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
			f2A[streamIndex[offst+8]] = omomega*f1A[offst+8] + omega*feq*rA[i] + omomega2*frc8A;
			f2B[streamIndex[offst+8]] = omomega*f1B[offst+8] + omega*feq*rB[i] + omomega2*frc8B;
			
		}					
	}
}



// --------------------------------------------------------
// D2Q9 implement bounce-back conditions:
// --------------------------------------------------------

__global__ void mcmp_bounce_back_D2Q9(float* f2A, 
									  float* f2B,
									  int* s,
									  int* nList,									  
									  int* streamIndex,
									  int nVoxels)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
								
		// --------------------------------------------------	
		// If the current voxel is solid, then bounce-back
		// the populations just received via streaming 
		// back to the neighboring voxel:
		// --------------------------------------------------
		
		if (s[i] == 1) {
			
			int offst = 9*i;
						
			// dir 1 bounce-back to nabor 3 as dir 3:
			if (s[nList[offst+3]] == 0) {
				f2A[streamIndex[offst+3]] = f2A[offst+1];
				f2B[streamIndex[offst+3]] = f2B[offst+1];
				f2A[offst+1] = 0.0;
				f2B[offst+1] = 0.0;
			}
			
			// dir 2 bounce-back to nabor 4 as dir 4:
			if (s[nList[offst+4]] == 0) {
				f2A[streamIndex[offst+4]] = f2A[offst+2];
				f2B[streamIndex[offst+4]] = f2B[offst+2];
				f2A[offst+2] = 0.0;
				f2B[offst+2] = 0.0;
			}
			
			// dir 3 bounce-back to nabor 1 as dir 1:
			if (s[nList[offst+1]] == 0) {
				f2A[streamIndex[offst+1]] = f2A[offst+3];
				f2B[streamIndex[offst+1]] = f2B[offst+3];
				f2A[offst+3] = 0.0;
				f2B[offst+3] = 0.0;
			}
			
			// dir 4 bounce-back to nabor 2 as dir 2:
			if (s[nList[offst+2]] == 0) {
				f2A[streamIndex[offst+2]] = f2A[offst+4];
				f2B[streamIndex[offst+2]] = f2B[offst+4];
				f2A[offst+4] = 0.0;
				f2B[offst+4] = 0.0;
			}
			
			// dir 5 bounce-back to nabor 7 as dir 7:
			if (s[nList[offst+7]] == 0) {
				f2A[streamIndex[offst+7]] = f2A[offst+5];
				f2B[streamIndex[offst+7]] = f2B[offst+5];
				f2A[offst+5] = 0.0;
				f2B[offst+5] = 0.0;
			}
			
			// dir 6 bounce-back to nabor 8 as dir 8:
			if (s[nList[offst+8]] == 0) {
				f2A[streamIndex[offst+8]] = f2A[offst+6];
				f2B[streamIndex[offst+8]] = f2B[offst+6];
				f2A[offst+6] = 0.0;
				f2B[offst+6] = 0.0;
			}
			
			// dir 7 bounce-back to nabor 5 as dir 5:
			if (s[nList[offst+5]] == 0) {
				f2A[streamIndex[offst+5]] = f2A[offst+7];
				f2B[streamIndex[offst+5]] = f2B[offst+7];
				f2A[offst+7] = 0.0;
				f2B[offst+7] = 0.0;
			}
			
			// dir 8 bounce-back to nabor 6 as dir 6:
			if (s[nList[offst+6]] == 0) {
				f2A[streamIndex[offst+6]] = f2A[offst+8];
				f2B[streamIndex[offst+6]] = f2B[offst+8];
				f2A[offst+8] = 0.0;
				f2B[offst+8] = 0.0;
			}
			
		}	
	}		
}



// --------------------------------------------------------
// D2Q9 implement bounce-back conditions for moving
// solids:
// --------------------------------------------------------

__global__ void mcmp_bounce_back_moving_D2Q9(float* f2A, 
									         float* f2B,
											 float* rA,
											 float* rB,
											 float* u,
											 float* v,
									         int* s,
									         int* nList,									  
									         int* streamIndex,
									         int nVoxels)
{

	// define voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
								
		// --------------------------------------------------	
		// If the current voxel is solid, then bounce-back
		// the populations just received via streaming 
		// back to the neighboring voxel:
		// --------------------------------------------------
		
		if (s[i] == 1) {
			
			int offst = 9*i;
			const float ws = 1.0/9.0;
			const float wd = 1.0/36.0;
						
			// dir 1 bounce-back to nabor 3 as dir 3:
			if (s[nList[offst+3]] == 0) {
				float evel = u[i];
				f2A[streamIndex[offst+3]] = f2A[offst+1] - 6.0*ws*rA[nList[offst+3]]*evel;
				f2B[streamIndex[offst+3]] = f2B[offst+1] - 6.0*ws*rB[nList[offst+3]]*evel;
				f2A[offst+1] = 0.0;
				f2B[offst+1] = 0.0;
			}
			
			// dir 2 bounce-back to nabor 4 as dir 4:
			if (s[nList[offst+4]] == 0) {
				float evel = v[i];
				f2A[streamIndex[offst+4]] = f2A[offst+2] - 6.0*ws*rA[nList[offst+4]]*evel;
				f2B[streamIndex[offst+4]] = f2B[offst+2] - 6.0*ws*rB[nList[offst+4]]*evel;
				f2A[offst+2] = 0.0;
				f2B[offst+2] = 0.0;
			}
			
			// dir 3 bounce-back to nabor 1 as dir 1:
			if (s[nList[offst+1]] == 0) {
				float evel = -u[i];
				f2A[streamIndex[offst+1]] = f2A[offst+3] - 6.0*ws*rA[nList[offst+1]]*evel;
				f2B[streamIndex[offst+1]] = f2B[offst+3] - 6.0*ws*rB[nList[offst+1]]*evel;
				f2A[offst+3] = 0.0;
				f2B[offst+3] = 0.0;
			}
			
			// dir 4 bounce-back to nabor 2 as dir 2:
			if (s[nList[offst+2]] == 0) {
				float evel = -v[i];
				f2A[streamIndex[offst+2]] = f2A[offst+4] - 6.0*ws*rA[nList[offst+2]]*evel;
				f2B[streamIndex[offst+2]] = f2B[offst+4] - 6.0*ws*rB[nList[offst+2]]*evel;
				f2A[offst+4] = 0.0;
				f2B[offst+4] = 0.0;
			}
			
			// dir 5 bounce-back to nabor 7 as dir 7:
			if (s[nList[offst+7]] == 0) {
				float evel = u[i] + v[i];
				f2A[streamIndex[offst+7]] = f2A[offst+5] - 6.0*wd*rA[nList[offst+7]]*evel;
				f2B[streamIndex[offst+7]] = f2B[offst+5] - 6.0*wd*rB[nList[offst+7]]*evel;
				f2A[offst+5] = 0.0;
				f2B[offst+5] = 0.0;
			}
			
			// dir 6 bounce-back to nabor 8 as dir 8:
			if (s[nList[offst+8]] == 0) {
				float evel = -u[i] + v[i];
				f2A[streamIndex[offst+8]] = f2A[offst+6] - 6.0*wd*rA[nList[offst+8]]*evel;
				f2B[streamIndex[offst+8]] = f2B[offst+6] - 6.0*wd*rB[nList[offst+8]]*evel;
				f2A[offst+6] = 0.0;
				f2B[offst+6] = 0.0;
			}
			
			// dir 7 bounce-back to nabor 5 as dir 5:
			if (s[nList[offst+5]] == 0) {
				float evel = -u[i] - v[i];
				f2A[streamIndex[offst+5]] = f2A[offst+7] - 6.0*wd*rA[nList[offst+5]]*evel;
				f2B[streamIndex[offst+5]] = f2B[offst+7] - 6.0*wd*rB[nList[offst+5]]*evel;
				f2A[offst+7] = 0.0;
				f2B[offst+7] = 0.0;
			}
			
			// dir 8 bounce-back to nabor 6 as dir 6:
			if (s[nList[offst+6]] == 0) {
				float evel = u[i] - v[i];
				f2A[streamIndex[offst+6]] = f2A[offst+8] - 6.0*wd*rA[nList[offst+6]]*evel;
				f2B[streamIndex[offst+6]] = f2B[offst+8] - 6.0*wd*rB[nList[offst+6]]*evel;
				f2A[offst+8] = 0.0;
				f2B[offst+8] = 0.0;
			}
			
		}	
	}		
}



