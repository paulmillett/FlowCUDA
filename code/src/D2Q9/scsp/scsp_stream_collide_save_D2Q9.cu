#include "hip/hip_runtime.h"

# include "scsp_stream_collide_save_D2Q9.cuh"
# include "../iolets/zou_he_BC_D2Q9.cuh"
# include <stdio.h>



// --------------------------------------------------------
// D2Q9 update kernel.
// This algorithm is based on the optimized "stream-collide-
// save" algorithm recommended by T. Kruger in the 
// textbook: "The Lattice Boltzmann Method: Principles
// and Practice".
// --------------------------------------------------------

__global__ 
void scsp_stream_collide_save_D2Q9(
	float* f1,
    float* f2,
	float* r,
	float* u,
	float* v,
	int* streamIndex,
	int* voxelType,
	iolet2D* iolets,
	float nu,
	int nVoxels,
	bool save)
{

	// -----------------------------------------------
	// define voxel:
	// -----------------------------------------------
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
		
		// --------------------------------------------------		
		// voxel-specific parameters:
		// --------------------------------------------------
		
		int vtype = voxelType[i];
		int offst = 9*i;	
		float ft[9];
		
		// --------------------------------------------------		
		// STREAMING - load populations from adjacent voxels,
		//             note	that streamIndex[] accounts for
		//             halfway bounceback conditions.
		// --------------------------------------------------
		
		ft[0] = f1[streamIndex[offst+0]];                   
		ft[1] = f1[streamIndex[offst+1]]; 
		ft[2] = f1[streamIndex[offst+2]];  
		ft[3] = f1[streamIndex[offst+3]];  
		ft[4] = f1[streamIndex[offst+4]];  
		ft[5] = f1[streamIndex[offst+5]]; 
		ft[6] = f1[streamIndex[offst+6]];  
		ft[7] = f1[streamIndex[offst+7]];  
		ft[8] = f1[streamIndex[offst+8]]; 		
				
		// --------------------------------------------------
		// COLLISION - this step is done only for fluid voxels;
		//             other treatments are performed for inlet/
		//             outlet voxels.  
		// --------------------------------------------------
		
		float rho = ft[0] + ft[1] + ft[2] + ft[3] + ft[4] + ft[5] + ft[6] + ft[7] + ft[8];
		float rhoinv = 1.0/rho;
		float ux = rhoinv*(ft[1] + ft[5] + ft[8] - (ft[3] + ft[6] + ft[7]));
		float vy = rhoinv*(ft[2] + ft[5] + ft[6] - (ft[4] + ft[7] + ft[8]));
		
		// Fluid voxel...
		if (vtype == 0) {
			
			// useful constants:
			const float tauinv = 2.0/(6.0*nu + 1.0);   // 1/tau
			const float omtauinv = 1.0 - tauinv;       // 1 - 1/tau
			const float tw0r = (4.0/9.0)*rho*tauinv;   // w[0]*rho/tau
			const float twsr = (1.0/9.0)*rho*tauinv;   // w[1-4]*rho/tau
			const float twdr = (1.0/36.0)*rho*tauinv;  // w[5-8]*rho/tau
			const float omusq = 1.0 - 1.5*(ux*ux + vy*vy);
			const float tux = 3.0*ux;
			const float tvy = 3.0*vy;	
			
			// collision calculations:	
			f2[offst+0] = omtauinv*ft[0] + tw0r*(omusq);		                      // dir-0
			float cidot3u = tux;
			f2[offst+1] = omtauinv*ft[1] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-1
			cidot3u = tvy;
			f2[offst+2] = omtauinv*ft[2] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-2
			cidot3u = -tux;
			f2[offst+3] = omtauinv*ft[3] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-3
			cidot3u = -tvy;
			f2[offst+4] = omtauinv*ft[4] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-4
			cidot3u = tux+tvy;
			f2[offst+5] = omtauinv*ft[5] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-5
			cidot3u = tvy-tux;
			f2[offst+6] = omtauinv*ft[6] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-6
			cidot3u = -(tux+tvy);
			f2[offst+7] = omtauinv*ft[7] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-7
			cidot3u = tux-tvy;
			f2[offst+8] = omtauinv*ft[8] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-8			
		
		}
		
		// Boundary Conditions:
		else if (vtype > 0) {
			
			// decide the type of iolet:
			const int ioi = vtype - 1;  // iolet index
			const int iotype = iolets[ioi].type;
			
			// Zou-He velocity boundary (East)...
			if (iotype == 1) {
				ux = iolets[ioi].uBC;
				vy = iolets[ioi].vBC;
				rho = (ft[0]+ft[2]+ft[4] + 2.0*(ft[1]+ft[5]+ft[8])) / (1.0 + ux);
				zou_he_velo_east_D2Q9(offst,f2,ft,ux,vy,rho);
			}
			
			// Zou-He velocity boundary (West)...
			else if (iotype == 2) {
				ux = iolets[ioi].uBC;
				vy = iolets[ioi].vBC;
				rho = (ft[0]+ft[2]+ft[4] + 2.0*(ft[3]+ft[7]+ft[6])) / (1.0 - ux);
				zou_he_velo_west_D2Q9(offst,f2,ft,ux,vy,rho);
			}
			
			// Zou-He velocity boundary (North)...
			else if (iotype == 3) {
				ux = iolets[ioi].uBC;
				vy = iolets[ioi].vBC;
				rho = (ft[0]+ft[1]+ft[3] + 2.0*(ft[2]+ft[5]+ft[6])) / (1.0 + vy);
				zou_he_velo_north_D2Q9(offst,f2,ft,ux,vy,rho);
			}	
			
			// Zou-He velocity boundary (South)...
			else if (iotype == 4) {
				ux = iolets[ioi].uBC;
				vy = iolets[ioi].vBC;
				rho = (ft[0]+ft[1]+ft[3] + 2.0*(ft[4]+ft[7]+ft[8])) / (1.0 - vy);
				zou_he_velo_south_D2Q9(offst,f2,ft,ux,vy,rho);
			}	
									
			// Zou-He pressure boundary (East)...
			else if (iotype == 11) {
				vy = iolets[ioi].vBC;
				rho = iolets[ioi].rBC;
				ux = (ft[0]+ft[2]+ft[4] + 2.0*(ft[1]+ft[5]+ft[8]))/rho - 1.0;				
				zou_he_pres_east_D2Q9(offst,f2,ft,ux,vy,rho);
			}	
			
			// Zou-He pressure boundary (West)...
			else if (iotype == 12) {			
				vy = iolets[ioi].vBC;
				rho = iolets[ioi].rBC;
				ux = (ft[0]+ft[2]+ft[4] + 2.0*(ft[3]+ft[7]+ft[6]))/rho - 1.0;
				zou_he_pres_west_D2Q9(offst,f2,ft,ux,vy,rho);
			}	
			
			// Zou-He pressure boundary (North)...
			else if (iotype == 13) {
				ux = iolets[ioi].uBC;
				rho = iolets[ioi].rBC;
				vy = (ft[0]+ft[1]+ft[3] + 2.0*(ft[2]+ft[5]+ft[6]))/rho - 1.0;
				zou_he_pres_north_D2Q9(offst,f2,ft,ux,vy,rho);
			}	
			
			// Zou-He pressure boundary (South)...
			else if (iotype == 14) {
				ux = iolets[ioi].uBC;
				rho = iolets[ioi].rBC;
				vy = (ft[0]+ft[1]+ft[3] + 2.0*(ft[4]+ft[7]+ft[8]))/rho - 1.0;
				zou_he_pres_south_D2Q9(offst,f2,ft,ux,vy,rho);
			}
									
		}
	
		// --------------------------------------------------		
		// SAVE - write macros to arrays 
		// --------------------------------------------------
		
		if (save) {
			r[i] = rho;
			u[i] = ux;
			v[i] = vy;
		}
							
	}
}
