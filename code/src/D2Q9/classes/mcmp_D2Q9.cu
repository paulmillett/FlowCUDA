
# include "mcmp_D2Q9.cuh"
# include "mcmp_D2Q9_includes.cuh"
# include "../../IO/GetPot"
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

mcmp_D2Q9::mcmp_D2Q9()
{
	Q = 9;
	GetPot inputParams("input.dat");	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = 1;
	if (nVoxels != Nx*Ny*Nz) cout << "nVoxels does not match Nx, Ny, Nz!" << endl;
	numIolets = inputParams("Lattice/numIolets",0);
	nu = inputParams("LBM/nu",0.1666666);
	gAB = inputParams("LBM/gAB",6.0);
	gAS = inputParams("LBM/gAS",4.5);
	gBS = inputParams("LBM/gBS",4.5); 
	omega = inputParams("LBM/omega",0.0);
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

mcmp_D2Q9::~mcmp_D2Q9()
{
		
}



// --------------------------------------------------------
// Allocate arrays:
// --------------------------------------------------------

void mcmp_D2Q9::allocate()
{
	// allocate array memory (host):
    uH = (float*)malloc(nVoxels*sizeof(float));
    vH = (float*)malloc(nVoxels*sizeof(float));
    rAH = (float*)malloc(nVoxels*sizeof(float));
	rBH = (float*)malloc(nVoxels*sizeof(float));
	sH = (int*)malloc(nVoxels*sizeof(int));
	xH = (int*)malloc(nVoxels*sizeof(int));
	yH = (int*)malloc(nVoxels*sizeof(int));
	nListH = (int*)malloc(nVoxels*Q*sizeof(int));
	voxelTypeH = (int*)malloc(nVoxels*sizeof(int));
	streamIndexH = (int*)malloc(nVoxels*Q*sizeof(int));	
	ioletsH = (iolet2D*)malloc(numIolets*sizeof(iolet2D));
			
	// allocate array memory (device):
	hipMalloc((void **) &u, nVoxels*sizeof(float));
	hipMalloc((void **) &v, nVoxels*sizeof(float));
	hipMalloc((void **) &rA, nVoxels*sizeof(float));
	hipMalloc((void **) &rB, nVoxels*sizeof(float));
	hipMalloc((void **) &f1A, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2A, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &f1B, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2B, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &FxA, nVoxels*sizeof(float));
	hipMalloc((void **) &FxB, nVoxels*sizeof(float));
	hipMalloc((void **) &FyA, nVoxels*sizeof(float));
	hipMalloc((void **) &FyB, nVoxels*sizeof(float));
	hipMalloc((void **) &s, nVoxels*sizeof(int));	
	hipMalloc((void **) &x, nVoxels*sizeof(int));	
	hipMalloc((void **) &y, nVoxels*sizeof(int));	
	hipMalloc((void **) &nList, nVoxels*Q*sizeof(int));	
	hipMalloc((void **) &voxelType, nVoxels*sizeof(int));
	hipMalloc((void **) &streamIndex, nVoxels*Q*sizeof(int));	
	hipMalloc((void **) &iolets, numIolets*sizeof(iolet2D));	
}



// --------------------------------------------------------
// Deallocate arrays:
// --------------------------------------------------------

void mcmp_D2Q9::deallocate()
{
	// free array memory (host):
	free(uH);
	free(vH);
	free(rAH);
	free(rBH);
	free(sH);
	free(xH);
	free(yH);
	free(nListH);
	free(voxelTypeH);
	free(streamIndexH);	
	free(ioletsH);	
		
	// free array memory (device):
	hipFree(u);
	hipFree(v);
	hipFree(rA);
	hipFree(rB);
	hipFree(f1A);
	hipFree(f2A);	
	hipFree(f1B);
	hipFree(f2B);
	hipFree(FxA);
	hipFree(FxB);
	hipFree(FyA);
	hipFree(FyB);
	hipFree(s);
	hipFree(x);
	hipFree(y);
	hipFree(nList);
	hipFree(voxelType);
	hipFree(streamIndex);
	hipFree(iolets);
}



// --------------------------------------------------------
// Copy arrays from host to device:
// --------------------------------------------------------

void mcmp_D2Q9::memcopy_host_to_device()
{
    hipMemcpy(u, uH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(v, vH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rA, rAH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rB, rBH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(s, sH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(x, xH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(y, yH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(nList, nListH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(voxelType, voxelTypeH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(streamIndex, streamIndexH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(iolets, ioletsH, sizeof(iolet2D)*numIolets, hipMemcpyHostToDevice);
}



// --------------------------------------------------------
// Copy arrays from device to host:
// --------------------------------------------------------

void mcmp_D2Q9::memcopy_device_to_host()
{
    hipMemcpy(uH, u, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(vH, v, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rAH, rA, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rBH, rB, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
}



// --------------------------------------------------------
// Initialize lattice as a "box":
// --------------------------------------------------------

void mcmp_D2Q9::create_lattice_box()
{
	GetPot inputParams("input.dat");	
	int flowDir = inputParams("Lattice/flowDir",0);
	int xLBC = inputParams("Lattice/xLBC",0);
	int xUBC = inputParams("Lattice/xUBC",0);
	int yLBC = inputParams("Lattice/yLBC",0);
	int yUBC = inputParams("Lattice/yUBC",0);			
	build_box_lattice_D2Q9(nVoxels,flowDir,Nx,Ny,
	                       xLBC,xUBC,yLBC,yUBC,
	                       voxelTypeH,nListH);
}



// --------------------------------------------------------
// Initialize lattice as a "box" with periodic BC's:
// --------------------------------------------------------

void mcmp_D2Q9::create_lattice_box_periodic()
{
	build_box_lattice_D2Q9(nVoxels,Nx,Ny,voxelTypeH,nListH);
}



// --------------------------------------------------------
// Initialize lattice from "file":
// --------------------------------------------------------

void mcmp_D2Q9::create_lattice_file()
{
	
}



// --------------------------------------------------------
// Build the streamIndex[] array for PUSH streaming:
// --------------------------------------------------------

void mcmp_D2Q9::stream_index_push()
{
	stream_index_push_D2Q9(nVoxels,nListH,streamIndexH);
}



// --------------------------------------------------------
// Build the streamIndex[] array for PULL streaming:
// --------------------------------------------------------

void mcmp_D2Q9::stream_index_pull()
{
	stream_index_pull_D2Q9(nVoxels,nListH,streamIndexH);
}



// --------------------------------------------------------
// Build the streamIndex[] array for PUSH streaming:
// --------------------------------------------------------

void mcmp_D2Q9::stream_index_push_bb()
{
	stream_index_push_bb_D2Q9(nVoxels,nListH,sH,streamIndexH);
}



// --------------------------------------------------------
// Read information about iolet:
// --------------------------------------------------------

void mcmp_D2Q9::read_iolet_info(int i, const char* name) 
{
	char namemod[20];
	GetPot inputParams("input.dat");
	if (i >= 0 and i < numIolets) {
		strcpy(namemod, name);
		strcat(namemod, "/type");
		ioletsH[i].type = inputParams(namemod,1);
		strcpy(namemod, name);
		strcat(namemod, "/uBC");
		ioletsH[i].uBC  = inputParams(namemod,0.0);
		strcpy(namemod, name);
		strcat(namemod, "/vBC");
		ioletsH[i].vBC  = inputParams(namemod,0.0);
		strcpy(namemod, name);
		strcat(namemod, "/rBC");
		ioletsH[i].rBC  = inputParams(namemod,1.0);
		strcpy(namemod, name);
		strcat(namemod, "/pBC");
		ioletsH[i].pBC  = inputParams(namemod,0.0);
	}
	else {
		cout << "iolet index is not correct" << endl;
	}
}



// --------------------------------------------------------
// Swap the populations 1 and 2 for both A and B:
// --------------------------------------------------------

void mcmp_D2Q9::swap_populations()
{
	float* tempA = f1A;
	float* tempB = f1B;
	f1A = f2A;
	f1B = f2B;
	f2A = tempA;
	f2B = tempB;
}



// --------------------------------------------------------
// Setters for host arrays:
// --------------------------------------------------------

void mcmp_D2Q9::setU(int i, float val)
{
	uH[i] = val;
}

void mcmp_D2Q9::setV(int i, float val)
{
	vH[i] = val;
}

void mcmp_D2Q9::setS(int i, int val)
{
	sH[i] = val;
}

void mcmp_D2Q9::setX(int i, int val)
{
	xH[i] = val;
}

void mcmp_D2Q9::setY(int i, int val)
{
	yH[i] = val;
}

void mcmp_D2Q9::setRA(int i, float val)
{
	rAH[i] = val;
}

void mcmp_D2Q9::setRB(int i, float val)
{
	rBH[i] = val;
}

void mcmp_D2Q9::setVoxelType(int i, int val)
{
	voxelTypeH[i] = val;
}



// --------------------------------------------------------
// Getters for host arrays:
// --------------------------------------------------------

float mcmp_D2Q9::getU(int i)
{
	return uH[i];
}

float mcmp_D2Q9::getV(int i)
{
	return vH[i];
}

int mcmp_D2Q9::getS(int i)
{
	return sH[i];
}

float mcmp_D2Q9::getRA(int i)
{
	return rAH[i];
}

float mcmp_D2Q9::getRB(int i)
{
	return rBH[i];
}



// --------------------------------------------------------
// Calls to Kernels:
// --------------------------------------------------------

void mcmp_D2Q9::initial_equilibrium_bb(int nBlocks, int nThreads)
{
	mcmp_initial_equilibrium_bb_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,u,v,nVoxels);	
}

void mcmp_D2Q9::initial_equilibrium_psm(int nBlocks, int nThreads)
{
	mcmp_initial_equilibrium_psm_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,u,v,nVoxels);	
}

void mcmp_D2Q9::initial_particles_on_lattice(float* prx, float* pry, float* prad, int* pIDgrid, 
	int nParts, int nBlocks, int nThreads)
{
	mcmp_initial_particles_on_lattice_D2Q9 
	<<<nBlocks,nThreads>>> (prx,pry,prad,x,y,s,pIDgrid,nVoxels,nParts);	
}

void mcmp_D2Q9::compute_density_bb(int nBlocks, int nThreads)
{
	mcmp_compute_density_bb_D2Q9
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,nVoxels);
}

void mcmp_D2Q9::compute_density_psm(int nBlocks, int nThreads)
{
	mcmp_compute_density_psm_D2Q9
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,nVoxels);
}

void mcmp_D2Q9::update_particles_on_lattice(float* prx, float* pry, float* pvx, float* pvy,
	float* prad, int* pIDgrid, int nParts, int nBlocks, int nThreads)
{
	mcmp_update_particles_on_lattice_D2Q9
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,u,v,prx,pry,pvx,pvy,prad,x,y,s,pIDgrid,nList,nVoxels,nParts);
}

void mcmp_D2Q9::update_particles_on_lattice_psm(float* prx, float* pry, float* B, 
	float* rInner, float* rOuter, int* pIDgrid, int nParts, int nBlocks, int nThreads)
{
	mcmp_update_particles_on_lattice_psm_D2Q9
	<<<nBlocks,nThreads>>> (B,prx,pry,rOuter,rInner,x,y,pIDgrid,nu,nVoxels,nParts);
}

void mcmp_D2Q9::compute_SC_forces_bb(int nBlocks, int nThreads)
{
	mcmp_compute_SC_forces_bb_D2Q9 
	<<<nBlocks,nThreads>>> (rA,rB,FxA,FxB,FyA,FyB,s,nList,gAB,gAS,gBS,nVoxels);	
}

void mcmp_D2Q9::compute_SC_forces_psm(float* B, float* pfx, float* pfy, int* pIDgrid, 
	int nBlocks, int nThreads)
{
	mcmp_compute_SC_forces_psm_D2Q9 
	<<<nBlocks,nThreads>>> (rA,rB,B,FxA,FxB,FyA,FyB,pfx,pfy,nList,pIDgrid,gAB,gAS,gBS,omega,nVoxels);	
}

void mcmp_D2Q9::compute_velocity_bb(int nBlocks, int nThreads)
{
	mcmp_compute_velocity_bb_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,FxA,FxB,FyA,FyB,u,v,s,nVoxels);
}

void mcmp_D2Q9::compute_velocity_psm(float* pvx, float* pvy, float* pfx, float* pfy, 
	float* B, int* pIDgrid, int nBlocks, int nThreads)
{
	mcmp_compute_velocity_psm_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,FxA,FxB,FyA,FyB,u,v,pvx,pvy,pfx,pfy,B,pIDgrid,nVoxels);
}

void mcmp_D2Q9::set_boundary_velocity_psm(int nBlocks, int nThreads)
{
	mcmp_set_boundary_velocity_psm_D2Q9 
	<<<nBlocks,nThreads>>> (rA,rB,FxA,FxB,FyA,FyB,u,v,y,Ny,nVoxels);
}

void mcmp_D2Q9::collide_stream_bb(int nBlocks, int nThreads)
{
	mcmp_collide_stream_bb_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,f2A,f2B,rA,rB,u,v,FxA,FxB,FyA,FyB,s,streamIndex,nu,nVoxels);
}

void mcmp_D2Q9::collide_stream_psm(float* pvx, float* pvy, float* B, int* pIDgrid, 
	float rApart, float rBpart, int nBlocks, int nThreads)
{
	mcmp_collide_stream_psm_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,f2A,f2B,rA,rB,u,v,FxA,FxB,FyA,FyB,pvx,pvy,B,pIDgrid,streamIndex,
	                        rApart,rBpart,nu,nVoxels);
}

void mcmp_D2Q9::bounce_back(int nBlocks, int nThreads)
{
	mcmp_bounce_back_D2Q9
	<<<nBlocks,nThreads>>> (f2A,f2B,s,nList,streamIndex,nVoxels);
}

void mcmp_D2Q9::bounce_back_moving(int nBlocks, int nThreads)
{
	mcmp_bounce_back_moving_D2Q9
	<<<nBlocks,nThreads>>> (f2A,f2B,rA,rB,u,v,s,nList,streamIndex,nVoxels);
}



// --------------------------------------------------------
// Wrtie output:
// --------------------------------------------------------

void mcmp_D2Q9::write_output(std::string tagname, int step)
{
	write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,rAH,rBH,uH,vH);
	write_vtk_structured_grid_2D("rA",step,Nx,Ny,Nz,rAH,uH,vH);
	write_vtk_structured_grid_2D("rB",step,Nx,Ny,Nz,rBH,uH,vH);
}









