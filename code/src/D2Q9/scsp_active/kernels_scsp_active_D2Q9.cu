#include "hip/hip_runtime.h"

# include "kernels_scsp_active_D2Q9.cuh"
# include <stdio.h>



// --------------------------------------------------------
// D2Q9 kernel to re-set the fluid forces to zero: 
// --------------------------------------------------------

__global__ void scsp_active_zero_forces_D2Q9(
	float2* F,
	int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;	
	if (i < nVoxels) {			
		F[i].x = 0.0;
		F[i].y = 0.0;
	}
}



// --------------------------------------------------------
// D2Q9 initialize kernel: 
// --------------------------------------------------------

__global__ void scsp_active_initial_equilibrium_D2Q9(
	float* f1,
	float* r,
	float2* u,
	int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {			
		// useful constants: 
		const float rho = r[i];
		const float ux = u[i].x;
		const float vy = u[i].y;
		const float w0r = rho*4.0/9.0;
		const float wsr = rho*1.0/9.0;
		const float wdr = rho*1.0/36.0;
		const float omusq = 1.0 - 1.5*(ux*ux + vy*vy);	
		const float tux = 3.0*ux;
		const float tvy = 3.0*vy;			
		// equilibrium populations:
		f1[9*i+0] = w0r*(omusq);		
		float cidot3u = tux;
		f1[9*i+1] = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
		cidot3u = tvy;
		f1[9*i+2] = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
		cidot3u = -tux;
		f1[9*i+3] = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
		cidot3u = -tvy;
		f1[9*i+4] = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));	
		cidot3u = tux+tvy;
		f1[9*i+5] = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
		cidot3u = tvy-tux;
		f1[9*i+6] = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
		cidot3u = -(tux+tvy);
		f1[9*i+7] = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
		cidot3u = tux-tvy;
		f1[9*i+8] = wdr*(omusq + cidot3u*(1.0+0.5*cidot3u));
	}
}



// --------------------------------------------------------
// D2Q9 update kernel.
// This algorithm is based on the optimized "stream-collide-
// save" algorithm recommended by T. Kruger in the 
// textbook: "The Lattice Boltzmann Method: Principles
// and Practice".
// --------------------------------------------------------

__global__ void scsp_active_stream_collide_save_D2Q9(
	float* f1,
    float* f2,
	float* r,
	float2* u,
	int* streamIndex,
	int* voxelType,
	float nu,
	int nVoxels)
{

	// -----------------------------------------------
	// define voxel:
	// -----------------------------------------------
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
		
		// --------------------------------------------------		
		// voxel-specific parameters:
		// --------------------------------------------------
		
		int offst = 9*i;	
		float ft[9];
		
		// --------------------------------------------------		
		// STREAMING - load populations from adjacent voxels,
		//             note	that streamIndex[] accounts for
		//             halfway bounceback conditions.
		// --------------------------------------------------
		
		ft[0] = f1[streamIndex[offst+0]];                   
		ft[1] = f1[streamIndex[offst+1]]; 
		ft[2] = f1[streamIndex[offst+2]];  
		ft[3] = f1[streamIndex[offst+3]];  
		ft[4] = f1[streamIndex[offst+4]];  
		ft[5] = f1[streamIndex[offst+5]]; 
		ft[6] = f1[streamIndex[offst+6]];  
		ft[7] = f1[streamIndex[offst+7]];  
		ft[8] = f1[streamIndex[offst+8]]; 		
						
		// --------------------------------------------------
		// MACROS - calculate the velocity and density.
		// --------------------------------------------------
		
		float rho = ft[0] + ft[1] + ft[2] + ft[3] + ft[4] + ft[5] + ft[6] + ft[7] + ft[8];
		float rhoinv = 1.0/rho;
		float ux = rhoinv*(ft[1] + ft[5] + ft[8] - (ft[3] + ft[6] + ft[7]));
		float vy = rhoinv*(ft[2] + ft[5] + ft[6] - (ft[4] + ft[7] + ft[8]));
		
		// --------------------------------------------------
		// COLLISION - perform the BGK collision operator.
		// --------------------------------------------------
			
		// useful constants:
		const float tauinv = 2.0/(6.0*nu + 1.0);   // 1/tau
		const float omtauinv = 1.0 - tauinv;       // 1 - 1/tau
		const float tw0r = (4.0/9.0)*rho*tauinv;   // w[0]*rho/tau
		const float twsr = (1.0/9.0)*rho*tauinv;   // w[1-4]*rho/tau
		const float twdr = (1.0/36.0)*rho*tauinv;  // w[5-8]*rho/tau
		const float omusq = 1.0 - 1.5*(ux*ux + vy*vy);
		const float tux = 3.0*ux;
		const float tvy = 3.0*vy;	
		
		// collision calculations:	
		f2[offst+0] = omtauinv*ft[0] + tw0r*(omusq);		                      // dir-0
		float cidot3u = tux;
		f2[offst+1] = omtauinv*ft[1] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-1
		cidot3u = tvy;
		f2[offst+2] = omtauinv*ft[2] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-2
		cidot3u = -tux;
		f2[offst+3] = omtauinv*ft[3] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-3
		cidot3u = -tvy;
		f2[offst+4] = omtauinv*ft[4] + twsr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-4
		cidot3u = tux+tvy;
		f2[offst+5] = omtauinv*ft[5] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-5
		cidot3u = tvy-tux;
		f2[offst+6] = omtauinv*ft[6] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-6
		cidot3u = -(tux+tvy);
		f2[offst+7] = omtauinv*ft[7] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-7
		cidot3u = tux-tvy;
		f2[offst+8] = omtauinv*ft[8] + twdr*(omusq + cidot3u*(1.0+0.5*cidot3u));  // dir-8			
	
		// --------------------------------------------------		
		// SAVE - write macros to arrays 
		// --------------------------------------------------
		
		r[i] = rho;
		u[i].x = ux;
		u[i].y = vy;
							
	}
}



// --------------------------------------------------------
// D2Q9 update kernel.
// This algorithm is based on the optimized "stream-collide-
// save" algorithm recommended by T. Kruger in the 
// textbook: "The Lattice Boltzmann Method: Principles
// and Practice".
// --------------------------------------------------------

__global__ void scsp_active_stream_collide_save_forcing_D2Q9(
	float* f1,
    float* f2,
	float* r,
	float2* u,
	float2* F,
	int* streamIndex,
	int* voxelType,
	float nu,
	int nVoxels)
{

	// -----------------------------------------------
	// define voxel:
	// -----------------------------------------------
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
		
		// --------------------------------------------------		
		// voxel-specific parameters:
		// --------------------------------------------------
		
		int offst = 9*i;	
		float ft[9];
		
		// --------------------------------------------------		
		// STREAMING - load populations from adjacent voxels,
		//             note	that streamIndex[] accounts for
		//             halfway bounceback conditions.
		// --------------------------------------------------
		
		ft[0] = f1[streamIndex[offst+0]];                   
		ft[1] = f1[streamIndex[offst+1]]; 
		ft[2] = f1[streamIndex[offst+2]];  
		ft[3] = f1[streamIndex[offst+3]];  
		ft[4] = f1[streamIndex[offst+4]];  
		ft[5] = f1[streamIndex[offst+5]]; 
		ft[6] = f1[streamIndex[offst+6]];  
		ft[7] = f1[streamIndex[offst+7]];  
		ft[8] = f1[streamIndex[offst+8]]; 	
				
		// --------------------------------------------------
		// MACROS - calculate the velocity and density (force
		//          corrected).
		// --------------------------------------------------	
				
		float rho = ft[0] + ft[1] + ft[2] + ft[3] + ft[4] + ft[5] + ft[6] + ft[7] + ft[8];
		float rhoinv = 1.0/rho;
		float ux = rhoinv*(ft[1] + ft[5] + ft[8] - (ft[3] + ft[6] + ft[7]) + 0.5*F[i].x);
		float vy = rhoinv*(ft[2] + ft[5] + ft[6] - (ft[4] + ft[7] + ft[8]) + 0.5*F[i].y);
		
		// --------------------------------------------------
		// COLLISION - perform the BGK collision operator
		//             with Guo forcing.
		// --------------------------------------------------
					
		// useful constants:
		const float w0 = 4.0/9.0;
		const float ws = 1.0/9.0;
		const float wd = 1.0/36.0;			
		const float omega = 2.0/(6.0*nu + 1.0);   // 1/tau
		const float omomega = 1.0 - omega;        // 1 - 1/tau
		const float omomega2 = 1.0 - 0.5*omega;   // 1 - 1/(2tau)
		const float omusq = 1.0 - 1.5*(ux*ux + vy*vy);
					
		// direction 0
		float evel = 0.0;       // e dot velocity
		float emiu = 0.0-ux;    // e minus u
		float emiv = 0.0-vy;    // e minus v
		float feq = w0*rho*omusq;
		float frc = w0*(F[i].x*(3.0*emiu) + F[i].y*(3.0*emiv));
		f2[offst+0] = omomega*ft[0] + omega*feq + omomega2*frc;
		
		// direction 1
		evel = ux;
		emiu = 1.0-ux;
		emiv = 0.0-vy;
		feq = ws*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = ws*(F[i].x*(3.0*emiu + 9.0*evel) + F[i].y*(3.0*emiv));
		f2[offst+1] = omomega*ft[1] + omega*feq + omomega2*frc;
		
		// direction 2
		evel = vy; 
		emiu = 0.0-ux;
		emiv = 1.0-vy;
		feq = ws*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = ws*(F[i].x*(3.0*emiu) + F[i].y*(3.0*emiv + 9.0*evel));
		f2[offst+2] = omomega*ft[2] + omega*feq + omomega2*frc;
		
		// direction 3
		evel = -ux;
		emiu = -1.0-ux;
		emiv =  0.0-vy;
		feq = ws*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = ws*(F[i].x*(3.0*emiu - 9.0*evel) + F[i].y*(3.0*emiv));
		f2[offst+3] = omomega*ft[3] + omega*feq + omomega2*frc;
		
		// direction 4
		evel = -vy;
		emiu =  0.0-ux;
		emiv = -1.0-vy;
		feq = ws*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = ws*(F[i].x*(3.0*emiu) + F[i].y*(3.0*emiv - 9.0*evel));
		f2[offst+4] = omomega*ft[4] + omega*feq + omomega2*frc;
		
		// direction 5
		evel = ux + vy;
		emiu = 1.0-ux;
		emiv = 1.0-vy;
		feq = wd*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = wd*(F[i].x*(3.0*emiu + 9.0*evel) + F[i].y*(3.0*emiv + 9.0*evel));
		f2[offst+5] = omomega*ft[5] + omega*feq + omomega2*frc;
		
		// direction 6
		evel = -ux + vy;
		emiu = -1.0-ux;
		emiv =  1.0-vy;
		feq = wd*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = wd*(F[i].x*(3.0*emiu - 9.0*evel) + F[i].y*(3.0*emiv + 9.0*evel));
		f2[offst+6] = omomega*ft[6] + omega*feq + omomega2*frc;
		
		// direction 7
		evel = -ux - vy;
		emiu = -1.0-ux;
		emiv = -1.0-vy;
		feq = wd*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = wd*(F[i].x*(3.0*emiu - 9.0*evel) + F[i].y*(3.0*emiv - 9.0*evel));
		f2[offst+7] = omomega*ft[7] + omega*feq + omomega2*frc;
		
		// direction 8
		evel = ux - vy;
		emiu =  1.0-ux;
		emiv = -1.0-vy;
		feq = wd*rho*(omusq + 3.0*evel + 4.5*evel*evel);
		frc = wd*(F[i].x*(3.0*emiu + 9.0*evel) + F[i].y*(3.0*emiv - 9.0*evel));
		f2[offst+8] = omomega*ft[8] + omega*feq + omomega2*frc;		
	
		// --------------------------------------------------		
		// SAVE - write macros to arrays 
		// --------------------------------------------------
		
		r[i] = rho;
		u[i].x = ux;
		u[i].y = vy;
					
	}
}



// --------------------------------------------------------
// D2Q9 update kernel for the orientation field.
// See: Tjhung et al. Soft Matter (2011) 7:7453
// --------------------------------------------------------

__global__ void scsp_active_update_orientation_D2Q9(
	float2* u,
	float2* p,
	float2* h,
	int* nList,
	float sf,
	float fricR,
	int nVoxels)
{
	
	// -----------------------------------------------
	// define voxel:
	// -----------------------------------------------
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
		
	if (i < nVoxels) {
		
		int offst = 9*i;	
		
		// calculate gradient of velocity field		
		tensor2D W;
		W.xx = (u[nList[offst+1]].x - u[nList[offst+3]].x) / 2.0;  // assume dx=1
		W.xy = (u[nList[offst+2]].x - u[nList[offst+4]].x) / 2.0;  // assume dx=1
		W.yx = (u[nList[offst+1]].y - u[nList[offst+3]].y) / 2.0;  // assume dx=1
		W.yy = (u[nList[offst+2]].y - u[nList[offst+4]].y) / 2.0;  // assume dx=1
				
		// calculate symmetric and anti-symmetric flow field contribution to dpdt:
		tensor2D D = 0.5*(W + transpose(W));
		tensor2D O = 0.5*(W - transpose(W));
		float2 dpdt1 = (sf*D - O)*p[i];
		
		// advection contribution to dpdt (see Wikipedia page on 'material derivative'):
		float pxE = p[nList[offst+1]].x;  // east
		float pxN = p[nList[offst+2]].x;  // north
		float pxW = p[nList[offst+3]].x;  // west
		float pxS = p[nList[offst+4]].x;  // south
		float pyE = p[nList[offst+1]].y;  // east
		float pyN = p[nList[offst+2]].y;  // north
		float pyW = p[nList[offst+3]].y;  // west
		float pyS = p[nList[offst+4]].y;  // south		
		float dpxdx = (pxE - pxW) / 2.0;  // assume dx=1
		float dpxdy = (pxN - pxS) / 2.0;  // assume dx=1
		float dpydx = (pyE - pyW) / 2.0;  // assume dx=1
		float dpydy = (pyN - pyS) / 2.0;  // assume dx=1
		float2 dpdt2 = make_float2(u[i].x*dpxdx + u[i].y*dpxdy, u[i].x*dpydx + u[i].y*dpydy);
		dpdt2 *= -1.0f;
		
		// molecular field contribution to dpdt:
		float2 dpdt3 = -h[i]/fricR;
				
		// update orientation field:
		p[i] += dpdt1 + dpdt2 + dpdt3;   // assume dt=1.0
		
		//p[i] = normalize(p[i]);
		
	}
}



// --------------------------------------------------------
// Kernel to calculate the active stress to be applied
// to the fluid.
// --------------------------------------------------------

__global__ void scsp_active_fluid_stress_D2Q9(
	float2* p,
	float2* h,
	tensor2D* stress,
	int* nList,
	float sf,
	float kapp,
	float activity,	
	int nVoxels)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {
				
		// calculate the elastic (passive) stress tensor:
		int offst = 9*i;
		tensor2D ph = dyadic(p[i],h[i]);
		tensor2D phT = transpose(ph);
		tensor2D symph = 0.5*(ph - phT);
		tensor2D asymph = 0.5*sf*(ph + phT);
		stress[i] = symph - asymph;		
		tensor2D dp;
		dp.xx = (p[nList[offst+1]].x - p[nList[offst+3]].x) / 2.0;  // assume dx=1
		dp.xy = (p[nList[offst+2]].x - p[nList[offst+4]].x) / 2.0;  // assume dx=1
		dp.yx = (p[nList[offst+1]].y - p[nList[offst+3]].y) / 2.0;  // assume dx=1
		dp.yy = (p[nList[offst+2]].y - p[nList[offst+4]].y) / 2.0;  // assume dx=1
		stress[i] += kapp*dp*transpose(dp);
						
		// calculate active stress tensor:
		stress[i] += -activity*dyadic(p[i]);
	}
}



// --------------------------------------------------------
// Kernel to calculate the active forces to be applied
// to the fluid.
// --------------------------------------------------------

__global__ void scsp_active_fluid_forces_D2Q9(
	float2* F,
	tensor2D* stress,
	int* nList,
	int nVoxels)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {
		int offst = 9*i;	
		// divergence of stress tensor:
		float dsxxdx = (stress[nList[offst+1]].xx - stress[nList[offst+3]].xx) / 2.0;  // assume dx=1
		float dsxydy = (stress[nList[offst+2]].xy - stress[nList[offst+4]].xy) / 2.0;  // assume dx=1
		float dsyxdx = (stress[nList[offst+1]].yx - stress[nList[offst+3]].yx) / 2.0;  // assume dx=1
		float dsyydy = (stress[nList[offst+2]].yy - stress[nList[offst+4]].yy) / 2.0;  // assume dx=1
		float2 force = make_float2(dsxxdx + dsxydy, dsyxdx + dsyydy);
		F[i] += force;
	}
	
}



// --------------------------------------------------------
// Kernel to calculate the molecular field "h" = dFdp
// --------------------------------------------------------

__global__ void scsp_active_fluid_molecular_field_D2Q9(
	float2* h,
	float2* p,
	tensor2D* stress,
	int* nList,
	float alpha,
	float kapp,
	int nVoxels)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {		
		int offst = 9*i;	
		// molecular field:
		float px  = p[i].x;
		float py  = p[i].y;
		float pxE = p[nList[offst+1]].x;  // east
		float pxN = p[nList[offst+2]].x;  // north
		float pxW = p[nList[offst+3]].x;  // west
		float pxS = p[nList[offst+4]].x;  // south
		float pyE = p[nList[offst+1]].y;  // east
		float pyN = p[nList[offst+2]].y;  // north
		float pyW = p[nList[offst+3]].y;  // west
		float pyS = p[nList[offst+4]].y;  // south		
		float pmag = sqrt(px*px + py*py);
		float dfdpmag = alpha*(pmag*pmag*pmag - pmag);
		float laplpx = (pxE + pxW + pxN + pxS - 4.0*px);   // assume dx=1
		float laplpy = (pyE + pyW + pyN + pyS - 4.0*py);   // assume dx=1
		h[i].x = dfdpmag*px - kapp*laplpx;
		h[i].y = dfdpmag*py - kapp*laplpy;		
	}
}



// --------------------------------------------------------
// Kernel to calculate the molecular field "h" = dFdp
// here including the order parameter 'phi'
// --------------------------------------------------------

__global__ void scsp_active_fluid_molecular_field_with_phi_D2Q9(
	float* phi,
	float2* h,
	float2* p,
	tensor2D* stress,
	int* nList,
	float alpha,
	float kapp,
	int nVoxels)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {		
		int offst = 9*i;	
		// molecular field:
		float px  = p[i].x;
		float py  = p[i].y;
		float pxE = p[nList[offst+1]].x;  // east
		float pxN = p[nList[offst+2]].x;  // north
		float pxW = p[nList[offst+3]].x;  // west
		float pxS = p[nList[offst+4]].x;  // south
		float pyE = p[nList[offst+1]].y;  // east
		float pyN = p[nList[offst+2]].y;  // north
		float pyW = p[nList[offst+3]].y;  // west
		float pyS = p[nList[offst+4]].y;  // south		
		float pmag = sqrt(px*px + py*py);
		float phicoeff = (phi[i] - 0.5)/0.5;   // phi_critical = 0.5
		float dfdpmag = alpha*(pmag*pmag*pmag - pmag*phicoeff);
		float laplpx = (pxE + pxW + pxN + pxS - 4.0*px);   // assume dx=1
		float laplpy = (pyE + pyW + pyN + pyS - 4.0*py);   // assume dx=1
		h[i].x = dfdpmag*px - kapp*laplpx;
		h[i].y = dfdpmag*py - kapp*laplpy;		
	}
}



// --------------------------------------------------------
// Kernel to calculate the chemical potential of the 
// order parameter:
// --------------------------------------------------------

__global__ void scsp_active_fluid_chemical_potential_D2Q9(
	float* phi,
	float* chempot,
	int* nList,
	float a,
	float kapphi,
	int nVoxels)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {		
		int offst = 9*i;
		float phii = phi[i];
		float phiE = phi[nList[offst+1]];  // east
		float phiN = phi[nList[offst+2]];  // north
		float phiW = phi[nList[offst+3]];  // west
		float phiS = phi[nList[offst+4]];  // south
		float lapl = (phiE + phiW + phiN + phiS - 4.0*phii);  // assume dx=1
		chempot[i] = a*(4.0*phii*phii*phii - 6.0*phii*phii + 2.0*phii) - kapphi*lapl;
	}		
}



// --------------------------------------------------------
// Kernel to calculate the interfacial capillary force
// the fluid:
// --------------------------------------------------------

__global__ void scsp_active_fluid_capillary_force_D2Q9(
	float* phi,
	float* chempot,
	float2* F,
	int* nList,
	int nVoxels)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {		
		int offst = 9*i;
		float gradphix = (phi[nList[offst+1]] - phi[nList[offst+3]])/2.0;  // assume dx=1
		float gradphiy = (phi[nList[offst+2]] - phi[nList[offst+4]])/2.0;  // assume dx=1
		float2 capF = chempot[i]*make_float2(gradphix,gradphiy);
		F[i] += capF;
	}		
}



// --------------------------------------------------------
// Kernel to update the order parameter phi:
// --------------------------------------------------------

__global__ void scsp_active_fluid_update_phi_D2Q9(
	float* phi,
	float* chempot,
	float2* u,
	int* nList,
	float mob,
	int nVoxels)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nVoxels) {		
		int offst = 9*i;
		float cpi = chempot[i];
		float cpE = chempot[nList[offst+1]];  // east
		float cpN = chempot[nList[offst+2]];  // north
		float cpW = chempot[nList[offst+3]];  // west
		float cpS = chempot[nList[offst+4]];  // south
		float lapl = (cpE + cpW + cpN + cpS - 4.0*cpi);  // assume dx=1		
		float gradphix = (phi[nList[offst+1]] - phi[nList[offst+3]])/2.0;  // assume dx=1
		float gradphiy = (phi[nList[offst+2]] - phi[nList[offst+4]])/2.0;  // assume dx=1
		float2 gradphi = make_float2(gradphix,gradphiy);
		phi[i] += mob*lapl - dot(u[i],gradphi);   // assume dt=1
	}		
}



