#include "hip/hip_runtime.h"

# include "class_scsp_active_3phi_D2Q9.cuh"
# include "kernels_scsp_active_D2Q9.cuh"
# include "../../IO/GetPot"
# include <math.h>
# include <iostream>
# include <iomanip>
# include <fstream>
# include <sstream>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

class_scsp_active_3phi_D2Q9::class_scsp_active_3phi_D2Q9()
{
	Q = 9;
	GetPot inputParams("input.dat");	
	nVoxels = inputParams("Lattice/nVoxels",0);
	numIolets = inputParams("Lattice/numIolets",0);
	nu = inputParams("LBM/nu",0.1666666);
	nu_in = inputParams("LBM/nu2",0.1666666);
	nu_out = nu;
	sf = inputParams("LBM/sf",1.0);
	fricR = inputParams("LBM/fricR",1.0);
	activity = inputParams("LBM/activity",0.0);
	alpha = inputParams("LBM/alpha",1.0);
	beta = inputParams("LBM/beta",0.0);
	kapp = inputParams("LBM/kapp",1.0);
	kapphi = inputParams("LBM/kapphi",1.0);
	mob = inputParams("LBM/mob",1.0);
	a = inputParams("LBM/a",1.0);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = inputParams("Lattice/Nz",1);	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

class_scsp_active_3phi_D2Q9::~class_scsp_active_3phi_D2Q9()
{
		
}



// --------------------------------------------------------
// Allocate arrays:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::allocate()
{
	// allocate array memory (host):
    rH = (float*)malloc(nVoxels*sizeof(float));
	phi1H = (float*)malloc(nVoxels*sizeof(float));
	phi2H = (float*)malloc(nVoxels*sizeof(float));
	phi3H = (float*)malloc(nVoxels*sizeof(float));
	uH = (float2*)malloc(nVoxels*sizeof(float2));
	pH = (float2*)malloc(nVoxels*sizeof(float2));
	hH = (float2*)malloc(nVoxels*sizeof(float2));
	nListH = (int*)malloc(nVoxels*Q*sizeof(int));
	voxelTypeH = (int*)malloc(nVoxels*sizeof(int));
	streamIndexH = (int*)malloc(nVoxels*Q*sizeof(int));	
	phisum0H = (float*)malloc(3*sizeof(float));
			
	// allocate array memory (device):
	hipMalloc((void **) &r, nVoxels*sizeof(float));
	hipMalloc((void **) &u, nVoxels*sizeof(float2));
	hipMalloc((void **) &F, nVoxels*sizeof(float2));
	hipMalloc((void **) &p, nVoxels*sizeof(float2));
	hipMalloc((void **) &h, nVoxels*sizeof(float2));
	hipMalloc((void **) &f1, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &phi1, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &phi2, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &phi3, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &chempot1, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &chempot2, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &chempot3, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &stress, nVoxels*sizeof(tensor2D));		
	hipMalloc((void **) &voxelType, nVoxels*sizeof(int));
	hipMalloc((void **) &streamIndex, nVoxels*Q*sizeof(int));	
	hipMalloc((void **) &nList, nVoxels*Q*sizeof(int));
	hipMalloc((void **) &phisum, 3*sizeof(float));
	hipMalloc((void **) &phisum0, 3*sizeof(float));
}



// --------------------------------------------------------
// Deallocate arrays:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::deallocate()
{
	// free array memory (host):
	free(uH);
	free(pH);
	free(hH);
	free(rH);
	free(phi1H);
	free(phi2H);
	free(phi3H);
	free(nListH);
	free(voxelTypeH);
	free(streamIndexH);
	free(phisum0H);
		
	// free array memory (device):
	hipFree(F);
	hipFree(u);
	hipFree(p);
	hipFree(h);
	hipFree(r);
	hipFree(f1);
	hipFree(f2);
	hipFree(phi1);
	hipFree(phi2);
	hipFree(phi3);
	hipFree(chempot1);
	hipFree(chempot2);
	hipFree(chempot3);
	hipFree(stress);
	hipFree(voxelType);
	hipFree(streamIndex);
	hipFree(nList);
	hipFree(phisum);
	hipFree(phisum0);
}



// --------------------------------------------------------
// Copy arrays from host to device:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::memcopy_host_to_device()
{
    hipMemcpy(r, rH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(u, uH, sizeof(float2)*nVoxels, hipMemcpyHostToDevice);	
	hipMemcpy(p, pH, sizeof(float2)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi1, phi1H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi2, phi2H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi3, phi3H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(nList, nListH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(voxelType, voxelTypeH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(streamIndex, streamIndexH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
}



// --------------------------------------------------------
// Copy arrays from device to host:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::memcopy_device_to_host()
{
    hipMemcpy(rH, r, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(uH, u, sizeof(float2)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(pH, p, sizeof(float2)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(hH, h, sizeof(float2)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(phi1H, phi1, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(phi2H, phi2, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(phi3H, phi3, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
}



// --------------------------------------------------------
// Initialize lattice as a "box" with periodic BC's:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::create_lattice_box_periodic()
{
	build_box_lattice_D2Q9(nVoxels,Nx,Ny,voxelTypeH,nListH);
}



// --------------------------------------------------------
// Initialize lattice as a "box" with periodic BC's:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::create_lattice_box_shear()
{
	build_box_lattice_shear_D2Q9(nVoxels,Nx,Ny,voxelTypeH,nListH);
}



// --------------------------------------------------------
// Build the streamIndex[] array for PULL streaming:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::stream_index_pull()
{
	stream_index_pull_D2Q9(nVoxels,nListH,streamIndexH);
}



// --------------------------------------------------------
// Setters for host arrays:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::setU(int i, float val)
{
	uH[i].x = val;
}

void class_scsp_active_3phi_D2Q9::setV(int i, float val)
{
	uH[i].y = val;
}

void class_scsp_active_3phi_D2Q9::setPx(int i, float val)
{
	pH[i].x = val;
}

void class_scsp_active_3phi_D2Q9::setPy(int i, float val)
{
	pH[i].y = val;
}

void class_scsp_active_3phi_D2Q9::setR(int i, float val)
{
	rH[i] = val;
}

void class_scsp_active_3phi_D2Q9::setPhi1(int i, float val)
{
	phi1H[i] = val;
}

void class_scsp_active_3phi_D2Q9::setPhi2(int i, float val)
{
	phi2H[i] = val;
}

void class_scsp_active_3phi_D2Q9::setPhi3(int i, float val)
{
	phi3H[i] = val;
}

void class_scsp_active_3phi_D2Q9::setVoxelType(int i, int val)
{
	voxelTypeH[i] = val;
}

void class_scsp_active_3phi_D2Q9::setPhiSum(float sum1, float sum2, float sum3)
{
	phisum0H[0] = sum1; phisum0H[1] = sum2; phisum0H[2] = sum3;
	hipMemcpy(phisum0, phisum0H, sizeof(float)*3, hipMemcpyHostToDevice);
}



// --------------------------------------------------------
// Getters for host arrays:
// --------------------------------------------------------

float class_scsp_active_3phi_D2Q9::getU(int i)
{
	return uH[i].x;
}

float class_scsp_active_3phi_D2Q9::getV(int i)
{
	return uH[i].y;
}

float class_scsp_active_3phi_D2Q9::getR(int i)
{
	return rH[i];
}

float class_scsp_active_3phi_D2Q9::getPhi1(int i)
{
	return phi1H[i];
}

float class_scsp_active_3phi_D2Q9::getPhi2(int i)
{
	return phi2H[i];
}

float class_scsp_active_3phi_D2Q9::getPhi3(int i)
{
	return phi3H[i];
}



// --------------------------------------------------------
// Call to "scsp_initial_equilibrium_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::initial_equilibrium(int nBlocks, int nThreads)
{
	scsp_active_initial_equilibrium_D2Q9 
	<<<nBlocks,nThreads>>> (f1,r,u,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_zero_forces_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::zero_forces(int nBlocks, int nThreads)
{
	scsp_active_zero_forces_D2Q9 
	<<<nBlocks,nThreads>>> (F,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_stream_collide_save_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::stream_collide_save(int nBlocks, int nThreads)
{
	scsp_active_stream_collide_save_D2Q9 
	<<<nBlocks,nThreads>>> (f1,f2,r,u,streamIndex,voxelType,nu,nVoxels);
	float* temp = f1;
	f1 = f2;
	f2 = temp;
}



// --------------------------------------------------------
// Call to "scsp_stream_collide_save_forcing_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::stream_collide_save_forcing(int nBlocks, int nThreads)
{
	scsp_active_stream_collide_save_forcing_D2Q9 
	<<<nBlocks,nThreads>>> (f1,f2,r,u,F,streamIndex,voxelType,nu,nVoxels);
	float* temp = f1;
	f1 = f2;
	f2 = temp;
}



// --------------------------------------------------------
// Call to "scsp_active_stream_collide_save_forcing_varvisc_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::stream_collide_save_forcing_varvisc(int nBlocks, int nThreads)
{
	scsp_active_stream_collide_save_forcing_varvisc_D2Q9 
	<<<nBlocks,nThreads>>> (f1,f2,r,phi2,u,F,streamIndex,voxelType,nu_in,nu_out,nVoxels);
	float* temp = f1;
	f1 = f2;
	f2 = temp;
}



// --------------------------------------------------------
// Call to "scsp_active_set_boundary_velocity_D2Q9" kernel:
// NOTE: This should be called AFTER the collide-streaming
//       step.  It should be the last calculation for the 
//       fluid update.  
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::set_wall_velocity_ydir(float uWall, int nBlocks, int nThreads)
{
	scsp_active_set_boundary_velocity_D2Q9 
	<<<nBlocks,nThreads>>> (uWall,f1,u,r,Nx,Ny,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_update_orientation_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_update_orientation(int nBlocks, int nThreads)
{
	scsp_active_update_orientation_D2Q9 
	<<<nBlocks,nThreads>>> (u,p,h,nList,sf,fricR,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_update_orientation_diffusive_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_update_orientation_diffusive(int nBlocks, int nThreads)
{
	scsp_active_update_orientation_diffusive_D2Q9 
	<<<nBlocks,nThreads>>> (p,h,nList,fricR,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_stress_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_stress(int nBlocks, int nThreads)
{
	scsp_active_fluid_stress_D2Q9 
	<<<nBlocks,nThreads>>> (p,h,stress,nList,sf,kapp,activity,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_forces_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_forces(int nBlocks, int nThreads)
{
	scsp_active_fluid_forces_D2Q9 
	<<<nBlocks,nThreads>>> (F,stress,nList,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_molecular_field_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_molecular_field(int nBlocks, int nThreads)
{
	scsp_active_fluid_molecular_field_D2Q9 
	<<<nBlocks,nThreads>>> (h,p,stress,nList,alpha,kapp,nVoxels);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_molecular_field_with_phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_molecular_field_with_phi(int nBlocks, int nThreads)
{
	scsp_active_fluid_molecular_field_with_phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,h,p,stress,nList,alpha,kapp,beta,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_chemical_potential_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_chemical_potential(int nBlocks, int nThreads)
{
	scsp_active_fluid_chemical_potential_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,chempot1,chempot2,chempot3,p,nList,a,alpha,kapphi,beta,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_capillary_force_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_capillary_force(int nBlocks, int nThreads)
{
	scsp_active_fluid_capillary_force_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,chempot1,chempot2,chempot3,F,nList,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_update_phi_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_update_phi(int nBlocks, int nThreads)
{
	scsp_active_fluid_update_phi_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,chempot1,chempot2,chempot3,u,nList,mob,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_update_phi_3phi_alternative_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_update_phi_alternative(int nBlocks, int nThreads)
{
	scsp_active_fluid_update_phi_3phi_alternative_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,chempot1,chempot2,chempot3,u,nList,mob,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_zero_phisum_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_zero_phisum_3phi(int nBlocks, int nThreads)
{
	scsp_active_fluid_zero_phisum_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phisum,nVoxels);	
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_sum_phi_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_sum_phi_3phi(int nBlocks, int nThreads)
{
	scsp_active_fluid_sum_phi_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,phisum,nVoxels);	
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_enforce_conservation_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_enforce_conservation_3phi(int nBlocks, int nThreads)
{
	scsp_active_fluid_enforce_conservation_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,phisum,phisum0,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_update_phi_diffusive_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_update_phi_diffusive(int nBlocks, int nThreads)
{
	scsp_active_fluid_update_phi_diffusive_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,chempot1,u,nList,mob,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_set_velocity_field_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_set_velocity_field(int nBlocks, int nThreads)
{
	scsp_active_fluid_set_velocity_field_D2Q9 
	<<<nBlocks,nThreads>>> (u,p,0.00005,nVoxels);
}







// --------------------------------------------------------
// Wrtie output:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::write_output(std::string tagname, int step, int iskip, int jskip)
{
	
	// -----------------------------------------------
	//	Define the file location and name:
	// -----------------------------------------------
	
	ofstream outfile;
	std::stringstream filenamecombine;
	filenamecombine << "vtkoutput/" << tagname << "_" << step << ".vtk";
	string filename = filenamecombine.str();
	outfile.open(filename.c_str(), ios::out | ios::app);
	
	// -----------------------------------------------
	//	find output dimensions considering
	//  iskip, jskip:
	// -----------------------------------------------
	
	int Nxs = Nx/iskip;
	int Nys = Ny/jskip;
	if (Nx%2 && iskip>1) Nxs++;  // if odd, then add 1
	if (Ny%2 && jskip>1) Nys++;
	
	// -----------------------------------------------
	//	Write the 'vtk' file header:
	// -----------------------------------------------
	
	string d = "   ";
	outfile << "# vtk DataFile Version 3.1" << endl;
	outfile << "VTK file containing grid data" << endl;
	outfile << "ASCII" << endl;
	outfile << " " << endl;
	outfile << "DATASET STRUCTURED_POINTS" << endl;
	outfile << "DIMENSIONS" << d << Nxs << d << Nys << d << Nz << endl;
	outfile << "ORIGIN " << d << 0 << d << 0 << d << 0 << endl;
	outfile << "SPACING" << d << 1.0*iskip << d << 1.0*jskip << d << 1.0 << endl;
	outfile << " " << endl;
	outfile << "POINT_DATA " << Nxs*Nys*Nz << endl;
	outfile << "SCALARS " << tagname << " float" << endl;
	outfile << "LOOKUP_TABLE default" << endl;
	
	// -----------------------------------------------
	// Write the 'rho' data:
	// NOTE: x-data increases fastest,
	//       then y-data
	// -----------------------------------------------
	
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j+=jskip) {
			for (int i=0; i<Nx; i+=iskip) {
				int ndx = k*Nx*Ny + j*Nx + i;
				outfile << fixed << setprecision(5) << phi1H[ndx] - phi2H[ndx] << endl;
			}
		}
	}	
	
	// -----------------------------------------------				
	// Write the 'velocity' data:
	// NOTE: x-data increases fastest,
	//       then y-data	
	// -----------------------------------------------
	
	outfile << "   " << endl;
	outfile << "VECTORS Velocity float" << endl;		
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j+=jskip) {
			for (int i=0; i<Nx; i+=iskip) {
				int ndx = k*Nx*Ny + j*Nx + i;
				outfile << fixed << setprecision(5) << uH[ndx].x << " "
					                                << uH[ndx].y << " " 
													<< 0.0 << endl;
			}
		}
	}
	
	// -----------------------------------------------				
	// Write the 'orientation' data:
	// NOTE: x-data increases fastest,
	//       then y-data	
	// -----------------------------------------------
	
	outfile << "   " << endl;
	outfile << "VECTORS Orientation float" << endl;		
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j+=jskip) {
			for (int i=0; i<Nx; i+=iskip) {
				int ndx = k*Nx*Ny + j*Nx + i;
				outfile << fixed << setprecision(5) << pH[ndx].x << " "
					                                << pH[ndx].y << " " 
													<< 0.0 << endl;
			}
		}
	}
		
	// -----------------------------------------------
	//	Close the file:
	// -----------------------------------------------
	
	outfile.close();
	
}



// --------------------------------------------------------
// Wrtie output for the droplet properties:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::write_output_droplet_properties(int step)
{
	
	// -----------------------------------------
	// Define the file location and name:
	// -----------------------------------------
	
	ofstream outfile;
	std::stringstream filenamecombine;
	filenamecombine << "vtkoutput/" << "droplet_data.dat";
	string filename = filenamecombine.str();
	outfile.open(filename.c_str(), ios::out | ios::app);
	
	// -----------------------------------------
	// Calculate volume of droplets:
	// -----------------------------------------
	
	float vol1 = 0.0;
	float vol2 = 0.0;
	float vol3 = 0.0;
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {
			int ndx = j*Nx + i;
			vol1 += phi1H[ndx];
			vol2 += phi2H[ndx];
			vol3 += phi3H[ndx];
		}
	}
	
	// -----------------------------------------
	// Enforce conservation of volume of fluids:
	// -----------------------------------------
	
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {
			int ndx = j*Nx + i;
			phi1H[ndx] += (phisum0H[0] - vol1)/float(nVoxels);
			phi2H[ndx] += (phisum0H[1] - vol2)/float(nVoxels);
			phi3H[ndx] += (phisum0H[2] - vol3)/float(nVoxels);
		}
	}	
	hipMemcpy(phi1, phi1H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi2, phi2H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi3, phi3H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	
	// -----------------------------------------
	// Calculate center-of-mass of droplets,
	// taking into account the PBC's in the 
	// x-direction (see the Wiki page on c.o.m)
	// -----------------------------------------
	
	float p0ave1 = 0.0;  // droplet 1
	float q0ave1 = 0.0;
	float p0ave2 = 0.0;  // droplet 2
	float q0ave2 = 0.0;
	float yf1 = 0.0;
	float yf2 = 0.0;
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {
			int ndx = j*Nx + i;
			// x-dir
			float t0x = (float(i)/float(Nx))*2*M_PI;
			float p0x = cos(t0x);
			float q0x = sin(t0x);
			p0ave1 += p0x*phi1H[ndx];
			q0ave1 += q0x*phi1H[ndx];	
			p0ave2 += p0x*phi2H[ndx];
			q0ave2 += q0x*phi2H[ndx];
			// y-dir
			yf1 += float(j)*phi1H[ndx];
			yf2 += float(j)*phi2H[ndx];	
		}
	}
	float pxave1 = p0ave1/vol1;
	float qxave1 = q0ave1/vol1;
	float pxave2 = p0ave2/vol2;
	float qxave2 = q0ave2/vol2;
	float txave1 = atan2(-qxave1,-pxave1) + M_PI;
	float txave2 = atan2(-qxave2,-pxave2) + M_PI;
	float xf1 = float(Nx)*txave1/(2*M_PI);
	float xf2 = float(Nx)*txave2/(2*M_PI);
	yf1 /= vol1;
	yf2 /= vol2;
	
	// -----------------------------------------
	// Calculate velocity of droplets:
	// -----------------------------------------
		
	if (step == 0) {
		velx1 = 0.0; vely1 = 0.0;
		velx2 = 0.0; vely2 = 0.0;	
	} 
	else {
		float dtstep = float(step - stepprev);
		float dx1 = xf1 - xf1prev;
		float dy1 = yf1 - yf1prev;
		dx1 = dx1 - roundf(dx1/float(Nx))*float(Nx);
		velx1 = dx1/dtstep;
		vely1 = dy1/dtstep;
		float dx2 = xf2 - xf2prev;
		float dy2 = yf2 - yf2prev;
		dx2 = dx2 - roundf(dx2/float(Nx))*float(Nx);
		velx2 = dx2/dtstep;
		vely2 = dy2/dtstep;
	}	
	
	stepprev = step;
	xf1prev = xf1; yf1prev = yf1;
	xf2prev = xf2; yf2prev = yf2;
		
	// -----------------------------------------
	// Print to file:
	// -----------------------------------------
		
	outfile << fixed << setprecision(1) << setw(10) << step << " "
		             << setprecision(4) << setw(12) << vol1 << " " 
					 << setprecision(4) << setw(10) << xf1  << " " 
					 << setprecision(4) << setw(10)	<< yf1  << " "
					 << setprecision(7) << setw(11)	<< velx1 << " "
					 << setprecision(7) << setw(11)	<< vely1 << " "
					 << setprecision(4) << setw(12) << vol2 << " " 
					 << setprecision(4) << setw(10) << xf2  << " " 
					 << setprecision(4) << setw(10) << yf2  << " " 
					 << setprecision(7) << setw(11)	<< velx2 << " "
					 << setprecision(7) << setw(11)	<< vely2 << " " << endl;	
		
}





