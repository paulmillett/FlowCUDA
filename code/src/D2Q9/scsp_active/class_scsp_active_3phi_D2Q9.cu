#include "hip/hip_runtime.h"

# include "class_scsp_active_3phi_D2Q9.cuh"
# include "kernels_scsp_active_D2Q9.cuh"
# include "../../IO/GetPot"
# include <math.h>
# include <iostream>
# include <iomanip>
# include <fstream>
# include <sstream>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

class_scsp_active_3phi_D2Q9::class_scsp_active_3phi_D2Q9()
{
	Q = 9;
	GetPot inputParams("input.dat");	
	nVoxels = inputParams("Lattice/nVoxels",0);
	numIolets = inputParams("Lattice/numIolets",0);
	nu = inputParams("LBM/nu",0.1666666);
	sf = inputParams("LBM/sf",1.0);
	fricR = inputParams("LBM/fricR",1.0);
	activity = inputParams("LBM/activity",0.0);
	alpha = inputParams("LBM/alpha",1.0);
	beta = inputParams("LBM/beta",0.0);
	kapp = inputParams("LBM/kapp",1.0);
	kapphi = inputParams("LBM/kapphi",1.0);
	mob = inputParams("LBM/mob",1.0);
	a = inputParams("LBM/a",1.0);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = inputParams("Lattice/Nz",1);	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

class_scsp_active_3phi_D2Q9::~class_scsp_active_3phi_D2Q9()
{
		
}



// --------------------------------------------------------
// Allocate arrays:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::allocate()
{
	// allocate array memory (host):
    rH = (float*)malloc(nVoxels*sizeof(float));
	phi1H = (float*)malloc(nVoxels*sizeof(float));
	phi2H = (float*)malloc(nVoxels*sizeof(float));
	phi3H = (float*)malloc(nVoxels*sizeof(float));
	uH = (float2*)malloc(nVoxels*sizeof(float2));
	pH = (float2*)malloc(nVoxels*sizeof(float2));
	hH = (float2*)malloc(nVoxels*sizeof(float2));
	nListH = (int*)malloc(nVoxels*Q*sizeof(int));
	voxelTypeH = (int*)malloc(nVoxels*sizeof(int));
	streamIndexH = (int*)malloc(nVoxels*Q*sizeof(int));	
			
	// allocate array memory (device):
	hipMalloc((void **) &r, nVoxels*sizeof(float));
	hipMalloc((void **) &u, nVoxels*sizeof(float2));
	hipMalloc((void **) &F, nVoxels*sizeof(float2));
	hipMalloc((void **) &p, nVoxels*sizeof(float2));
	hipMalloc((void **) &h, nVoxels*sizeof(float2));
	hipMalloc((void **) &f1, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &phi1, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &phi2, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &phi3, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &chempot1, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &chempot2, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &chempot3, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &stress, nVoxels*sizeof(tensor2D));		
	hipMalloc((void **) &voxelType, nVoxels*sizeof(int));
	hipMalloc((void **) &streamIndex, nVoxels*Q*sizeof(int));	
	hipMalloc((void **) &nList, nVoxels*Q*sizeof(int));	
}



// --------------------------------------------------------
// Deallocate arrays:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::deallocate()
{
	// free array memory (host):
	free(uH);
	free(pH);
	free(hH);
	free(rH);
	free(phi1H);
	free(phi2H);
	free(phi3H);
	free(nListH);
	free(voxelTypeH);
	free(streamIndexH);	
		
	// free array memory (device):
	hipFree(F);
	hipFree(u);
	hipFree(p);
	hipFree(h);
	hipFree(r);
	hipFree(f1);
	hipFree(f2);
	hipFree(phi1);
	hipFree(phi2);
	hipFree(phi3);
	hipFree(chempot1);
	hipFree(chempot2);
	hipFree(chempot3);
	hipFree(stress);
	hipFree(voxelType);
	hipFree(streamIndex);
	hipFree(nList);
}



// --------------------------------------------------------
// Copy arrays from host to device:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::memcopy_host_to_device()
{
    hipMemcpy(r, rH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(u, uH, sizeof(float2)*nVoxels, hipMemcpyHostToDevice);	
	hipMemcpy(p, pH, sizeof(float2)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi1, phi1H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi2, phi2H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(phi3, phi3H, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(nList, nListH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(voxelType, voxelTypeH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(streamIndex, streamIndexH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
}



// --------------------------------------------------------
// Copy arrays from device to host:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::memcopy_device_to_host()
{
    hipMemcpy(rH, r, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(uH, u, sizeof(float2)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(pH, p, sizeof(float2)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(hH, h, sizeof(float2)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(phi1H, phi1, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(phi2H, phi2, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(phi3H, phi3, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
}



// --------------------------------------------------------
// Initialize lattice as a "box" with periodic BC's:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::create_lattice_box_periodic()
{
	build_box_lattice_D2Q9(nVoxels,Nx,Ny,voxelTypeH,nListH);
}



// --------------------------------------------------------
// Build the streamIndex[] array for PULL streaming:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::stream_index_pull()
{
	stream_index_pull_D2Q9(nVoxels,nListH,streamIndexH);
}



// --------------------------------------------------------
// Setters for host arrays:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::setU(int i, float val)
{
	uH[i].x = val;
}

void class_scsp_active_3phi_D2Q9::setV(int i, float val)
{
	uH[i].y = val;
}

void class_scsp_active_3phi_D2Q9::setPx(int i, float val)
{
	pH[i].x = val;
}

void class_scsp_active_3phi_D2Q9::setPy(int i, float val)
{
	pH[i].y = val;
}

void class_scsp_active_3phi_D2Q9::setR(int i, float val)
{
	rH[i] = val;
}

void class_scsp_active_3phi_D2Q9::setPhi1(int i, float val)
{
	phi1H[i] = val;
}

void class_scsp_active_3phi_D2Q9::setPhi2(int i, float val)
{
	phi2H[i] = val;
}

void class_scsp_active_3phi_D2Q9::setPhi3(int i, float val)
{
	phi3H[i] = val;
}

void class_scsp_active_3phi_D2Q9::setVoxelType(int i, int val)
{
	voxelTypeH[i] = val;
}



// --------------------------------------------------------
// Getters for host arrays:
// --------------------------------------------------------

float class_scsp_active_3phi_D2Q9::getU(int i)
{
	return uH[i].x;
}

float class_scsp_active_3phi_D2Q9::getV(int i)
{
	return uH[i].y;
}

float class_scsp_active_3phi_D2Q9::getR(int i)
{
	return rH[i];
}

float class_scsp_active_3phi_D2Q9::getPhi1(int i)
{
	return phi1H[i];
}

float class_scsp_active_3phi_D2Q9::getPhi2(int i)
{
	return phi2H[i];
}

float class_scsp_active_3phi_D2Q9::getPhi3(int i)
{
	return phi3H[i];
}



// --------------------------------------------------------
// Call to "scsp_initial_equilibrium_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::initial_equilibrium(int nBlocks, int nThreads)
{
	scsp_active_initial_equilibrium_D2Q9 
	<<<nBlocks,nThreads>>> (f1,r,u,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_zero_forces_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::zero_forces(int nBlocks, int nThreads)
{
	scsp_active_zero_forces_D2Q9 
	<<<nBlocks,nThreads>>> (F,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_stream_collide_save_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::stream_collide_save(int nBlocks, int nThreads)
{
	scsp_active_stream_collide_save_D2Q9 
	<<<nBlocks,nThreads>>> (f1,f2,r,u,streamIndex,voxelType,nu,nVoxels);
	float* temp = f1;
	f1 = f2;
	f2 = temp;
}



// --------------------------------------------------------
// Call to "scsp_stream_collide_save_forcing_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::stream_collide_save_forcing(int nBlocks, int nThreads)
{
	scsp_active_stream_collide_save_forcing_D2Q9 
	<<<nBlocks,nThreads>>> (f1,f2,r,u,F,streamIndex,voxelType,nu,nVoxels);
	float* temp = f1;
	f1 = f2;
	f2 = temp;
}



// --------------------------------------------------------
// Call to "scsp_active_update_orientation_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_update_orientation(int nBlocks, int nThreads)
{
	scsp_active_update_orientation_D2Q9 
	<<<nBlocks,nThreads>>> (u,p,h,nList,sf,fricR,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_update_orientation_diffusive_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_update_orientation_diffusive(int nBlocks, int nThreads)
{
	scsp_active_update_orientation_diffusive_D2Q9 
	<<<nBlocks,nThreads>>> (p,h,nList,fricR,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_stress_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_stress(int nBlocks, int nThreads)
{
	scsp_active_fluid_stress_D2Q9 
	<<<nBlocks,nThreads>>> (p,h,stress,nList,sf,kapp,activity,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_forces_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_forces(int nBlocks, int nThreads)
{
	scsp_active_fluid_forces_D2Q9 
	<<<nBlocks,nThreads>>> (F,stress,nList,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_molecular_field_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_molecular_field(int nBlocks, int nThreads)
{
	scsp_active_fluid_molecular_field_D2Q9 
	<<<nBlocks,nThreads>>> (h,p,stress,nList,alpha,kapp,nVoxels);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_molecular_field_with_phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_molecular_field_with_phi(int nBlocks, int nThreads)
{
	scsp_active_fluid_molecular_field_with_phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,h,p,stress,nList,alpha,kapp,beta,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_chemical_potential_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_chemical_potential(int nBlocks, int nThreads)
{
	scsp_active_fluid_chemical_potential_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,chempot1,chempot2,chempot3,p,nList,a,alpha,kapphi,beta,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_capillary_force_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_capillary_force(int nBlocks, int nThreads)
{
	scsp_active_fluid_capillary_force_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,chempot1,chempot2,chempot3,F,nList,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_update_phi_3phi_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_update_phi(int nBlocks, int nThreads)
{
	scsp_active_fluid_update_phi_3phi_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,phi2,phi3,chempot1,chempot2,chempot3,u,nList,mob,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_update_phi_diffusive_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_update_phi_diffusive(int nBlocks, int nThreads)
{
	scsp_active_fluid_update_phi_diffusive_D2Q9 
	<<<nBlocks,nThreads>>> (phi1,chempot1,u,nList,mob,nVoxels);
}



// --------------------------------------------------------
// Call to "scsp_active_fluid_set_velocity_field_D2Q9" kernel:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::scsp_active_fluid_set_velocity_field(int nBlocks, int nThreads)
{
	scsp_active_fluid_set_velocity_field_D2Q9 
	<<<nBlocks,nThreads>>> (u,p,0.00005,nVoxels);
}







// --------------------------------------------------------
// Wrtie output:
// --------------------------------------------------------

void class_scsp_active_3phi_D2Q9::write_output(std::string tagname, int step, int iskip, int jskip)
{
	
	// -----------------------------------------------
	//	Define the file location and name:
	// -----------------------------------------------
	
	ofstream outfile;
	std::stringstream filenamecombine;
	filenamecombine << "vtkoutput/" << tagname << "_" << step << ".vtk";
	string filename = filenamecombine.str();
	outfile.open(filename.c_str(), ios::out | ios::app);
	
	// -----------------------------------------------
	//	find output dimensions considering
	//  iskip, jskip:
	// -----------------------------------------------
	
	int Nxs = Nx/iskip;
	int Nys = Ny/jskip;
	if (Nx%2 && iskip>1) Nxs++;  // if odd, then add 1
	if (Ny%2 && jskip>1) Nys++;
	
	// -----------------------------------------------
	//	Write the 'vtk' file header:
	// -----------------------------------------------
	
	string d = "   ";
	outfile << "# vtk DataFile Version 3.1" << endl;
	outfile << "VTK file containing grid data" << endl;
	outfile << "ASCII" << endl;
	outfile << " " << endl;
	outfile << "DATASET STRUCTURED_POINTS" << endl;
	outfile << "DIMENSIONS" << d << Nxs << d << Nys << d << Nz << endl;
	outfile << "ORIGIN " << d << 0 << d << 0 << d << 0 << endl;
	outfile << "SPACING" << d << 1.0*iskip << d << 1.0*jskip << d << 1.0 << endl;
	outfile << " " << endl;
	outfile << "POINT_DATA " << Nxs*Nys*Nz << endl;
	outfile << "SCALARS " << tagname << " float" << endl;
	outfile << "LOOKUP_TABLE default" << endl;
	
	// -----------------------------------------------
	// Write the 'rho' data:
	// NOTE: x-data increases fastest,
	//       then y-data
	// -----------------------------------------------
	
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j+=jskip) {
			for (int i=0; i<Nx; i+=iskip) {
				int ndx = k*Nx*Ny + j*Nx + i;
				outfile << fixed << setprecision(5) << phi2H[ndx] << endl;
			}
		}
	}	
	
	// -----------------------------------------------				
	// Write the 'velocity' data:
	// NOTE: x-data increases fastest,
	//       then y-data	
	// -----------------------------------------------
	
	outfile << "   " << endl;
	outfile << "VECTORS Velocity float" << endl;		
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j+=jskip) {
			for (int i=0; i<Nx; i+=iskip) {
				int ndx = k*Nx*Ny + j*Nx + i;
				outfile << fixed << setprecision(5) << uH[ndx].x << " "
					                                << uH[ndx].y << " " 
													<< 0.0 << endl;
			}
		}
	}
	
	// -----------------------------------------------				
	// Write the 'orientation' data:
	// NOTE: x-data increases fastest,
	//       then y-data	
	// -----------------------------------------------
	
	outfile << "   " << endl;
	outfile << "VECTORS Orientation float" << endl;		
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j+=jskip) {
			for (int i=0; i<Nx; i+=iskip) {
				int ndx = k*Nx*Ny + j*Nx + i;
				outfile << fixed << setprecision(5) << pH[ndx].x << " "
					                                << pH[ndx].y << " " 
													<< 0.0 << endl;
			}
		}
	}
		
	// -----------------------------------------------
	//	Close the file:
	// -----------------------------------------------
	
	outfile.close();
	
}








