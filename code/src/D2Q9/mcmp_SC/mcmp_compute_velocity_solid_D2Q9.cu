#include "hip/hip_runtime.h"

# include "mcmp_compute_velocity_solid_D2Q9.cuh"
# include <stdio.h>

// --------------------------------------------------------
// D2Q9 compute velocity (barycentric) for the system: 
// --------------------------------------------------------

__global__ void mcmp_compute_velocity_solid_D2Q9(float* fA,
                                                 float* fB,
										         float* rA,
										         float* rB,
												 float* rS,
										         float* FxA,
										         float* FxB,
										         float* FyA,
										         float* FyB,
												 float* u,
										         float* v,
												 int* pID,
												 particle2D* p,
										         int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nVoxels) {
		int offst = i*9;			
		float uA = fA[offst+1] + fA[offst+5] + fA[offst+8] - (fA[offst+3] + fA[offst+6] + fA[offst+7]) + 0.5*FxA[i];
		float uB = fB[offst+1] + fB[offst+5] + fB[offst+8] - (fB[offst+3] + fB[offst+6] + fB[offst+7]) + 0.5*FxB[i];
		float vA = fA[offst+2] + fA[offst+5] + fA[offst+6] - (fA[offst+4] + fA[offst+7] + fA[offst+8]) + 0.5*FyA[i];
		float vB = fB[offst+2] + fB[offst+5] + fB[offst+6] - (fB[offst+4] + fB[offst+7] + fB[offst+8]) + 0.5*FyB[i];
		float rTotal = rA[i] + rB[i] + rS[i];		
		float rSVelx = 0.0;
		float rSVely = 0.0;
		int partID = pID[i]; 
		if (partID >= 0) {
			rSVelx = rS[i]*p[partID].vx;
			rSVely = rS[i]*p[partID].vy;
		}
		u[i] = (uA + uB + rSVelx)/rTotal;
		v[i] = (vA + vB + rSVely)/rTotal;				
	}
}