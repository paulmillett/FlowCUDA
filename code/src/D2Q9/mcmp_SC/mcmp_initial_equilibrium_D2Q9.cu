#include "hip/hip_runtime.h"

# include "mcmp_initial_equilibrium_D2Q9.cuh"
# include <stdio.h>

// --------------------------------------------------------
// D2Q9 initialize kernel: 
// --------------------------------------------------------

__global__ void mcmp_initial_equilibrium_D2Q9(float* fA,
                                              float* fB,
										      float* rA,
											  float* rB,
										      float* u,
										      float* v,
										      int nVoxels)
{
	// define current voxel:
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	// initialize populations to equilibrium values:
	if (i < nVoxels) {	
		
		int offst = 9*i;
		const float w0 = 4.0/9.0;
		const float ws = 1.0/9.0;
		const float wd = 1.0/36.0;
		const float omusq = 1.0 - 1.5*(u[i]*u[i] + v[i]*v[i]);
		
		// dir 0
		float feq = w0*omusq;
		fA[offst+0] = feq*rA[i];
		fB[offst+0] = feq*rB[i];
		
		// dir 1
		float evel = u[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+1] = feq*rA[i];
		fB[offst+1] = feq*rB[i];
		
		// dir 2
		evel = v[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+2] = feq*rA[i];
		fB[offst+2] = feq*rB[i];
		
		// dir 3
		evel = -u[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+3] = feq*rA[i];
		fB[offst+3] = feq*rB[i];
		
		// dir 4
		evel = -v[i];
		feq = ws*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+4] = feq*rA[i];
		fB[offst+4] = feq*rB[i];
		
		// dir 5
		evel = u[i] + v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+5] = feq*rA[i];
		fB[offst+5] = feq*rB[i];
		
		// dir 6
		evel = -u[i] + v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+6] = feq*rA[i];
		fB[offst+6] = feq*rB[i];
		
		// dir 7
		evel = -u[i] - v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+7] = feq*rA[i];
		fB[offst+7] = feq*rB[i];
		
		// dir 8
		evel = u[i] - v[i];
		feq = wd*(omusq + 3.0*evel + 4.5*evel*evel);
		fA[offst+8] = feq*rA[i];
		fB[offst+8] = feq*rB[i];
				
	}
}