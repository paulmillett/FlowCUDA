#include "hip/hip_runtime.h"

# include "scsp_3D_capsules_duct_oval_margination.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_3D_capsules_duct_oval_margination::scsp_3D_capsules_duct_oval_margination() : lbm(),ibm(),poissonRBC(),poissonPLT()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = inputParams("Lattice/Nz",1);	
	chA = inputParams("Lattice/chA",float(Ny-1));
	chB = inputParams("Lattice/chB",float(Nz-1));
	chA /= 2.0;
	chB /= 2.0;
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	int sizeIBM = ibm.get_max_array_size();
	int sizeMAX = max(sizeIBM,nVoxels);	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (sizeMAX+(nThreads-1))/nThreads;  // integer division
	
	cout << "largest array size = " << sizeMAX << endl;
	cout << "nBlocks = " << nBlocks << ", nThreads = " << nThreads << endl;
	
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	nStepsEquilibrate = inputParams("Time/nStepsEquilibrate",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	bodyForx = inputParams("LBM/bodyForx",0.0);
	Re = inputParams("LBM/Re",2.0);
	umax = inputParams("LBM/umax",0.1);
	Q0 = inputParams("LBM/Q0",0.0);
	
	// ----------------------------------------------
	// Immersed-Boundary parameters:
	// ----------------------------------------------
		
	a1 = inputParams("IBM/a1",10.0);
	a2 = inputParams("IBM/a2",10.0);
	Ca1 = inputParams("IBM/Ca1",1.0);
	Ca2 = inputParams("IBM/Ca2",1.0);
	float ksmax = inputParams("IBM/ksmax",0.002);
	gam = inputParams("IBM/gamma",0.1);
	ibmFile1 = inputParams("IBM/ibmFile1","rbc.dat");
	ibmFile2 = inputParams("IBM/ibmFile2","sphere.dat");
	ibmUpdate = inputParams("IBM/ibmUpdate","verlet");
	initRandom = inputParams("IBM/initRandom",1);
	nu_in = 5.0/6.0;   // internal RBC visc
	nu_out = 1.0/6.0;  // plasma visc
	
	// ----------------------------------------------
	// IBM set flags for PBC's:
	// ----------------------------------------------
	
	ibm.set_pbcFlag(1,0,0);
		
	// ----------------------------------------------
	// iolets parameters:
	// ----------------------------------------------
	
	numIolets = inputParams("Lattice/numIolets",2);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","polydata");
	iskip = inputParams("Output/iskip",1);
	jskip = inputParams("Output/jskip",1);
	kskip = inputParams("Output/kskip",1);
	nVTKOutputs = inputParams("Output/nVTKOutputs",0);
	precision = inputParams("Output/precision",3);
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	lbm.allocate_forces();
	lbm.allocate_solid();
	ibm.allocate();	
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_3D_capsules_duct_oval_margination::~scsp_3D_capsules_duct_oval_margination()
{
	lbm.deallocate();
	ibm.deallocate();	
	poissonRBC.deallocate();
	poissonPLT.deallocate();
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_3D_capsules_duct_oval_margination::initSystem()
{
		
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// define the solid walls:
	// ----------------------------------------------
	
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j++) {
			for (int i=0; i<Nx; i++) {
				int ndx = k*Nx*Ny + j*Nx + i;
				int Si = 0;				
				// set up solid walls
				float y = float(j) - (float(Ny/2) + 1.0);
				float z = float(k) - (float(Nz/2) + 1.0);
				if (y*y/chA/chA + z*z/chB/chB > 1.0) Si = 1;				
				lbm.setS(ndx,Si);
			}
		}
	}
		
	// ----------------------------------------------
	// create the lattice for channel flow:
	// ----------------------------------------------	
	
	lbm.create_lattice_box_periodic_solid_walls();
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
		
	lbm.stream_index_pull();
			
	// ----------------------------------------------			
	// initialize velocities: 
	// ----------------------------------------------

	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
		lbm.setW(i,0.0);
		lbm.setR(i,1.0);
	}
	
	// ----------------------------------------------			
	// initialize immersed boundary info: 
	// ----------------------------------------------
		
	ibm.read_ibm_information(ibmFile1,ibmFile2);		
	ibm.duplicate_cells();
	ibm.assign_cellIDs_to_nodes();
	ibm.assign_refNode_to_cells();	
	ibm.set_cells_radii_binary();
	ibm.set_cells_types_binary();
		
	// ----------------------------------------------
	// determine membrane parameters (see function
	// below), then calculate reference flux for no
	// capsules:
	// ----------------------------------------------
	
	calcMembraneParams();
			
	// ----------------------------------------------
	// build the binMap array for neighbor lists: 
	// ----------------------------------------------
	
	ibm.build_binMap(nBlocks,nThreads); 
		
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	ibm.memcopy_host_to_device();
		
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);	
	
	// ----------------------------------------------
	// calculate rest geometries for membrane: 
	// ----------------------------------------------
	
	ibm.rest_geometries_skalak(nBlocks,nThreads);
	
	// ----------------------------------------------
	// set the random number seed: 
	// ----------------------------------------------
	
	srand(time(NULL));
	
	// ----------------------------------------------
	// shrink and randomly disperse cells: 
	// ----------------------------------------------
		
	if (initRandom) {		
		
		float a = max(a1,a2);
		ibm.randomize_platelets_and_rbcs(3.0,a+2.0);
		ibm.stepIBM_no_fluid_rbcs_platelets(20000,false,nBlocks,nThreads);  // here, only RBC's move
		ibm.stepIBM_no_fluid(10000,true,nBlocks,nThreads);   // here, both RBC's and PLT's move
			
		/*
		float scale = 0.2;   // 0.7;
		float a = max(a1,a2);
		ibm.shrink_and_randomize_cells(scale,2.0,a+2.0);
		ibm.scale_equilibrium_cell_size(scale,nBlocks,nThreads);
		
		
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Relaxing capsules..." << endl;
		
		int relaxSteps = 0;  // 90000
		scale = 1.0/scale;
		ibm.relax_node_positions_skalak(relaxSteps,scale,0.1,nBlocks,nThreads);	
		ibm.relax_node_positions_skalak(relaxSteps,1.0,0.1,nBlocks,nThreads);
		
		cout << "... done relaxing" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;	
		*/
	}
	
	// ----------------------------------------------
	// initialize poisson solver:
	// ----------------------------------------------
	
	poissonRBC.initialize(Nx,Ny,Nz);
	poissonPLT.initialize(Nx,Ny,Nz);
	poissonRBC.solve_poisson(ibm.faces,ibm.r,ibm.cells,ibm.nFaces,1,nBlocks,nThreads);
	poissonRBC.write_output("indicatorRBC",0,iskip,jskip,kskip,precision);
	poissonPLT.solve_poisson(ibm.faces,ibm.r,ibm.cells,ibm.nFaces,2,nBlocks,nThreads);
	poissonPLT.write_output("indicatorPLT",0,iskip,jskip,kskip,precision);
			
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	ibm.memcopy_device_to_host();
	writeOutput("macros",0);
	
	// ----------------------------------------------
	// set IBM velocities & forces to zero: 
	// ----------------------------------------------
	
	ibm.zero_velocities_forces(nBlocks,nThreads);
	
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_3D_capsules_duct_oval_margination::cycleForward(int stepsPerCycle, int currentCycle)
{
		
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// if simulation just started, perform 
	// equilibration:
	// ----------------------------------------------
	
	if (cummulativeSteps == 0) {
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Equilibrating for " << nStepsEquilibrate << " steps..." << endl;
		for (int i=0; i<nStepsEquilibrate; i++) {
			if (i%10000 == 0) cout << "equilibration step " << i << endl;
			if (i%5 == 0) poissonRBC.solve_poisson(ibm.faces,ibm.r,ibm.cells,ibm.nFaces,1,nBlocks,nThreads);
			ibm.stepIBM(lbm,nBlocks,nThreads);
			lbm.add_body_force(bodyForx,0.0,0.0,nBlocks,nThreads);
			lbm.stream_collide_save_forcing_varvisc(poissonRBC.indicator,nu_in,nu_out,nBlocks,nThreads);
			hipDeviceSynchronize();
		}
		cout << " " << endl;
		cout << "... done equilibrating!" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;
	}
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
		
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;
		if (cummulativeSteps%5 == 0) poissonRBC.solve_poisson(ibm.faces,ibm.r,ibm.cells,ibm.nFaces,1,nBlocks,nThreads);
		ibm.stepIBM(lbm,nBlocks,nThreads);
		lbm.add_body_force(bodyForx,0.0,0.0,nBlocks,nThreads);
		lbm.stream_collide_save_forcing_varvisc(poissonRBC.indicator,nu_in,nu_out,nBlocks,nThreads);
		hipDeviceSynchronize();
	}
	
	cout << cummulativeSteps << endl;	
		
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
	ibm.memcopy_device_to_host();    
	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
		
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_3D_capsules_duct_oval_margination::writeOutput(std::string tagname, int step)
{				
		
	if (step == 0) {
		// only print out vtk files
		lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision); 
		ibm.write_output("ibm",step);
		poissonRBC.volume_fraction_analysis("vol_frac_RBC",0.4);
		poissonPLT.volume_fraction_analysis("vol_frac_PLT",0.4);
	}
	
	if (step > 0) { 
		// analyze membrane geometry:
		ibm.capsule_geometry_analysis(step);
		ibm.output_capsule_data();
		// need to perform PLT poisson solver because it is not performed during
		// regular time steps
		poissonPLT.solve_poisson(ibm.faces,ibm.r,ibm.cells,ibm.nFaces,2,nBlocks,nThreads);
		poissonRBC.volume_fraction_analysis("vol_frac_RBC",0.4);
		poissonPLT.volume_fraction_analysis("vol_frac_PLT",0.4);
	
		// calculate relative viscosity:
		lbm.calculate_relative_viscosity("relative_viscosity_thru_time",Q0,step);
		
		// write vtk output for LBM and IBM:
		int intervalVTK = nSteps/nVTKOutputs;
		if (nVTKOutputs == 0) intervalVTK = nSteps;
		if (step%intervalVTK == 0) {
			lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision);
			ibm.write_output("ibm",step);
			//poissonRBC.write_output("indicatorRBC",step,iskip,jskip,kskip,precision);
			//poissonPLT.write_output("indicatorPLT",step,iskip,jskip,kskip,precision);
		}
		
		// print out final averaged flow profile:
		if (step == nSteps) {
			lbm.print_flow_rate_xdir("flow_data",step);			
		}
	}	
}



// --------------------------------------------------------
// Calculate membrane elastic parameters.  Here, we
// calculate the appropriate values of nu, ks, and bodyForx
// that satisfy the given Re and Ca subject to the 
// conditions that maximum u < umax and ks < ksmax:
// --------------------------------------------------------

void scsp_3D_capsules_duct_oval_margination::calcMembraneParams()
{
	// 'GetPot' object containing input parameters:
	GetPot inputParams("input.dat");
	float Kv = inputParams("IBM/kv",0.0);
	float C = inputParams("IBM/C",10.0);
	int nCells1 = inputParams("IBM/nCells1",1);
	int nCells2 = inputParams("IBM/nCells2",0);
	
	// assumed parameters:
	float rho = 1.0;
	float w = float(Ny)/2.0;
	float h = float(Nz)/2.0;
	float Dh = 4.0*(4.0*w*h)/(4.0*(w+h));
	float infsum = calcInfSum(w,h);	
	
	// per cell calculations:
	umax = 2.0*Re*nu_out/Dh;
	bodyForx = umax*nu_out*M_PI*M_PI*M_PI/(16.0*w*w*infsum);
	for (int i=0; i<nCells1+nCells2; i++) {
		float rad_i = ibm.cellsH[i].rad;
		float Ca_i = 0.1;
		if (i<nCells1)  Ca_i = Ca1;
		if (i>=nCells1) Ca_i = Ca2;
		float Ks = rho*umax*umax*rad_i/(Ca_i*Re);
		float Kb = Ks*rad_i*rad_i*0.00287*sqrt(3);		
		ibm.set_cell_mechanical_props(i,Ks,Kb,Kv,C,Ca_i);
	}
	
	// shear rates:
	float gamma_aver_ydir = umax/w;
	float gamma_wall_ydir = wall_shear_rate("y");
	float gamma_aver_zdir = umax/h;
	float gamma_wall_zdir = wall_shear_rate("z");
	
	// reference flux:
	calcRefFlux();
	
	// output the results:
	cout << "  " << endl;
	cout << "hydraulic diameter = " << Dh << endl;
	cout << "umax (bare fluid) = " << umax << endl;
	cout << "fx = " << bodyForx << endl;
	cout << "aver shear stress in z-dir = " << gamma_aver_zdir << endl;
	cout << "wall shear stress in z-dir = " << gamma_wall_zdir << endl;
	cout << "aver shear stress in y-dir = " << gamma_aver_ydir << endl;
	cout << "wall shear stress in y-dir = " << gamma_wall_ydir << endl;
	cout << "  " << endl;
	
}



// --------------------------------------------------------
// Calculate infinite sum associated with solution
// to velocity profile in rectanglular channel:
// --------------------------------------------------------

float scsp_3D_capsules_duct_oval_margination::calcInfSum(float w, float h)
{
	float outval = 0.0;
	// take first 40 terms of infinite sum
	for (int n = 1; n<80; n=n+2) {
		float nf = float(n);
		float pref = pow(-1.0,(nf-1.0)/2)/(nf*nf*nf);
		float term = pref*(1 - 1/cosh(nf*M_PI*h/2.0/w));
		outval += term;
	}
	return outval;
}



// --------------------------------------------------------
// Calculate reference flux for the chosen values of w, h,
// bodyForx, and nu:
// --------------------------------------------------------

void scsp_3D_capsules_duct_oval_margination::calcRefFlux()
{
	// parameters:
	float w = float(Ny)/2.0;
	float h = float(Nz)/2.0;
	Q0 = 0.0;
	
	// calculate solution for velocity at every
	// site in the y-z plane:
	for (int j=0; j<Ny; j++) {
		for (int k=0; k<Nz; k++) {
			float y = float(j) - w;
			float z = float(k) - h;
			float u0 = velocity_at_point(y,z,w,h);
			Q0 += u0;
		}
	}
	
	// output the results:
	cout << "reference flux = " << Q0 << endl;
	cout << "  " << endl;		
}



// --------------------------------------------------------
// Calculate wall shear rate:
// --------------------------------------------------------

float scsp_3D_capsules_duct_oval_margination::wall_shear_rate(std::string dir)
{
	// parameters:
	float w = float(Ny)/2.0;
	float h = float(Nz)/2.0;	
	// calculate solution for velocity at two points
	// and determine forward finite difference for shear rate
	if (dir == "z") {
		float u0 = velocity_at_point(0.0,h,w,h);
		float u1 = velocity_at_point(0.0,h-1.0,w,h);
		return (u1-u0)/1.0;
	} else if (dir == "y") {
		float u0 = velocity_at_point(w,0.0,w,h);
		float u1 = velocity_at_point(w-1,0.0,w,h);
		return (u1-u0)/1.0;
	} else {
		return 0.0;
	}
}



// --------------------------------------------------------
// Calculate velocity at point:
// --------------------------------------------------------

float scsp_3D_capsules_duct_oval_margination::velocity_at_point(float y, float z, float w, float h)
{
	float sumval = 0.0;
	// take first 40 terms of infinite sum
	for (int n = 1; n<80; n=n+2) {
		float nf = float(n);
		float pref = pow(-1.0,(nf-1.0)/2)/(nf*nf*nf);
		float term = pref*(1 - cosh(nf*M_PI*z/2/w) / cosh(nf*M_PI*h/2/w)) * cos(nf*M_PI*y/2/w);
		sumval += term;
	}
	return (16*bodyForx*w*w/nu/pow(M_PI,3))*sumval;
}

