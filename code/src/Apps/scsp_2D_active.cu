#include "hip/hip_runtime.h"

# include "scsp_2D_active.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_2D_active::scsp_2D_active() : lbm()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	
	// ----------------------------------------------
	// Output parameters:
	// ----------------------------------------------
	
	iskip = inputParams("Output/iskip",1);
	jskip = inputParams("Output/jskip",1);
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_2D_active::~scsp_2D_active()
{	
	lbm.deallocate();
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_2D_active::initSystem()
{
		
	// ----------------------------------------------
	// create the lattice using "box" function. 
	// ----------------------------------------------	
		
	lbm.create_lattice_box_periodic();
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
	
	lbm.stream_index_pull();
		
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
		
	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
		lbm.setR(i,1.0);	
	}
	
	// ----------------------------------------------			
	// initialize orientation: 
	// ----------------------------------------------
		
	for (int i=0; i<nVoxels; i++) {
		float theta = 2.0*M_PI*((float)rand()/RAND_MAX - 0.5); 
		float px = 1.0;
		float py = 0.0;
		float pxr = px*cos(theta) - py*sin(theta);
		float pyr = px*sin(theta) + py*cos(theta);
		lbm.setPx(i,pxr);
		lbm.setPy(i,pyr);	
	}
		
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);
	
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_2D_active::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;	
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
	
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;		
		lbm.zero_forces(nBlocks,nThreads);
		lbm.scsp_active_fluid_molecular_field(nBlocks,nThreads);
		lbm.scsp_active_fluid_stress(nBlocks,nThreads);
		lbm.scsp_active_fluid_forces(nBlocks,nThreads);
		lbm.scsp_active_update_orientation(nBlocks,nThreads);		
		lbm.stream_collide_save_forcing(nBlocks,nThreads);
		hipDeviceSynchronize();
	}
	
	cout << cummulativeSteps << endl;	
	
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
		
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
	
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_2D_active::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// function location:
	// "io/write_vtk_output.cuh"
	// ----------------------------------------------
	
	lbm.write_output(tagname,step,iskip,jskip);
			
}







