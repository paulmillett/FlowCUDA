#include "hip/hip_runtime.h"

# include "mcmp_2D_capbridge_dip.cuh"
# include "../IO/GetPot"
# include <math.h>
# include <string> 
using namespace std;   



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

mcmp_2D_capbridge_dip::mcmp_2D_capbridge_dip() : lbm()
{	
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",9);	
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
		
	// ----------------------------------------------
	// Particles parameters:
	// ----------------------------------------------
	
	nParts = inputParams("Particles/nParts",1);	
	rApart = inputParams("Particles/rApart",0.5);
	rBpart = inputParams("Particles/rBpart",0.5);			
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","structured");
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

mcmp_2D_capbridge_dip::~mcmp_2D_capbridge_dip()
{
	lbm.deallocate();
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void mcmp_2D_capbridge_dip::initSystem()
{
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the periodic lattice:
	// ----------------------------------------------	
	
	lbm.create_lattice_box_periodic();	
			
	// ----------------------------------------------			
	// initialize particles: 
	// ----------------------------------------------
	
	lbm.setPrx(0,420.0);
	lbm.setPry(0,250.0);
	lbm.setPrInner(0,40.0);
	lbm.setPrOuter(0,45.0);
	
	lbm.setPrx(1,580.0);
	lbm.setPry(1,250.0);
	lbm.setPrInner(1,70.0);
	lbm.setPrOuter(1,75.0);
			
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
	
	// initialize solid field:
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {		
			int ndx = j*Nx + i;	
			lbm.setX(ndx,i);
			lbm.setY(ndx,j);
			float Bi = 0.0;
			for (int k=0; k<nParts; k++) {
				float dx = float(i) - lbm.getPrx(k);
				float dy = float(j) - lbm.getPry(k);
				float rr = sqrt(dx*dx + dy*dy);				
				if (rr <= lbm.getPrOuter(k)) {
					if (rr < lbm.getPrInner(k)) {
						Bi = 1.0;
					}
					else {
						float rsc = rr - lbm.getPrInner(k);
						Bi = 1.0 - rsc/(lbm.getPrOuter(k) - lbm.getPrInner(k));
					}
				}	
			}			
			if (i > 400 && i < 600 && j > 220 && j < 280) {
				lbm.setRA(ndx,1.0*(1.0-Bi) + rApart*Bi);
				lbm.setRB(ndx,0.02*(1.0-Bi) + rBpart*Bi);				
			}
			else {
				lbm.setRA(ndx,0.02*(1.0-Bi) + rApart*Bi);
				lbm.setRB(ndx,1.0*(1.0-Bi) + rBpart*Bi);
				
			}		 
		}
	}
			
	// initialize velocity fields
	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
	}	
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
	
	lbm.stream_index_push();	
			
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------	
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium_dip(nBlocks,nThreads);
		
}



// --------------------------------------------------------
// Step forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void mcmp_2D_capbridge_dip::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------

	for (int step=0; step<stepsPerCycle; step++) {
		
		cummulativeSteps++;
		
		// ------------------------------
		// zero particle forces:
		// ------------------------------
		
		lbm.zero_particle_forces_dip(nBlocks,nThreads);
		
		// ------------------------------
		// update density fields:
		// ------------------------------
		
		lbm.map_particles_to_lattice_dip(nBlocks,nThreads);
		lbm.compute_density_dip(nBlocks,nThreads);
		hipDeviceSynchronize();
		
		// ------------------------------
		// update fluid fields:											   
		// ------------------------------ 
		
		lbm.compute_SC_forces_dip(nBlocks,nThreads);
		lbm.compute_velocity_dip(nBlocks,nThreads);
		lbm.collide_stream_dip(nBlocks,nThreads);  		
		lbm.swap_populations();	
		
		// ------------------------------
		// update particles:											   
		// ------------------------------ 
		
		lbm.fix_particle_velocity_dip(0.005,nBlocks,nThreads);
		lbm.move_particles_dip(nBlocks,nThreads);
		hipDeviceSynchronize();
				
	}
	
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
		
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);	
	
}



// --------------------------------------------------------
// Write output:
// --------------------------------------------------------

void mcmp_2D_capbridge_dip::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// function location:
	// "io/write_vtk_output.cuh"
	// ----------------------------------------------
	
	lbm.write_output(tagname,step); 

}










