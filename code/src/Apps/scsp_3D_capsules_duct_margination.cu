#include "hip/hip_runtime.h"

# include "scsp_3D_capsules_duct_margination.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_3D_capsules_duct_margination::scsp_3D_capsules_duct_margination() : lbm(),ibm()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = inputParams("Lattice/Nz",1);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	//nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	nBlocks = (1230720+(nThreads-1))/nThreads;  // integer division
	
	cout << "nBlocks = " << nBlocks << ", nThreads = " << nThreads << endl;
	
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	nStepsEquilibrate = inputParams("Time/nStepsEquilibrate",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	bodyForx = inputParams("LBM/bodyForx",0.0);
	Re = inputParams("LBM/Re",2.0);
	umax = inputParams("LBM/umax",0.1);
	Q0 = inputParams("LBM/Q0",0.0);
	
	// ----------------------------------------------
	// Immersed-Boundary parameters:
	// ----------------------------------------------
		
	a1 = inputParams("IBM/a1",10.0);
	a2 = inputParams("IBM/a2",10.0);
	Ca1 = inputParams("IBM/Ca1",1.0);
	Ca2 = inputParams("IBM/Ca2",1.0);
	float ksmax = inputParams("IBM/ksmax",0.002);
	gam = inputParams("IBM/gamma",0.1);
	ibmFile1 = inputParams("IBM/ibmFile1","rbc.dat");
	ibmFile2 = inputParams("IBM/ibmFile2","sphere.dat");
	ibmUpdate = inputParams("IBM/ibmUpdate","verlet");
	initRandom = inputParams("IBM/initRandom",1);
	
	// ----------------------------------------------
	// IBM set flags for PBC's:
	// ----------------------------------------------
	
	ibm.set_pbcFlag(1,0,0);
		
	// ----------------------------------------------
	// iolets parameters:
	// ----------------------------------------------
	
	numIolets = inputParams("Lattice/numIolets",2);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","polydata");
	iskip = inputParams("Output/iskip",1);
	jskip = inputParams("Output/jskip",1);
	kskip = inputParams("Output/kskip",1);
	nVTKOutputs = inputParams("Output/nVTKOutputs",0);
	precision = inputParams("Output/precision",3);
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	lbm.allocate_forces();
	ibm.allocate();	
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_3D_capsules_duct_margination::~scsp_3D_capsules_duct_margination()
{
	lbm.deallocate();
	ibm.deallocate();	
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_3D_capsules_duct_margination::initSystem()
{
		
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
		
	// ----------------------------------------------
	// create the lattice for channel flow:
	// ----------------------------------------------	
	
	lbm.create_lattice_box_channel();
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
		
	lbm.stream_index_pull();
			
	// ----------------------------------------------			
	// initialize velocities: 
	// ----------------------------------------------

	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
		lbm.setW(i,0.0);
		lbm.setR(i,1.0);
	}
	
	// ----------------------------------------------			
	// initialize immersed boundary info: 
	// ----------------------------------------------
		
	ibm.read_ibm_information(ibmFile1,ibmFile2);		
	ibm.duplicate_cells();
	ibm.assign_cellIDs_to_nodes();
	ibm.assign_refNode_to_cells();	
	ibm.set_cells_radii_binary();
	ibm.set_cells_types_binary();
		
	// ----------------------------------------------
	// determine membrane parameters (see function
	// below), then calculate reference flux for no
	// capsules:
	// ----------------------------------------------
	
	calcMembraneParams();
			
	// ----------------------------------------------
	// build the binMap array for neighbor lists: 
	// ----------------------------------------------
	
	ibm.build_binMap(nBlocks,nThreads); 
		
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	ibm.memcopy_host_to_device();
		
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);	
	
	// ----------------------------------------------
	// calculate rest geometries for membrane: 
	// ----------------------------------------------
	
	ibm.rest_geometries_skalak(nBlocks,nThreads);
	
	// ----------------------------------------------
	// set the random number seed: 
	// ----------------------------------------------
	
	srand(time(NULL));
	
	// ----------------------------------------------
	// shrink and randomly disperse cells: 
	// ----------------------------------------------
		
	if (initRandom) {
		
		
		float a = max(a1,a2);
		ibm.randomize_cells(a+2.0);
		//ibm.stepIBM_no_fluid(20000,true,nBlocks,nThreads); 
		
		
		/*
		float scale = 0.2;   // 0.7;
		float a = max(a1,a2);
		ibm.shrink_and_randomize_cells(scale,2.0,a+2.0);
		ibm.scale_equilibrium_cell_size(scale,nBlocks,nThreads);
		
		
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Relaxing capsules..." << endl;
		
		int relaxSteps = 0;  // 90000
		scale = 1.0/scale;
		ibm.relax_node_positions_skalak(relaxSteps,scale,0.1,nBlocks,nThreads);	
		ibm.relax_node_positions_skalak(relaxSteps,1.0,0.1,nBlocks,nThreads);
		
		cout << "... done relaxing" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;	
		*/
	}
			
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	ibm.memcopy_device_to_host();
	writeOutput("macros",0);
	
	// ----------------------------------------------
	// set IBM velocities & forces to zero: 
	// ----------------------------------------------
	
	ibm.zero_velocities_forces(nBlocks,nThreads);
	
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_3D_capsules_duct_margination::cycleForward(int stepsPerCycle, int currentCycle)
{
		
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// if simulation just started, perform 
	// equilibration:
	// ----------------------------------------------
	
	if (cummulativeSteps == 0) {
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Equilibrating for " << nStepsEquilibrate << " steps..." << endl;
		for (int i=0; i<nStepsEquilibrate; i++) {
			if (i%10000 == 0) cout << "equilibration step " << i << endl;
			
			ibm.stepIBM_no_fluid(1,false,nBlocks,nThreads); 
			
			/*
			ibm.stepIBM(lbm,nBlocks,nThreads);
			lbm.add_body_force(bodyForx,0.0,0.0,nBlocks,nThreads);
			lbm.stream_collide_save_forcing(nBlocks,nThreads);
			*/	
			hipDeviceSynchronize();
			
		}
		cout << " " << endl;
		cout << "... done equilibrating!" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;
	}
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
		
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;
		
		ibm.stepIBM_no_fluid(1,false,nBlocks,nThreads);
		
		/*
		ibm.stepIBM(lbm,nBlocks,nThreads);
		lbm.add_body_force(bodyForx,0.0,0.0,nBlocks,nThreads);
		lbm.stream_collide_save_forcing(nBlocks,nThreads);
		*/
		hipDeviceSynchronize();
	}
	
	cout << cummulativeSteps << endl;	
		
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
	ibm.memcopy_device_to_host();    
	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
		
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_3D_capsules_duct_margination::writeOutput(std::string tagname, int step)
{				
		
	if (step == 0) {
		// only print out vtk files
		lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision); 
		ibm.write_output("ibm",step);
	}
	
	if (step > 0) { 
		// analyze membrane geometry:
		ibm.capsule_geometry_analysis(step);
		ibm.output_capsule_data();
	
		// calculate relative viscosity:
		lbm.calculate_relative_viscosity("relative_viscosity_thru_time",Q0,step);
		
		// write vtk output for LBM and IBM:
		int intervalVTK = nSteps/nVTKOutputs;
		if (nVTKOutputs == 0) intervalVTK = nSteps;
		if (step%intervalVTK == 0) {
			lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision);
			ibm.write_output("ibm",step);
		}
		
		// print out final averaged flow profile:
		if (step == nSteps) {
			lbm.print_flow_rate_xdir("flow_data",step);			
		}
	}	
}



// --------------------------------------------------------
// Calculate membrane elastic parameters.  Here, we
// calculate the appropriate values of nu, ks, and bodyForx
// that satisfy the given Re and Ca subject to the 
// conditions that maximum u < umax and ks < ksmax:
// --------------------------------------------------------

void scsp_3D_capsules_duct_margination::calcMembraneParams()
{
	// 'GetPot' object containing input parameters:
	GetPot inputParams("input.dat");
	float Kv = inputParams("IBM/kv",0.0);
	float C = inputParams("IBM/C",2.0);
	int nCells1 = inputParams("IBM/nCells1",1);
	int nCells2 = inputParams("IBM/nCells2",0);
	
	// assumed parameters:
	float rho = 1.0;
	float w = float(Ny)/2.0;
	float h = float(Nz)/2.0;
	float Dh = 4.0*(4.0*w*h)/(4.0*(w+h));
	float infsum = calcInfSum(w,h);	
	
	// my calculations:
	umax = 2.0*Re*nu/Dh;
	bodyForx = umax*nu*M_PI*M_PI*M_PI/(16.0*w*w*infsum);
	for (int i=0; i<nCells1+nCells2; i++) {
		float rad_i = ibm.cellsH[i].rad;
		float Ca_i = 0.1;
		if (i<nCells1)  Ca_i = Ca1;
		if (i>=nCells1) Ca_i = Ca2;
		float Ks = rho*umax*umax*rad_i/(Ca_i*Re);
		float Kb = Ks*rad_i*rad_i*0.00287*sqrt(3);		
		ibm.set_cell_mechanical_props(i,Ks,Kb,Kv,C,Ca_i);
	}	
}



// --------------------------------------------------------
// Calculate infinite sum associated with solution
// to velocity profile in rectanglular channel:
// --------------------------------------------------------

float scsp_3D_capsules_duct_margination::calcInfSum(float w, float h)
{
	float outval = 0.0;
	// take first 40 terms of infinite sum
	for (int n = 1; n<80; n=n+2) {
		float nf = float(n);
		float pref = pow(-1.0,(nf-1.0)/2)/(nf*nf*nf);
		float term = pref*(1 - 1/cosh(nf*M_PI*h/2.0/w));
		outval += term;
	}
	return outval;
}

