
# include "scsp_3D_iolets.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_3D_iolets::scsp_3D_iolets() : lbm()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
		
	// ----------------------------------------------
	// iolets parameters:
	// ----------------------------------------------
	
	numIolets = inputParams("Lattice/numIolets",2);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","polydata");
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	lbm.allocate_voxel_positions();
		
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_3D_iolets::~scsp_3D_iolets()
{	
	lbm.deallocate();	
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_3D_iolets::initSystem()
{
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the lattice using "box" function.
	// ----------------------------------------------	
	
	if (latticeSource == "box") {
		lbm.create_lattice_box();
	}	
	
	// ----------------------------------------------
	// create the lattice by reading from file.	
	// input integer = 1 = read x[],y[],z[],voxelType[],nList[]
	//               = 2 = read x[],y[],z[],voxelType[]
	//               = 3 = read x[],y[],z[] 
	// ----------------------------------------------	
	
	if (latticeSource == "file") {
		lbm.read_lattice_geometry(1);
	}		
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
		
	lbm.stream_index_pull();
	
	// ----------------------------------------------			
	// initialize inlets/outlets: 
	// ----------------------------------------------
	
	lbm.read_iolet_info(0,"Iolet1");
	lbm.read_iolet_info(1,"Iolet2");
			
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
	
	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
		lbm.setW(i,0.0);
		lbm.setR(i,1.0);		
	}
		
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
		
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);		

}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_3D_iolets::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	bool save = false;
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
	
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;		
		if (step == (stepsPerCycle-1)) save = true;		
		lbm.stream_collide_save(nBlocks,nThreads,save);
		hipDeviceSynchronize();
	}
	    	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
	writeOutput("macros",cummulativeSteps);
	
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_3D_iolets::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// ----------------------------------------------
	
	if (vtkFormat == "structured") {
		int precision = 3;
		lbm.vtk_structured_output_ruvw(tagname,step,1,1,1,precision);
	}
	
	else if (vtkFormat == "polydata") {
		lbm.vtk_polydata_output_ruvw(tagname,step);
	}
	
}







