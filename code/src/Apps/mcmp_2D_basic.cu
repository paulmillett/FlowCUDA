#include "hip/hip_runtime.h"

# include "mcmp_2D_basic.cuh"
# include "../D2Q9/mcmp_SC/kernels_mcmp_SC_D2Q9.cuh"
# include "../D2Q9/init/stream_index_builder_D2Q9.cuh"
# include "../D2Q9/init/lattice_builders_D2Q9.cuh"
# include "../IO/GetPot"
# include "../IO/write_vtk_output.cuh"
# include <math.h>
# include <string> 
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

mcmp_2D_basic::mcmp_2D_basic() 
{	
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",9);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	gAB = inputParams("LBM/gAB",6.0);
	potType = inputParams("LBM/potType",1);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","structured");
	
	// ----------------------------------------------
	// allocate array memory (host):
	// ----------------------------------------------
	
    uH = (float*)malloc(nVoxels*sizeof(float));
    vH = (float*)malloc(nVoxels*sizeof(float));
    rAH = (float*)malloc(nVoxels*sizeof(float));
	rBH = (float*)malloc(nVoxels*sizeof(float));
	prH = (float*)malloc(nVoxels*sizeof(float));
	nListH = (int*)malloc(nVoxels*Q*sizeof(int));	
	voxelTypeH = (int*)malloc(nVoxels*sizeof(int));
	streamIndexH = (int*)malloc(nVoxels*Q*sizeof(int));
	
	// ----------------------------------------------
	// allocate array memory (device):
	// ----------------------------------------------
	
	hipMalloc((void **) &u, nVoxels*sizeof(float));
	hipMalloc((void **) &v, nVoxels*sizeof(float));
	hipMalloc((void **) &rA, nVoxels*sizeof(float));
	hipMalloc((void **) &rB, nVoxels*sizeof(float));
	hipMalloc((void **) &pr, nVoxels*sizeof(float));
	hipMalloc((void **) &f1A, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f1B, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2A, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2B, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &FxA, nVoxels*sizeof(float));
	hipMalloc((void **) &FxB, nVoxels*sizeof(float));
	hipMalloc((void **) &FyA, nVoxels*sizeof(float));
	hipMalloc((void **) &FyB, nVoxels*sizeof(float));	
	hipMalloc((void **) &nList, nVoxels*Q*sizeof(int));
	hipMalloc((void **) &voxelType, nVoxels*sizeof(int));
	hipMalloc((void **) &streamIndex, nVoxels*Q*sizeof(int));
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

mcmp_2D_basic::~mcmp_2D_basic()
{
	
	// ----------------------------------------------
	// free array memory (host):
	// ----------------------------------------------
	
	free(uH);
	free(vH);
	free(rAH);
	free(rBH);
	free(nListH);
	free(voxelTypeH);
	free(streamIndexH);
	
	// ----------------------------------------------
	// free array memory (device):
	// ----------------------------------------------
	
	hipFree(u);
	hipFree(v);
	hipFree(rA);
	hipFree(rB);
	hipFree(f1A);
	hipFree(f2A);
	hipFree(f1B);
	hipFree(f2B);
	hipFree(FxA);
	hipFree(FxB);
	hipFree(FyA);
	hipFree(FyB);
	hipFree(nList);
	hipFree(voxelType);
	hipFree(streamIndex);
	
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void mcmp_2D_basic::initSystem()
{
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the lattice using "box" function.
	// function location:
	// "lattice/lattice_builders_D2Q9.cuh"	 
	// ----------------------------------------------	
	
	if (latticeSource == "box") {
		Nx = inputParams("Lattice/Nx",0);
		Ny = inputParams("Lattice/Ny",0);
		Nz = inputParams("Lattice/Nz",0);
		build_box_lattice_D2Q9(nVoxels,Nx,Ny,voxelTypeH,nListH);
	}	
		
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// function location:
	// "D2Q9/stream_index_builder_D2Q9.cuh"
	// ----------------------------------------------
		
	stream_index_push_D2Q9(nVoxels,nListH,streamIndexH);
	
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
	
	std::string initType = inputParams("LBM/initType","random");
	
	if (initType == "random") {
		srand(time(NULL));   // set the random seed
		for (int i=0; i<nVoxels; i++) {
			uH[i] = 0.0;
			vH[i] = 0.0;
			float ranA = (float)rand()/RAND_MAX;
			float ranB = (float)rand()/RAND_MAX;
			rAH[i] = 0.5 + 0.1*(ranA-0.5);
			rBH[i] = 0.5 + 0.1*(ranB-0.5);
		}
	}
	
	if (initType == "droplet") {
		float rInner = inputParams("LBM/rInner",10.0);	
		float rOuter = inputParams("LBM/rOuter",15.0);	
		for (int j=0; j<Ny; j++) {
			for (int i=0; i<Nx; i++) {
				int ndx = j*Nx + i;
				rAH[ndx] = 0.0;
				float dx = float(i) - float(Nx/2);
				float dy = float(j) - float(Ny/2);		
				float r2 = dx*dx + dy*dy;
				float r = sqrt(r2);
				if (r <= rOuter) {
					if (r < rInner) {
						rAH[ndx] = 1.0;
					}
					else {
						float rsc = r - rInner;
						rAH[ndx] = 1.0*exp(-rsc*rsc/5.0);
					}
				}
				rBH[ndx] = 1.0 - rAH[ndx];				
			}
		}
	}	
	
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------	
	// copy arrays from host to device: 
	// ----------------------------------------------
	
    hipMemcpy(u, uH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(v, vH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rA, rAH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rB, rBH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(nList, nListH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(voxelType, voxelTypeH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(streamIndex, streamIndexH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	mcmp_initial_equilibrium_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,u,v,nVoxels);	
		
}



// --------------------------------------------------------
// Step forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void mcmp_2D_basic::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------

	for (int step=0; step<stepsPerCycle; step++) {
		
		cummulativeSteps++;
		
		mcmp_compute_density_D2Q9 
		<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,nVoxels);
				
		hipDeviceSynchronize();
		
		if (potType == 1) {
			mcmp_compute_SC_forces_1_D2Q9 
			<<<nBlocks,nThreads>>> (rA,rB,FxA,FxB,FyA,FyB,nList,gAB,nVoxels);	
		}
		else if (potType == 2) {
			mcmp_compute_SC_forces_2_D2Q9 
			<<<nBlocks,nThreads>>> (rA,rB,FxA,FxB,FyA,FyB,nList,gAB,nVoxels);
		}													  
		
		mcmp_compute_velocity_D2Q9 
		<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,FxA,FxB,FyA,FyB,u,v,nVoxels);
		
		mcmp_collide_stream_D2Q9 
		<<<nBlocks,nThreads>>> (f1A,f1B,f2A,f2B,rA,rB,u,v,FxA,FxB,FyA,FyB,streamIndex,nu,nVoxels);
														 
		float* tempA = f1A;
		float* tempB = f1B;
		f1A = f2A;
		f1B = f2B;
		f2A = tempA;
		f2B = tempB;
		
		hipDeviceSynchronize();
		
	}
	
	// ----------------------------------------------
	// compute EOS pressure:
	// ----------------------------------------------
	
	mcmp_compute_SC_pressure_D2Q9 
	<<<nBlocks,nThreads>>> (rA,rB,pr,gAB,nVoxels);
	
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
    hipMemcpy(uH, u, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(vH, v, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rAH, rA, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(prH, pr, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);	
	
}



// --------------------------------------------------------
// Write output:
// --------------------------------------------------------

void mcmp_2D_basic::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// function location:
	// "io/write_vtk_output.cuh"
	// ----------------------------------------------
	
	if (vtkFormat == "structured") {
		write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,prH,uH,vH);
		//write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,rAH,rBH,uH,vH);
	}
	
}










