#include "hip/hip_runtime.h"

# include "mcmp_2D_dropsurface.cuh"
# include "../D2Q9/mcmp_SC/kernels_mcmp_SC_D2Q9.cuh"
# include "../D2Q9/mcmp_SC/kernels_mcmp_SC_solid_D2Q9.cuh"
# include "../D2Q9/init/stream_index_builder_D2Q9.cuh"
# include "../D2Q9/particles/map_particles_to_grid_D2Q9.cuh"
# include "../D2Q9/init/lattice_builders_D2Q9.cuh"
# include "../IO/GetPot"
# include "../IO/write_vtk_output.cuh"
# include <math.h>
# include <string> 
using namespace std;   



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

mcmp_2D_dropsurface::mcmp_2D_dropsurface() 
{	
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",9);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	gAB = inputParams("LBM/gAB",6.0);
	gAS = inputParams("LBM/gAS",6.0);
	gBS = inputParams("LBM/gBS",6.0); 
	potType = inputParams("LBM/potType",1);
	nParticles = inputParams("LBM/nParticles",1);
	rAinS = inputParams("LBM/rAinS",0.5);
	rBinS = inputParams("LBM/rBinS",0.5);
			
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","structured");
	
	// ----------------------------------------------
	// allocate array memory (host):
	// ----------------------------------------------
	
    uH = (float*)malloc(nVoxels*sizeof(float));
    vH = (float*)malloc(nVoxels*sizeof(float));
    rAH = (float*)malloc(nVoxels*sizeof(float));
	rBH = (float*)malloc(nVoxels*sizeof(float));
	rSH = (float*)malloc(nVoxels*sizeof(float));
	xH = (int*)malloc(nVoxels*sizeof(int));
	yH = (int*)malloc(nVoxels*sizeof(int));
	nListH = (int*)malloc(nVoxels*Q*sizeof(int));	
	voxelTypeH = (int*)malloc(nVoxels*sizeof(int));
	streamIndexH = (int*)malloc(nVoxels*Q*sizeof(int));
	pH = (particle2D*)malloc(nParticles*sizeof(particle2D));
	pIDH = (int*)malloc(nVoxels*sizeof(int));
	
	// ----------------------------------------------
	// allocate array memory (device):
	// ----------------------------------------------
	
	hipMalloc((void **) &u, nVoxels*sizeof(float));
	hipMalloc((void **) &v, nVoxels*sizeof(float));
	hipMalloc((void **) &rA, nVoxels*sizeof(float));
	hipMalloc((void **) &rB, nVoxels*sizeof(float));
	hipMalloc((void **) &rS, nVoxels*sizeof(float));
	hipMalloc((void **) &f1A, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f1B, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2A, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2B, nVoxels*Q*sizeof(float));	
	hipMalloc((void **) &FxA, nVoxels*sizeof(float));
	hipMalloc((void **) &FxB, nVoxels*sizeof(float));
	hipMalloc((void **) &FyA, nVoxels*sizeof(float));
	hipMalloc((void **) &FyB, nVoxels*sizeof(float));	
	hipMalloc((void **) &x, nVoxels*sizeof(int));
	hipMalloc((void **) &y, nVoxels*sizeof(int));
	hipMalloc((void **) &pID, nVoxels*sizeof(int));
	hipMalloc((void **) &nList, nVoxels*Q*sizeof(int));
	hipMalloc((void **) &voxelType, nVoxels*sizeof(int));
	hipMalloc((void **) &streamIndex, nVoxels*Q*sizeof(int));
	hipMalloc((void **) &p, nParticles*sizeof(particle2D));
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

mcmp_2D_dropsurface::~mcmp_2D_dropsurface()
{
	
	// ----------------------------------------------
	// free array memory (host):
	// ----------------------------------------------
	
	free(uH);
	free(vH);
	free(rAH);
	free(rBH);
	free(rSH);
	free(xH);
	free(yH);
	free(nListH);
	free(voxelTypeH);
	free(streamIndexH);
	free(pH);
	free(pIDH);
	
	// ----------------------------------------------
	// free array memory (device):
	// ----------------------------------------------
	
	hipFree(u);
	hipFree(v);
	hipFree(rA);
	hipFree(rB);
	hipFree(rS);
	hipFree(f1A);
	hipFree(f2A);
	hipFree(f1B);
	hipFree(f2B);
	hipFree(FxA);
	hipFree(FxB);
	hipFree(FyA);
	hipFree(FyB);
	hipFree(x);
	hipFree(y);
	hipFree(pID);
	hipFree(nList);
	hipFree(voxelType);
	hipFree(streamIndex);
	hipFree(p);
	
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void mcmp_2D_dropsurface::initSystem()
{
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the lattice using "box" function.
	// function location:
	// "lattice/lattice_builders_D2Q9.cuh"	 
	// ----------------------------------------------	
	
	if (latticeSource == "box") {
		Nx = inputParams("Lattice/Nx",0);
		Ny = inputParams("Lattice/Ny",0);
		Nz = inputParams("Lattice/Nz",0);
		build_box_lattice_D2Q9(nVoxels,Nx,Ny,voxelTypeH,nListH);
		for (int j=0; j<Ny; j++) {
			for (int i=0; i<Nx; i++) {
				int ndx = j*Nx + i;
				xH[ndx] = i;
				yH[ndx] = j;
			}
		}
	}	
		
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// function location:
	// "D2Q9/stream_index_builder_D2Q9.cuh"
	// ----------------------------------------------
		
	stream_index_push_D2Q9(nVoxels,nListH,streamIndexH);
		
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
			
	// initialize solid field:
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {		
			int ndx = j*Nx + i;	
			rSH[ndx] = 0.0;	
			pIDH[ndx] = -1;		
			float dy = abs(float(j) - 20.0);
			if (dy <= 15.0) {
				if (dy < 12.0) {
					rSH[ndx] = 1.0;
				}
				else {
					float rsc = dy - 12.0;
					rSH[ndx] = exp(-rsc*rsc/3.0);
				}
				pIDH[ndx] = 0;
			}
		}
	}
		
	// initialize density fields: 
	float rInner = inputParams("LBM/rInner",10.0);	
	float rOuter = inputParams("LBM/rOuter",15.0);
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {
			int ndx = j*Nx + i;
			float rhoA = 0.02;
			float rhoB = 0.0;			
			float dx = float(i) - float(Nx/2);
			float dy = float(j) - 40.0;		
			float r2 = dx*dx + dy*dy;
			float r = sqrt(r2);
			if (r <= rOuter && j > 20.0) {
				if (r < rInner) {
					rhoA = 1.0;
				}
				else {
					float rsc = r - rInner;
					rhoA = 1.0*exp(-rsc*rsc/5.0);
				}
			}
			rhoB = 0.02 + 1.0 - rhoA;			
			rAH[ndx] = rhoA*(1.0-rSH[ndx]) + rAinS*(rSH[ndx]);
			rBH[ndx] = rhoB*(1.0-rSH[ndx]) + rBinS*(rSH[ndx]);			
		}
	}
			
	// initialize velocity fields
	for (int i=0; i<nVoxels; i++) {
		uH[i] = 0.0;
		vH[i] = 0.0;	
	}	
			
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------	
	// copy arrays from host to device: 
	// ----------------------------------------------
	
    hipMemcpy(u, uH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(v, vH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rA, rAH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rB, rBH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(rS, rSH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(x, xH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(y, yH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(nList, nListH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(voxelType, voxelTypeH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(streamIndex, streamIndexH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(p, pH, sizeof(particle2D)*nParticles, hipMemcpyHostToDevice);
	hipMemcpy(pID, pIDH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	mcmp_initial_equilibrium_D2Q9 
	<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,u,v,nVoxels);	
		
}



// --------------------------------------------------------
// Step forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void mcmp_2D_dropsurface::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------

	for (int step=0; step<stepsPerCycle; step++) {
		
		cummulativeSteps++;
		
		// ------------------------------
		// update density fields:
		// ------------------------------
		
		mcmp_compute_density_D2Q9 
		<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,nVoxels);
				
		hipDeviceSynchronize();
		
		// ------------------------------
		// update fluid fields:											   
		// ------------------------------ 
		
		if (potType == 1) {
			mcmp_compute_SC_forces_solid_1_D2Q9 
			<<<nBlocks,nThreads>>> (rA,rB,rS,FxA,FxB,FyA,FyB,nList,gAB,gAS,gBS,nVoxels);	
		}
		else if (potType == 2) {
			mcmp_compute_SC_forces_solid_2_D2Q9 
			<<<nBlocks,nThreads>>> (rA,rB,rS,FxA,FxB,FyA,FyB,nList,gAB,gAS,gBS,nVoxels);
		}	
		else if (potType == 3) {
			mcmp_compute_SC_forces_solid_3_D2Q9 
			<<<nBlocks,nThreads>>> (rA,rB,rS,FxA,FxB,FyA,FyB,nList,gAB,rAinS,rBinS,nVoxels);
		}			
					
		mcmp_compute_velocity_solid_D2Q9 
		<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,rS,FxA,FxB,FyA,FyB,u,v,pID,p,nVoxels);
		
		//mcmp_compute_velocity_D2Q9 
		//<<<nBlocks,nThreads>>> (f1A,f1B,rA,rB,FxA,FxB,FyA,FyB,u,v,nVoxels);
				
		mcmp_collide_stream_D2Q9 
		<<<nBlocks,nThreads>>> (f1A,f1B,f2A,f2B,rA,rB,u,v,FxA,FxB,FyA,FyB,streamIndex,nu,nVoxels);
		
		//mcmp_collide_stream_solid_D2Q9 
		//<<<nBlocks,nThreads>>> (f1A,f1B,f2A,f2B,rA,rB,rS,u,v,FxA,FxB,FyA,FyB,streamIndex,nu,nVoxels);
																 
		float* tempA = f1A;
		float* tempB = f1B;
		f1A = f2A;
		f1B = f2B;
		f2A = tempA;
		f2B = tempB;
		
		hipDeviceSynchronize();
				
	}
	
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
    hipMemcpy(uH, u, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(vH, v, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rAH, rA, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rBH, rB, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(rSH, rS, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);	
	
}



// --------------------------------------------------------
// Write output:
// --------------------------------------------------------

void mcmp_2D_dropsurface::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// function location:
	// "io/write_vtk_output.cuh"
	// ----------------------------------------------
	
	if (vtkFormat == "structured") {
		write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,rAH,uH,vH);
		//write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,rAH,rBH,rSH,uH,vH);
	}
	
}










