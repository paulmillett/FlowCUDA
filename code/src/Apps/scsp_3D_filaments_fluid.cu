#include "hip/hip_runtime.h"

# include "scsp_3D_filaments_fluid.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_3D_filaments_fluid::scsp_3D_filaments_fluid() : lbm(),filams()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = inputParams("Lattice/Nz",1);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	int sizeFIL = filams.get_max_array_size();	
	int sizeMAX = max(nVoxels,sizeFIL);	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (sizeMAX+(nThreads-1))/nThreads;  // integer division
	
	cout << "largest array size = " << sizeMAX << endl;
	cout << "nBlocks = " << nBlocks << ", nThreads = " << nThreads << endl;
		
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	nStepsEquilibrate = inputParams("Time/nStepsEquilibrate",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	shearVel = inputParams("LBM/shearVel",0.0);
	float Re = inputParams("LBM/Re",2.0);
	shearVel = 2.0*Re*nu/float(Nz);
	shearVel = 0.0;
	
	// ----------------------------------------------
	// Filaments Immersed-Boundary parameters:
	// ----------------------------------------------
		
	int nBeadsPerFilam = inputParams("IBM_FILAMS/nBeadsPerFilam",0);
	nFilams = inputParams("IBM_FILAMS/nFilams",1);
	ks = inputParams("IBM_FILAMS/ks",0.1);
	kb = inputParams("IBM_FILAMS/kb",0.1);
	fp = inputParams("IBM_FILAMS/fp",0.0);
	L0 = inputParams("IBM_FILAMS/L0",0.5);
	Pe = inputParams("IBM_FILAMS/Pe",0.0);
	PL = inputParams("IBM_FILAMS/PL",1.0);  // non-dimensional persistence length
	kT = inputParams("IBM_FILAMS/kT",0.0);
	gam = inputParams("IBM_FILAMS/gamma",0.1);
	nBeads = nBeadsPerFilam*nFilams;
	Lfil = float(nBeadsPerFilam)*L0;
	
	// ----------------------------------------------
	// IBM set flags for PBC's:
	// ----------------------------------------------
	
	filams.set_pbcFlag(1,1,0);
		
	// ----------------------------------------------
	// iolets parameters:
	// ----------------------------------------------
	
	numIolets = inputParams("Lattice/numIolets",2);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	iskip = inputParams("Output/iskip",1);
	jskip = inputParams("Output/jskip",1);
	kskip = inputParams("Output/kskip",1);
	nVTKOutputs = inputParams("Output/nVTKOutputs",0);
	precision = inputParams("Output/precision",3);
		
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	lbm.allocate_forces();
	filams.allocate();	
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_3D_filaments_fluid::~scsp_3D_filaments_fluid()
{
	lbm.deallocate();
	filams.deallocate();
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_3D_filaments_fluid::initSystem()
{
		
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the lattice assuming shear flow.
	// ----------------------------------------------	
	
	lbm.create_lattice_box_shear();
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
		
	lbm.stream_index_pull();
			
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
	
	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
		lbm.setW(i,0.0);
		lbm.setR(i,1.0);		
	}
	
	// ----------------------------------------------			
	// initialize filament immersed boundary info: 
	// ----------------------------------------------
	
	filams.create_first_filament();
	filams.duplicate_filaments();
	filams.assign_filamIDs_to_beads();
	
	fp = Pe*kT/Lfil/Lfil;
	kb = PL*kT;
	up = fp*L0/gam;  // active velocity per bead
	filams.set_ks(ks);
	filams.set_kb(kb);
	filams.set_fp(fp);
	filams.set_up(up);
	filams.set_filams_radii(0.5);
	cout << "  " << endl;
	cout << "Filament kT = " << kT << endl;
	cout << "Filament ks = " << ks << endl;
	cout << "Filament kb = " << kb << endl;
	cout << "Filament fp = " << fp << endl;
	cout << "Filament up = " << up << endl;
			
	// ----------------------------------------------
	// build the binMap array for neighbor lists: 
	// ----------------------------------------------
	
	filams.build_binMap(nBlocks,nThreads);
		
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	filams.memcopy_host_to_device();
		
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);	
		
	// ----------------------------------------------
	// set the random number seed: 
	// ----------------------------------------------
	
	srand(time(NULL));
	
	// ----------------------------------------------
	// randomly disperse filaments: 
	// ----------------------------------------------
		
	filams.randomize_filaments(Lfil+2.0);
		
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	filams.memcopy_device_to_host();
	writeOutput("macros",0);
	
	// ----------------------------------------------
	// set IBM velocities & forces to zero: 
	// ----------------------------------------------
	
	filams.zero_bead_velocities_forces(nBlocks,nThreads);
	
	// ----------------------------------------------
	// initialize cuRand state for the thermal noise
	// force:
	// ----------------------------------------------
	
	filams.initialize_cuRand(nBlocks,nThreads);
		
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_3D_filaments_fluid::cycleForward(int stepsPerCycle, int currentCycle)
{
		
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// if simulation just started, perform 
	// equilibration:
	// ----------------------------------------------
	
	if (cummulativeSteps == 0) {
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Equilibrating for " << nStepsEquilibrate << " steps..." << endl;
		for (int i=0; i<nStepsEquilibrate; i++) {
			if (i%10000 == 0) cout << "equilibration step " << i << endl;
			filams.stepIBM_Euler(lbm,nBlocks,nThreads);
			lbm.stream_collide_save_forcing(nBlocks,nThreads);	
			lbm.set_boundary_shear_velocity(-shearVel,shearVel,nBlocks,nThreads);
			hipDeviceSynchronize();
		}
		cout << " " << endl;
		cout << "... done equilibrating!" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;
	}
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
		
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;
		filams.stepIBM_Euler(lbm,nBlocks,nThreads);
		lbm.stream_collide_save_forcing(nBlocks,nThreads);	
		lbm.set_boundary_shear_velocity(-shearVel,shearVel,nBlocks,nThreads);
		hipDeviceSynchronize();
	}
	
	cout << cummulativeSteps << endl;	
		
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
	filams.memcopy_device_to_host();    
	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
		
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_3D_filaments_fluid::writeOutput(std::string tagname, int step)
{				
	
	if (step == 0) {
		// only print out vtk files
		lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision); 
		filams.write_output("filaments",step);
	}
	
	if (step > 0) { 					
		// write vtk output for LBM and IBM:
		int intervalVTK = nSteps/nVTKOutputs;
		if (nVTKOutputs == 0) intervalVTK = nSteps;
		if (step%intervalVTK == 0) {
			lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision);
			filams.write_output("filaments",step);
		}
	}	
}








