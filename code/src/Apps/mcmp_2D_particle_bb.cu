#include "hip/hip_runtime.h"

# include "mcmp_2D_particle_bb.cuh"
# include "../IO/GetPot"
# include <math.h>
# include <string> 
using namespace std;   



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

mcmp_2D_particle_bb::mcmp_2D_particle_bb() : lbm(), parts()
{	
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",9);	
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	gAB = inputParams("LBM/gAB",6.0);
	gAS = inputParams("LBM/gAS",6.0);
	gBS = inputParams("LBM/gBS",6.0); 	
	
	// ----------------------------------------------
	// Particles parameters:
	// ----------------------------------------------
	
	nParts = inputParams("Particles/nParts",1);				
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","structured");
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	parts.allocate();	
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

mcmp_2D_particle_bb::~mcmp_2D_particle_bb()
{
	lbm.deallocate();
	parts.deallocate();	
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void mcmp_2D_particle_bb::initSystem()
{
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the periodic lattice:
	// ----------------------------------------------	
	
	lbm.create_lattice_box_periodic();	
			
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
	
	// particle's initial position:
	parts.xH[0] = 100.0;
	parts.yH[0] = 75.0;
	parts.radH[0] = 20.0;
	
	// initialize solid field:
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {		
			int ndx = j*Nx + i;	
			lbm.setX(ndx,i);
			lbm.setY(ndx,j);
			lbm.setS(ndx,0);
			float dx = float(i) - parts.xH[0];
			float dy = float(j) - parts.yH[0];
			float rr = sqrt(dx*dx + dy*dy);
			if (rr <= parts.radH[0]) lbm.setS(ndx,1); 
		}
	}
	
	// initialize density fields: 
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {
			int ndx = j*Nx + i;
			int sij = lbm.getS(ndx);			
			float rhoA = 0.99;
			float rhoB = 0.01;					
			lbm.setRA(ndx,rhoA*float(1 - sij));
			lbm.setRB(ndx,rhoB*float(1 - sij));						
		}
	}
		
	// initialize velocity fields
	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
	}	
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
	
	lbm.stream_index_push();	
			
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------	
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	parts.memcopy_host_to_device();
	
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium_bb(nBlocks,nThreads);
	lbm.initial_particles_on_lattice(parts.x,parts.y,parts.rad,parts.pIDgrid,nParts,nBlocks,nThreads);
		
}



// --------------------------------------------------------
// Step forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void mcmp_2D_particle_bb::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------

	for (int step=0; step<stepsPerCycle; step++) {
		
		cummulativeSteps++;
		
		// ------------------------------
		// update density fields:
		// ------------------------------
		
		lbm.update_particles_on_lattice(parts.x,parts.y,parts.vx,parts.vy,parts.rad,
		                                parts.pIDgrid,nParts,nBlocks,nThreads);
		lbm.compute_density_bb(nBlocks,nThreads);
		hipDeviceSynchronize();
		
		// ------------------------------
		// update fluid fields:											   
		// ------------------------------ 
		
		lbm.compute_SC_forces_bb(nBlocks,nThreads);
		lbm.compute_velocity_bb(nBlocks,nThreads);
		lbm.collide_stream_bb(nBlocks,nThreads);
		lbm.bounce_back_moving(nBlocks,nThreads);
		lbm.swap_populations();				
		hipDeviceSynchronize();
				
	}
	
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
		
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);	
	
}



// --------------------------------------------------------
// Write output:
// --------------------------------------------------------

void mcmp_2D_particle_bb::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// function location:
	// "io/write_vtk_output.cuh"
	// ----------------------------------------------
	
	lbm.write_output(tagname,step); 

}










