#include "hip/hip_runtime.h"

# include "scsp_2D_active_droplet_3phi.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_2D_active_droplet_3phi::scsp_2D_active_droplet_3phi() : lbm()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",9);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	dropRad = inputParams("LBM/dropRad",10.0);
	
	// ----------------------------------------------
	// Output parameters:
	// ----------------------------------------------
	
	iskip = inputParams("Output/iskip",1);
	jskip = inputParams("Output/jskip",1);
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_2D_active_droplet_3phi::~scsp_2D_active_droplet_3phi()
{	
	lbm.deallocate();
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_2D_active_droplet_3phi::initSystem()
{
		
	// ----------------------------------------------
	// create the lattice using "box" function. 
	// ----------------------------------------------	
		
	lbm.create_lattice_box_periodic();
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
	
	lbm.stream_index_pull();
		
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
		
	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
		lbm.setR(i,1.0);	
	}
	
	// ----------------------------------------------			
	// initialize order parameter phi: 
	// ----------------------------------------------
	
	for (int j=0; j<Ny; j++) {
		for (int i=0; i<Nx; i++) {		
			int ndx = j*Nx + i;	
			float phi1 = 0.0;
			float phi2 = 0.0;
			// phi 1 domain:
			float centx = float(Nx)/2.0 + (2.0/3.0)*dropRad;
			float centy = float(Ny)/2.0;
			float dx = float(i) - centx;
			float dy = float(j) - centy;
			float r = sqrt(dx*dx + dy*dy);
			if (r < dropRad && i > Nx/2) phi1 = 1.0;
			// phi 2 domain:
			centx = float(Nx)/2.0 - (2.0/3.0)*dropRad;
			centy = float(Ny)/2.0;
			dx = float(i) - centx;
			dy = float(j) - centy;
			r = sqrt(dx*dx + dy*dy);
			if (r < dropRad && i < Nx/2) phi2 = 1.0;
			// set values:
			lbm.setPhi1(ndx,phi1);
			lbm.setPhi2(ndx,phi2);
			lbm.setPhi3(ndx,1.0 - phi1 - phi2);			
		}
	}
	
	// ----------------------------------------------			
	// relax phi fields:
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();	
	for (int i=0; i<10000; i++) {
		lbm.scsp_active_fluid_chemical_potential(nBlocks,nThreads);
		lbm.scsp_active_fluid_update_phi(nBlocks,nThreads);
	}	
	lbm.memcopy_device_to_host();	
	
	// ----------------------------------------------			
	// initialize orientation: 
	// ----------------------------------------------
		
	for (int i=0; i<nVoxels; i++) {
		float phi = lbm.getPhi1(i);
		float theta = 0.0;   //2.0*M_PI*((float)rand()/RAND_MAX - 0.5); 
		float px = 1.0;
		float py = 0.0;
		float pxr = px*cos(theta) - py*sin(theta);
		float pyr = px*sin(theta) + py*cos(theta);
		lbm.setPx(i,pxr*phi);
		lbm.setPy(i,pyr*phi);	
	}
		
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);
	
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_2D_active_droplet_3phi::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;	
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
	
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;		
		lbm.zero_forces(nBlocks,nThreads);
		lbm.scsp_active_fluid_chemical_potential(nBlocks,nThreads);
		lbm.scsp_active_fluid_molecular_field_with_phi(nBlocks,nThreads);		
		lbm.scsp_active_fluid_stress(nBlocks,nThreads);
		lbm.scsp_active_fluid_forces(nBlocks,nThreads);
		lbm.scsp_active_fluid_capillary_force(nBlocks,nThreads);
		lbm.scsp_active_update_orientation(nBlocks,nThreads);
		lbm.scsp_active_fluid_update_phi(nBlocks,nThreads);
		lbm.stream_collide_save_forcing(nBlocks,nThreads);
		hipDeviceSynchronize();
	}
	
	cout << cummulativeSteps << endl;	
	
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
		
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
	
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_2D_active_droplet_3phi::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// function location:
	// "io/write_vtk_output.cuh"
	// ----------------------------------------------
	
	lbm.write_output(tagname,step,iskip,jskip);
			
}







