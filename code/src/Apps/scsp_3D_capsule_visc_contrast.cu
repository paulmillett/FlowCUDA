
# include "scsp_3D_capsule_visc_contrast.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_3D_capsule_visc_contrast::scsp_3D_capsule_visc_contrast() : lbm(),ibm(),poisson()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = inputParams("Lattice/Nz",1);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	nStepsEquilibrate = inputParams("Time/nStepsEquilibrate",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	shearVel = inputParams("LBM/shearVel",0.0);
	float Re = inputParams("LBM/Re",2.0);
	
	// ----------------------------------------------
	// Immersed-Boundary parameters:
	// ----------------------------------------------
		
	int nNodesPerCell = inputParams("IBM/nNodesPerCell",0);
	nCells = inputParams("IBM/nCells",1);
	nNodes = nNodesPerCell*nCells;
	a = inputParams("IBM/a",6.0);
	float Ca = inputParams("IBM/Ca",1.0);
	gam = inputParams("IBM/gamma",0.1);
	ibmFile = inputParams("IBM/ibmFile","sphere.dat");
	ibmUpdate = inputParams("IBM/ibmUpdate","verlet");
	initRandom = inputParams("IBM/initRandom",1);	
	
	// ----------------------------------------------
	// IBM set flags for PBC's:
	// ----------------------------------------------
	
	ibm.set_pbcFlag(1,1,0);
		
	// ----------------------------------------------
	// iolets parameters:
	// ----------------------------------------------
	
	numIolets = inputParams("Lattice/numIolets",2);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","polydata");
	iskip = inputParams("Output/iskip",1);
	jskip = inputParams("Output/jskip",1);
	kskip = inputParams("Output/kskip",1);
	nVTKOutputs = inputParams("Output/nVTKOutputs",0);
	precision = inputParams("Output/precision",3);
		
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	lbm.allocate_forces();
	ibm.allocate();	
	
	// ----------------------------------------------
	// determine membrane parameters (see function
	// below), then calculate reference flux for no
	// capsules:
	// ----------------------------------------------
	
	calcMembraneParams(Re,Ca);
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_3D_capsule_visc_contrast::~scsp_3D_capsule_visc_contrast()
{
	lbm.deallocate();
	ibm.deallocate();
	poisson.deallocate();
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_3D_capsule_visc_contrast::initSystem()
{
		
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the lattice assuming shear flow.
	// ----------------------------------------------	
	
	lbm.create_lattice_box_shear();
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
		
	lbm.stream_index_pull();
			
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
	
	for (int i=0; i<nVoxels; i++) {
		lbm.setU(i,0.0);
		lbm.setV(i,0.0);
		lbm.setW(i,0.0);
		lbm.setR(i,1.0);		
	}
	
	// ----------------------------------------------			
	// initialize immersed boundary info: 
	// ----------------------------------------------
	
	ibm.read_ibm_information(ibmFile);
	ibm.duplicate_cells();
	ibm.assign_cellIDs_to_nodes();
	ibm.assign_refNode_to_cells();
	
	// ----------------------------------------------			
	// rescale capsule sizes for normal distribution: 
	// ----------------------------------------------
	
	cellSizes = inputParams("IBM/cellSizes","uniform");
	float stddevA = inputParams("IBM/stddevA",0.0);
	ibm.rescale_cell_radii(a,stddevA,cellSizes);
	
	// ----------------------------------------------
	// build the binMap array for neighbor lists: 
	// ----------------------------------------------
	
	ibm.build_binMap(nBlocks,nThreads);
		
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	ibm.memcopy_host_to_device();
		
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);	
	
	// ----------------------------------------------
	// calculate rest geometries for membrane: 
	// ----------------------------------------------
	
	ibm.rest_geometries_skalak(nBlocks,nThreads);
	
	// ----------------------------------------------
	// set the random number seed: 
	// ----------------------------------------------
	
	srand(time(NULL));
	
	// ----------------------------------------------
	// shrink and randomly disperse cells: 
	// ----------------------------------------------
		
	if (initRandom) {
		float scale = 1.0;   // 0.7;
		ibm.shrink_and_randomize_cells(scale,2.0,a+2.0);
		ibm.scale_equilibrium_cell_size(scale,nBlocks,nThreads);
	
		
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Relaxing capsules..." << endl;
		
		scale = 1.0/scale;
		ibm.relax_node_positions_skalak(90000,scale,0.1,nBlocks,nThreads);	
		ibm.relax_node_positions_skalak(90000,1.0,0.1,nBlocks,nThreads);
		
		cout << "... done relaxing" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;	
		
	}
		
	// ----------------------------------------------
	// initialize poisson solver:
	// ----------------------------------------------
	
	poisson.initialize(Nx,Ny,Nz);
	poisson.solve_poisson(ibm.faces,ibm.r,ibm.nFaces,nBlocks,nThreads);
	poisson.write_output("indicator",0,iskip,jskip,kskip,precision);
	
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	ibm.memcopy_device_to_host();
	writeOutput("macros",0);
	
	// ----------------------------------------------
	// set IBM velocities & forces to zero: 
	// ----------------------------------------------
	
	ibm.zero_velocities_forces(nBlocks,nThreads);
		
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_3D_capsule_visc_contrast::cycleForward(int stepsPerCycle, int currentCycle)
{
		
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// if simulation just started, perform 
	// equilibration:
	// ----------------------------------------------
	
	if (cummulativeSteps == 0) {
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Equilibrating for " << nStepsEquilibrate << " steps..." << endl;
		for (int i=0; i<nStepsEquilibrate; i++) {
			if (i%10000 == 0) cout << "equilibration step " << i << endl;
			poisson.solve_poisson(ibm.faces,ibm.r,ibm.nFaces,nBlocks,nThreads);
			ibm.stepIBM(lbm,nBlocks,nThreads);			
			lbm.stream_collide_save_forcing(nBlocks,nThreads);
			lbm.set_boundary_shear_velocity(-shearVel,shearVel,nBlocks,nThreads);
			hipDeviceSynchronize();
		}
		cout << " " << endl;
		cout << "... done equilibrating!" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;
	}
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
		
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;
		poisson.solve_poisson(ibm.faces,ibm.r,ibm.nFaces,nBlocks,nThreads);
		ibm.stepIBM(lbm,nBlocks,nThreads);		
		lbm.stream_collide_save_forcing(nBlocks,nThreads);	
		lbm.set_boundary_shear_velocity(-shearVel,shearVel,nBlocks,nThreads);
		hipDeviceSynchronize();
	}
	
	cout << cummulativeSteps << endl;	
		
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
	ibm.memcopy_device_to_host();    
	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
		
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_3D_capsule_visc_contrast::writeOutput(std::string tagname, int step)
{				
	
	if (step == 0) {
		// only print out vtk files
		lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision); 
		ibm.write_output("ibm",step);
	}
	
	if (step > 0) { 
		// analyze membrane geometry:
		ibm.membrane_geometry_analysis("capdata",step);
			
		// write vtk output for LBM and IBM:
		int intervalVTK = nSteps/nVTKOutputs;
		if (nVTKOutputs == 0) intervalVTK = nSteps;
		if (step%intervalVTK == 0) {
			lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision);
			ibm.write_output("ibm",step);
			poisson.write_output("indicator",step,iskip,jskip,kskip,precision);
		}
		
		// print out final averaged flow profile:
		if (step == nSteps) {
			lbm.print_flow_rate_xdir("flow_data",step);			
		}
	}	
}



// --------------------------------------------------------
// Calculate membrane elastic parameters.  Here, we
// calculate the appropriate values of nu, ks, and bodyForx
// that satisfy the given Re and Ca subject to the 
// conditions that maximum u < umax and ks < ksmax:
// --------------------------------------------------------

void scsp_3D_capsule_visc_contrast::calcMembraneParams(float Re, float Ca)
{
	// 'GetPot' object containing input parameters:
	GetPot inputParams("input.dat");
	cellProps = inputParams("IBM/cellProps","uniform");
	float stddevCa = inputParams("IBM/stddevCa",0.0);
	float Kv = inputParams("IBM/kv",0.0);
	float C = inputParams("IBM/C",2.0);
	float rho = 1.0;
	float h = float(Nz)/2.0;
	shearVel = Re*nu/h;
	
	// set the mechanical properties:
	ibm.calculate_cell_membrane_props(Re,Ca,stddevCa,a,h,rho,shearVel,Kv,C,cellProps);
}









