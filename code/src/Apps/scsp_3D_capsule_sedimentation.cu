#include "hip/hip_runtime.h"

# include "scsp_3D_capsule_sedimentation.cuh"
# include "../IO/GetPot"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_3D_capsule_sedimentation::scsp_3D_capsule_sedimentation() : lbm(),ibm()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);
	Nx = inputParams("Lattice/Nx",1);
	Ny = inputParams("Lattice/Ny",1);
	Nz = inputParams("Lattice/Nz",1);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	nStepsEquilibrate = inputParams("Time/nStepsEquilibrate",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	float Re = inputParams("LBM/Re",2.0);
	float umax = inputParams("LBM/umax",0.1);
	
	// ----------------------------------------------
	// Immersed-Boundary parameters:
	// ----------------------------------------------
	
	int nNodesPerCell = inputParams("IBM/nNodesPerCell",0);
	int nCells = inputParams("IBM/nCells",1);
	nNodes = nNodesPerCell*nCells;
	a = inputParams("IBM/a",10.0);
	fx = inputParams("IBM/fx",0.0);
	float Ca = inputParams("IBM/Ca",1.0);	
	float ksmax = inputParams("IBM/ksmax",0.002);
	gam = inputParams("IBM/gamma",0.1);
	ibmUpdate = inputParams("IBM/ibmUpdate","verlet");
	
	// ----------------------------------------------
	// IBM set flags for PBC's:
	// ----------------------------------------------
	
	ibm.set_pbcFlag(1,0,0);
		
	// ----------------------------------------------
	// iolets parameters:
	// ----------------------------------------------
	
	numIolets = inputParams("Lattice/numIolets",2);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","polydata");
	iskip = inputParams("Output/iskip",1);
	jskip = inputParams("Output/jskip",1);
	kskip = inputParams("Output/kskip",1);
	nVTKOutputs = inputParams("Output/nVTKOutputs",0);
	
	// ----------------------------------------------
	// allocate array memory (host & device):
	// ----------------------------------------------
	
	lbm.allocate();
	lbm.allocate_forces();
	ibm.allocate();	
	
	// ----------------------------------------------
	// determine membrane parameters (see function
	// below), then calculate reference flux for no
	// capsules:
	// ----------------------------------------------
	
	calcMembraneParams(Re,Ca,umax,ksmax);
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_3D_capsule_sedimentation::~scsp_3D_capsule_sedimentation()
{
	lbm.deallocate();
	ibm.deallocate();	
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_3D_capsule_sedimentation::initSystem()
{
		
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
		
	// ----------------------------------------------
	// create the lattice for channel flow:
	// ----------------------------------------------	
	
	lbm.create_lattice_box_channel();
	
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// ----------------------------------------------
		
	lbm.stream_index_pull();
			
	// ----------------------------------------------			
	// initialize velocities: 
	// ----------------------------------------------
	
	float h = float(Nz)/2.0;
	float w = float(Ny)/2.0;
	
	for (int k=0; k<Nz; k++) {
		for (int j=0; j<Ny; j++) {
			for (int i=0; i<Nx; i++) {
				int ndx = k*Nx*Ny + j*Nx + i;								
				lbm.setU(ndx,0.0);
				lbm.setV(ndx,0.0);
				lbm.setW(ndx,0.0);
				lbm.setR(ndx,1.0);
			}
		}
	}
	
	// ----------------------------------------------			
	// initialize immersed boundary info: 
	// ----------------------------------------------
		
	ibm.read_ibm_information("sphere.dat");
	ibm.duplicate_cells();
	ibm.assign_cellIDs_to_nodes();
	ibm.assign_refNode_to_cells();	
	
	// ----------------------------------------------
	// initialize position of cell: 
	// ----------------------------------------------
	
	float xshift = 20.0;
	float yshift = float(Ny)/2.0 + 0.5;
	float zshift = float(Nz)/2.0 + 0.5;
	
	ibm.shift_node_positions(0,xshift,yshift,zshift);
			
	// ----------------------------------------------
	// build the binMap array for neighbor lists: 
	// ----------------------------------------------
	
	ibm.build_binMap(nBlocks,nThreads); 
		
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
	lbm.memcopy_host_to_device();
	ibm.memcopy_host_to_device();
		
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	lbm.initial_equilibrium(nBlocks,nThreads);	
	
	// ----------------------------------------------
	// calculate rest geometries for membrane: 
	// ----------------------------------------------
	
	ibm.rest_geometries_skalak(nBlocks,nThreads);
	
	// ----------------------------------------------
	// set the random number seed: 
	// ----------------------------------------------
	
	srand(time(NULL));
								
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	ibm.memcopy_device_to_host();
	writeOutput("macros",0);
	
	// ----------------------------------------------
	// set IBM velocities & forces to zero: 
	// ----------------------------------------------
	
	ibm.zero_velocities_forces(nBlocks,nThreads);
	
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_3D_capsule_sedimentation::cycleForward(int stepsPerCycle, int currentCycle)
{
		
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;
	
	// ----------------------------------------------
	// if simulation just started, perform 
	// equilibration:
	// ----------------------------------------------
	
	if (cummulativeSteps == 0) {
		cout << " " << endl;
		cout << "-----------------------------------------------" << endl;
		cout << "Equilibrating for " << nStepsEquilibrate << " steps..." << endl;
		for (int i=0; i<nStepsEquilibrate; i++) {
			if (i%10000 == 0) cout << "equilibration step " << i << endl;
			// decide on update type:
			if (ibmUpdate == "ibm") {
				stepIBM();
			} else if (ibmUpdate == "verlet") {
				stepVerlet();
			}		
		}
		cout << " " << endl;
		cout << "... done equilibrating!" << endl;
		cout << "-----------------------------------------------" << endl;
		cout << " " << endl;
	}
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
	
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;	
		// decide on update type:
		if (ibmUpdate == "ibm") {
			stepIBM();
		} else if (ibmUpdate == "verlet") {
			stepVerlet();
		}		
	}
	
	cout << cummulativeSteps << endl;	
		
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
	lbm.memcopy_device_to_host();
	ibm.memcopy_device_to_host();    
	
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
		
}



// --------------------------------------------------------
// Take a time-step with the traditional IBM approach:
// --------------------------------------------------------

void scsp_3D_capsule_sedimentation::stepIBM()
{
	// zero fluid forces:
	lbm.zero_forces(nBlocks,nThreads);
	
	// re-build bin lists for IBM nodes:
	ibm.reset_bin_lists(nBlocks,nThreads);
	ibm.build_bin_lists(nBlocks,nThreads);
			
	// compute IBM node forces:
	ibm.compute_node_forces_skalak(nBlocks,nThreads);
	ibm.nonbonded_node_interactions(nBlocks,nThreads);
	ibm.wall_forces_ydir_zdir(nBlocks,nThreads);
	ibm.add_xdir_force_to_nodes(nBlocks,nThreads,fx);
	lbm.interpolate_velocity_to_IBM(nBlocks,nThreads,ibm.nodes,nNodes);
			
	// update fluid:
	lbm.extrapolate_forces_from_IBM(nBlocks,nThreads,ibm.nodes,nNodes);
	lbm.stream_collide_save_forcing(nBlocks,nThreads);
	//lbm.set_channel_wall_velocity(0.0,nBlocks,nThreads);
	lbm.set_boundary_duct_density(nBlocks,nThreads);
	
	// update membrane:
	//ibm.update_node_positions(nBlocks,nThreads);
	ibm.update_node_positions_verlet_1(nBlocks,nThreads);
	
	// CUDA sync
	hipDeviceSynchronize();
}



// --------------------------------------------------------
// Take a time-step with the velocity-Verlet approach for IBM:
// --------------------------------------------------------

void scsp_3D_capsule_sedimentation::stepVerlet()
{
	// zero fluid forces:
	lbm.zero_forces(nBlocks,nThreads);
	
	// first step of IBM velocity verlet:
	ibm.update_node_positions_verlet_1(nBlocks,nThreads);
	
	// re-build bin lists for IBM nodes:
	ibm.reset_bin_lists(nBlocks,nThreads);
	ibm.build_bin_lists(nBlocks,nThreads);
			
	// compute IBM node forces:
	ibm.compute_node_forces_skalak(nBlocks,nThreads);
	ibm.nonbonded_node_interactions(nBlocks,nThreads);
	ibm.wall_forces_ydir_zdir(nBlocks,nThreads);
	ibm.add_xdir_force_to_nodes(nBlocks,nThreads,fx);
			
	// update fluid:
	lbm.viscous_force_IBM_LBM(nBlocks,nThreads,gam,ibm.nodes,nNodes);
	lbm.stream_collide_save_forcing(nBlocks,nThreads);
	//lbm.set_channel_wall_velocity(0.0,nBlocks,nThreads);
	lbm.set_boundary_duct_density(nBlocks,nThreads);
	
	// second step of IBM velocity verlet:
	ibm.update_node_positions_verlet_2(nBlocks,nThreads);
	
	// CUDA sync		
	hipDeviceSynchronize();
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_3D_capsule_sedimentation::writeOutput(std::string tagname, int step)
{				
	
	int precision = 3;
	
	if (step == 0) {
		// only print out vtk files
		lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision); 
		ibm.write_output("ibm",step);
	}
	
	if (step > 0) { 
		// analyze membrane geometry:
		ibm.capsule_geometry_analysis(step);
		ibm.output_capsule_data();
			
		// write vtk output for LBM and IBM:
		int intervalVTK = nSteps/nVTKOutputs;
		if (nVTKOutputs == 0) intervalVTK = nSteps;
		if (step%intervalVTK == 0) {
			lbm.vtk_structured_output_ruvw(tagname,step,iskip,jskip,kskip,precision); 
			ibm.write_output("ibm",step);
		}
		
		// print out final averaged flow profile:
		if (step == nSteps) {
			lbm.print_flow_rate_xdir("flow_data",step);			
		}
	}	
}



// --------------------------------------------------------
// Calculate membrane elastic parameters.  Here, we
// calculate the appropriate values of nu, ks, and bodyForx
// that satisfy the given Re and Ca subject to the 
// conditions that maximum u < umax and ks < ksmax:
// --------------------------------------------------------

void scsp_3D_capsule_sedimentation::calcMembraneParams(float Re, float Ca, float umax, float Ksmax)
{
	// assumed parameters:
	float rho = 1.0;
	float w = float(Ny)/2.0;
	float h = float(Nz)/2.0;
	float Dh = 4.0*(4.0*w*h)/(4.0*(w+h));
	
	// my calculations:
	float Ks = 0.0009;
	float Kb = 0.01;   // Ks*a*a*0.00287*sqrt(3);
		
	// assign values for ks and nu:
	ibm.set_ks(Ks); 
	ibm.set_kb(Kb);
	//ibm.set_kv(0.5);
	ibm.set_ka(0.0007);
	ibm.set_kag(0.0);
	ibm.set_C(2.0);
	lbm.setNu(nu);   
	
	// output the results:
	cout << "  " << endl;
	cout << "hydraulic diameter = " << Dh << endl;
	cout << "umax = " << umax << endl;
	cout << "ks = " << Ks << endl;
	cout << "kb = " << Kb << endl;
	cout << "nu = " << nu << endl;
	cout << "fx = " << fx << endl;
	cout << "  " << endl;
	cout << "Re = " << Re << endl;
	cout << "Ca = " << rho*nu*umax*a/h/Ks << endl;
	cout << "  " << endl;
		
}



