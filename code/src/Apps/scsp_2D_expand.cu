#include "hip/hip_runtime.h"

# include "scsp_2D_expand.cuh"
# include "../D2Q9/scsp/scsp_initial_equilibrium_D2Q9.cuh"
# include "../D2Q9/scsp/scsp_stream_collide_save_forcing_D2Q9.cuh"
# include "../D2Q9/scsp/scsp_zero_forces_D2Q9.cuh"
# include "../D2Q9/init/stream_index_builder_D2Q9.cuh"
# include "../IBM/2D/compute_node_force_IBM2D.cuh"
# include "../IBM/2D/extrapolate_force_IBM2D.cuh"
# include "../IBM/2D/interpolate_velocity_IBM2D.cuh"
# include "../IBM/2D/set_reference_node_positions_IBM2D.cuh"
# include "../IBM/2D/update_node_position_IBM2D.cuh"
# include "../IBM/2D/update_node_ref_position_IBM2D.cuh"
# include "../IO/GetPot"
# include "../IO/read_lattice_geometry.cuh"
# include "../IO/write_vtk_output.cuh"
# include "../Lattice/lattice_builders_D2Q9.cuh"
# include <string>
# include <math.h>
using namespace std;  



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

scsp_2D_expand::scsp_2D_expand()
{		
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	
	// ----------------------------------------------
	// lattice parameters:
	// ----------------------------------------------
	
	nVoxels = inputParams("Lattice/nVoxels",0);
	Q = inputParams("Lattice/Q",19);	
	
	// ----------------------------------------------
	// GPU parameters:
	// ----------------------------------------------
	
	nThreads = inputParams("GPU/nThreads",512);
	nBlocks = (nVoxels+(nThreads-1))/nThreads;  // integer division
	
	// ----------------------------------------------
	// time parameters:
	// ----------------------------------------------
	
	nSteps = inputParams("Time/nSteps",0);
	
	// ----------------------------------------------
	// Lattice Boltzmann parameters:
	// ----------------------------------------------
	
	nu = inputParams("LBM/nu",0.1666666);
	
	// ----------------------------------------------
	// Immersed-Boundary parameters:
	// ----------------------------------------------
	
	nNodes = inputParams("IBM/nNodes",0);
	kstiff = inputParams("IBM/kstiff",0.0);
	nBlocksIB = (nNodes+(nThreads-1))/nThreads; // integer division	
	
	// ----------------------------------------------
	// iolets parameters:
	// ----------------------------------------------
	
	numIolets = inputParams("Lattice/numIolets",2);
	
	// ----------------------------------------------
	// output parameters:
	// ----------------------------------------------
	
	vtkFormat = inputParams("Output/format","polydata");
	
	// ----------------------------------------------
	// allocate array memory (host):
	// ----------------------------------------------
	
    uH = (float*)malloc(nVoxels*sizeof(float));
    vH = (float*)malloc(nVoxels*sizeof(float));
    rH = (float*)malloc(nVoxels*sizeof(float));
	xIBH = (float*)malloc(nNodes*sizeof(float));
	yIBH = (float*)malloc(nNodes*sizeof(float));
	xIBH_start = (float*)malloc(nNodes*sizeof(float));
	yIBH_start = (float*)malloc(nNodes*sizeof(float));
	xIBH_end = (float*)malloc(nNodes*sizeof(float));
	yIBH_end = (float*)malloc(nNodes*sizeof(float));
	nListH = (int*)malloc(nVoxels*Q*sizeof(int));
	voxelTypeH = (int*)malloc(nVoxels*sizeof(int));
	streamIndexH = (int*)malloc(nVoxels*Q*sizeof(int));
	xH = (int*)malloc(nVoxels*sizeof(int));
	yH = (int*)malloc(nVoxels*sizeof(int));
	ioletsH = (iolet2D*)malloc(numIolets*sizeof(iolet2D));
	
	// ----------------------------------------------
	// allocate array memory (device):
	// ----------------------------------------------
	
	hipMalloc((void **) &u, nVoxels*sizeof(float));
	hipMalloc((void **) &v, nVoxels*sizeof(float));
	hipMalloc((void **) &r, nVoxels*sizeof(float));
	hipMalloc((void **) &f1, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &f2, nVoxels*Q*sizeof(float));
	hipMalloc((void **) &Fx, nVoxels*sizeof(float));
	hipMalloc((void **) &Fy, nVoxels*sizeof(float));
	hipMalloc((void **) &xIB, nNodes*sizeof(float));
	hipMalloc((void **) &yIB, nNodes*sizeof(float));
	hipMalloc((void **) &xIBref, nNodes*sizeof(float));
	hipMalloc((void **) &yIBref, nNodes*sizeof(float));
	hipMalloc((void **) &xIBref_start, nNodes*sizeof(float));
	hipMalloc((void **) &yIBref_start, nNodes*sizeof(float));
	hipMalloc((void **) &xIBref_end, nNodes*sizeof(float));
	hipMalloc((void **) &yIBref_end, nNodes*sizeof(float));
	hipMalloc((void **) &vxIB, nNodes*sizeof(float));
	hipMalloc((void **) &vyIB, nNodes*sizeof(float));
	hipMalloc((void **) &fxIB, nNodes*sizeof(float));
	hipMalloc((void **) &fyIB, nNodes*sizeof(float));
	hipMalloc((void **) &voxelType, nVoxels*sizeof(int));
	hipMalloc((void **) &streamIndex, nVoxels*Q*sizeof(int));	
	hipMalloc((void **) &iolets, numIolets*sizeof(iolet2D));
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

scsp_2D_expand::~scsp_2D_expand()
{
	
	// ----------------------------------------------
	// free array memory (host):
	// ----------------------------------------------
	
	free(uH);
	free(vH);
	free(rH);
	free(nListH);
	free(voxelTypeH);
	free(streamIndexH);
	free(xH);
	free(yH);
	free(ioletsH);
	free(xIBH);
	free(yIBH);
	free(xIBH_start);
	free(yIBH_start);
	free(xIBH_end);
	free(yIBH_end);
		
	// ----------------------------------------------
	// free array memory (device):
	// ----------------------------------------------
	
	hipFree(u);
	hipFree(v);
	hipFree(r);
	hipFree(f1);
	hipFree(f2);
	hipFree(Fx);
	hipFree(Fy);
	hipFree(voxelType);
	hipFree(streamIndex);
	hipFree(iolets);
	hipFree(xIB);
	hipFree(yIB);
	hipFree(xIBref);
	hipFree(yIBref);
	hipFree(xIBref_start);
	hipFree(yIBref_start);
	hipFree(xIBref_end);
	hipFree(yIBref_end);
	hipFree(vxIB);
	hipFree(vyIB);
	hipFree(fxIB);
	hipFree(fyIB);
	
}



// --------------------------------------------------------
// Initialize system:
// --------------------------------------------------------

void scsp_2D_expand::initSystem()
{
	
	// ----------------------------------------------
	// 'GetPot' object containing input parameters:
	// ----------------------------------------------
	
	GetPot inputParams("input.dat");
	string latticeSource = inputParams("Lattice/source","box");	
	
	// ----------------------------------------------
	// create the lattice using "box" function.
	// function location:
	// "lattice/lattice_builders_D2Q9.cuh"	 
	// ----------------------------------------------	
	
	if (latticeSource == "box") {
		Nx = inputParams("Lattice/Nx",0);
		Ny = inputParams("Lattice/Ny",0);
		Nz = 1;
		int flowDir = inputParams("Lattice/flowDir",0);
		int xLBC = inputParams("Lattice/xLBC",0);
		int xUBC = inputParams("Lattice/xUBC",0);
		int yLBC = inputParams("Lattice/yLBC",0);
		int yUBC = inputParams("Lattice/yUBC",0);			
		build_box_lattice_D2Q9(nVoxels,flowDir,Nx,Ny,
		                       xLBC,xUBC,yLBC,yUBC,
		                       voxelTypeH,nListH);
	}	
		
	// ----------------------------------------------		
	// build the streamIndex[] array.  
	// function location:
	// "D2Q9/init/stream_index_builder_D2Q9.cuh"
	// ----------------------------------------------
		
	stream_index_pull_D2Q9(nVoxels,nListH,streamIndexH);
	
	// ----------------------------------------------			
	// initialize inlets/outlets: 
	// ----------------------------------------------
	
	// I'm assuming there are 2 iolets!!!!
	ioletsH[0].type = inputParams("Iolet1/type",1);
	ioletsH[0].uBC = inputParams("Iolet1/uBC",0.0);
	ioletsH[0].vBC = inputParams("Iolet1/vBC",0.0);
	ioletsH[0].rBC = inputParams("Iolet1/rBC",1.0);
	ioletsH[0].pBC = inputParams("Iolet1/pBC",0.0);
	
	ioletsH[1].type = inputParams("Iolet2/type",1);
	ioletsH[1].uBC = inputParams("Iolet2/uBC",0.0);
	ioletsH[1].vBC = inputParams("Iolet2/vBC",0.0);
	ioletsH[1].rBC = inputParams("Iolet2/rBC",1.0);
	ioletsH[1].pBC = inputParams("Iolet2/pBC",0.0);	
		
	// ----------------------------------------------			
	// edit inlet condition: 
	// ----------------------------------------------
	
	for (int i=0; i<Nx; i++) {
		int j = Ny - 1;
		int ndx = j*Nx + i;
		if (i < 120 || i > 140) {
			voxelTypeH[ndx] = 0;
		} 
	}	
	
	// ----------------------------------------------			
	// initialize macros: 
	// ----------------------------------------------
	
	for (int i=0; i<nVoxels; i++) {
		uH[i] = 0.00;
		vH[i] = 0.00;
		rH[i] = 1.0;
	} 
	
	// ----------------------------------------------			
	// initialize immersed boundary info: 
	// ----------------------------------------------
			
	float xcent = 99.5;
	float ycent = 198.5;
	float radiusx = 50.0;
	float radiusy = 50.0;
	for (int i=0; i<nNodes; i++) { 
		xIBH_start[i] = xcent - radiusx*cos(1.0*M_PI*float(i)/(nNodes-1));
		yIBH_start[i] = ycent - radiusy*sin(1.0*M_PI*float(i)/(nNodes-1));
	}
	radiusx = 50.0;
	radiusy = 100.0;
	for (int i=0; i<nNodes; i++) { 
		xIBH_end[i] = xcent - radiusx*cos(1.0*M_PI*float(i)/(nNodes-1));
		yIBH_end[i] = ycent - radiusy*sin(1.0*M_PI*float(i)/(nNodes-1));		
	}
	
	for (int i=0; i<nNodes; i++) { 
		xIBH[i] = xIBH_start[i];
		yIBH[i] = yIBH_start[i];		
	}
	
	
	// ----------------------------------------------
	// write initial output file:
	// ----------------------------------------------
	
	writeOutput("macros",0);
	
	// ----------------------------------------------		
	// copy arrays from host to device: 
	// ----------------------------------------------
	
    hipMemcpy(u, uH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(v, vH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(r, rH, sizeof(float)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(xIB, xIBH, sizeof(float)*nNodes, hipMemcpyHostToDevice);
	hipMemcpy(yIB, yIBH, sizeof(float)*nNodes, hipMemcpyHostToDevice);
	hipMemcpy(xIBref_start, xIBH_start, sizeof(float)*nNodes, hipMemcpyHostToDevice);
	hipMemcpy(yIBref_start, yIBH_start, sizeof(float)*nNodes, hipMemcpyHostToDevice);
	hipMemcpy(xIBref_end, xIBH_end, sizeof(float)*nNodes, hipMemcpyHostToDevice);
	hipMemcpy(yIBref_end, yIBH_end, sizeof(float)*nNodes, hipMemcpyHostToDevice);
	hipMemcpy(voxelType, voxelTypeH, sizeof(int)*nVoxels, hipMemcpyHostToDevice);
	hipMemcpy(streamIndex, streamIndexH, sizeof(int)*nVoxels*Q, hipMemcpyHostToDevice);
	hipMemcpy(iolets, ioletsH, sizeof(iolet2D)*numIolets, hipMemcpyHostToDevice);
	
	// ----------------------------------------------
	// initialize equilibrium populations: 
	// ----------------------------------------------
	
	scsp_initial_equilibrium_D2Q9 
	<<<nBlocks,nThreads>>> (f1,r,u,v,nVoxels);	
	
	// ----------------------------------------------
	// define reference IBM node positions: 
	// ----------------------------------------------
	
	set_reference_node_positions_IBM2D
	<<<nBlocksIB,nThreads>>> (xIB,yIB,xIBref,yIBref,nNodes);
	
}



// --------------------------------------------------------
// Cycle forward
// (this function iterates the system by a certain 
//  number of time steps between print-outs):
// --------------------------------------------------------

void scsp_2D_expand::cycleForward(int stepsPerCycle, int currentCycle)
{
	
	// ----------------------------------------------
	// determine the cummulative number of steps at the
	// beginning of this cycle:
	// ----------------------------------------------
	
	int cummulativeSteps = stepsPerCycle*currentCycle;	
	
	// ----------------------------------------------
	// loop through this cycle:
	// ----------------------------------------------
	
	for (int step=0; step<stepsPerCycle; step++) {
		cummulativeSteps++;
		
		update_node_ref_position_IBM2D
		<<<nBlocksIB,nThreads>>> 
		(xIBref,yIBref,xIBref_start,yIBref_start,xIBref_end,yIBref_end,cummulativeSteps,nSteps,nNodes);
		
		scsp_zero_forces_D2Q9
		<<<nBlocks,nThreads>>> (Fx,Fy,nVoxels);
		
		compute_node_force_IBM2D
		<<<nBlocksIB,nThreads>>> (xIB,yIB,xIBref,yIBref,fxIB,fyIB,kstiff,nNodes);
		
		extrapolate_force_IBM2D
		<<<nBlocksIB,nThreads>>> (xIB,yIB,fxIB,fyIB,Fx,Fy,Nx,nNodes);
				
		scsp_stream_collide_save_forcing_D2Q9 
		<<<nBlocks,nThreads>>> (f1,f2,r,u,v,Fx,Fy,streamIndex,voxelType,iolets,nu,nVoxels);
		float* temp = f1;
		f1 = f2;
		f2 = temp;
		
		interpolate_velocity_IBM2D
		<<<nBlocksIB,nThreads>>> (xIB,yIB,vxIB,vyIB,u,v,Nx,nNodes);
		
		update_node_position_IBM2D
		<<<nBlocksIB,nThreads>>> (xIB,yIB,vxIB,vyIB,nNodes);														 
		
		hipDeviceSynchronize();
	}
	
	// ----------------------------------------------
	// copy arrays from device to host:
	// ----------------------------------------------
	
    hipMemcpy(rH, r, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(uH, u, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(vH, v, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(xIBH, xIB, sizeof(float)*nNodes, hipMemcpyDeviceToHost);
	hipMemcpy(yIBH, yIB, sizeof(float)*nNodes, hipMemcpyDeviceToHost);
		
	// ----------------------------------------------
	// write output from this cycle:
	// ----------------------------------------------
	
	writeOutput("macros",cummulativeSteps);
	
}



// --------------------------------------------------------
// Write output to file
// --------------------------------------------------------

void scsp_2D_expand::writeOutput(std::string tagname, int step)
{
	
	// ----------------------------------------------
	// decide which VTK file format to use for output
	// function location:
	// "io/write_vtk_output.cuh"
	// ----------------------------------------------
	
	write_vtk_structured_grid_2D(tagname,step,Nx,Ny,Nz,rH,uH,vH);
	write_vtk_immersed_boundary_2D("ibm",step,nNodes,xIBH,yIBH);
		
}







