#include "hip/hip_runtime.h"

# include "class_poisson_ibm3D.cuh"
# include <math.h>
# include <iostream>
using namespace std;



// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

class_poisson_ibm3D::class_poisson_ibm3D()
{
	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

class_poisson_ibm3D::~class_poisson_ibm3D()
{
		
}



// --------------------------------------------------------
// Initialize arrays:
// --------------------------------------------------------

void class_poisson_ibm3D::initialize(int Nxin, int Nyin, int Nzin)
{
	
	Nx = Nxin;
	Ny = Nyin;
	Nz = Nzin;
	nVoxels = Nx*Ny*Nz;
	
	// define cuFFT plan (here, the first dimension must be the slowest changing one i.e. Nz)
	hipfftPlan3d(&plan, Nz, Ny, Nx, HIPFFT_C2C);	
	
	// wave-vector arrays (host)
	float* kxH = new float[Nx];
	float* kyH = new float[Ny];
	float* kzH = new float[Nz];
	for (int i=0; i<=Nx/2; i++)   kxH[i] = float(i)*2*M_PI/float(Nx);
	for (int i=Nx/2+1; i<Nx; i++) kxH[i] = float(i-Nx)*2*M_PI/float(Nx);
	for (int j=0; j<=Ny/2; j++)   kyH[j] = float(j)*2*M_PI/float(Ny);
	for (int j=Ny/2+1; j<Ny; j++) kyH[j] = float(j-Ny)*2*M_PI/float(Ny);	
	for (int k=0; k<=Nz/2; k++)   kzH[k] = float(k)*2*M_PI/float(Nz);
	for (int k=Nz/2+1; k<Nz; k++) kzH[k] = float(k-Nz)*2*M_PI/float(Nz);
	
	// allocate host arrays
	indicatorH = (float*)malloc(nVoxels*sizeof(float));
	GH = (float3*)malloc(nVoxels*sizeof(float3));
	GxH = (float*)malloc(nVoxels*sizeof(float));
	GyH = (float*)malloc(nVoxels*sizeof(float));
	GzH = (float*)malloc(nVoxels*sizeof(float));
	
	// allocate device arrays
	hipMalloc((void**)&kx, sizeof(float)*Nx);
	hipMalloc((void**)&ky, sizeof(float)*Ny);
	hipMalloc((void**)&kz, sizeof(float)*Nz);
	hipMalloc((void**)&rhs, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&indicator, sizeof(float)*Nx*Ny*Nz);	
	hipMalloc((void**)&G, sizeof(float3)*Nx*Ny*Nz);
	
	// memcopy wave-vector arrays to device
	hipMemcpy(kx,kxH,sizeof(float)*Nx, hipMemcpyHostToDevice);
	hipMemcpy(ky,kyH,sizeof(float)*Ny, hipMemcpyHostToDevice);
	hipMemcpy(kz,kzH,sizeof(float)*Nz, hipMemcpyHostToDevice);
	
}



// --------------------------------------------------------
// Solve poisson equation:
// --------------------------------------------------------

void class_poisson_ibm3D::solve_poisson(triangle* faces, float3* r, int nFaces, int nBlocks, int nThreads)
{
	// zero the 'G' vector array:
	zero_G_poisson_IBM3D
	<<<nBlocks,nThreads>>> (G,nVoxels);	
		
	// extrapolate IBM interface normal vectors to fluid grid:
	extrapolate_interface_normal_poisson_IBM3D
	<<<nBlocks,nThreads>>> (r,G,Nx,Ny,Nz,nFaces,faces);	
	
	//test_interface_normal_poisson_IBM3D            // note this is just a test function
	//<<<nBlocks,nThreads>>> (G,Nx,Ny,Nz,nVoxels);	
	
	// calculate RHS of poisson equation (div.G):
	calculate_rhs_poisson_IBM3D
	<<<nBlocks,nThreads>>> (G,rhs,nVoxels,Nx,Ny,Nz);
	
	// forward FFT (in-place):
	hipfftExecC2C(plan, rhs, rhs, HIPFFT_FORWARD);
	
	// solve poisson equation in Fourier space:
	solve_poisson_inplace
	<<<nBlocks,nThreads>>> (rhs,kx,ky,kz,Nx,Ny,Nz);
	
	// inverse FFT (in-place):
	hipfftExecC2C(plan, rhs, rhs, HIPFFT_BACKWARD);
	
	// change solution from complex to real:
	complex2real
	<<<nBlocks,nThreads>>> (rhs,indicator,nVoxels);
}



// --------------------------------------------------------
// write output for the 'indicatorH' array:
// --------------------------------------------------------

void class_poisson_ibm3D::write_output(std::string tagname, int tagnum,
                                       int iskip, int jskip, int kskip, int precision)
{
	// first, do a memcopy from device to host:
	hipMemcpy(indicatorH, indicator, sizeof(float)*nVoxels, hipMemcpyDeviceToHost);
	hipMemcpy(GH, G, sizeof(float3)*nVoxels, hipMemcpyDeviceToHost);
	for (int i=0; i<nVoxels; i++) {
		GxH[i] = GH[i].x;
		GyH[i] = GH[i].y;
		GzH[i] = GH[i].z;
	}
	// second, write the output:
	//write_vtk_structured_grid(tagname,tagnum,Nx,Ny,Nz,indicatorH,iskip,jskip,kskip,precision);
	write_vtk_structured_grid(tagname,tagnum,Nx,Ny,Nz,indicatorH,GxH,GyH,GzH,iskip,jskip,kskip,precision);
}



// --------------------------------------------------------
// Deallocate arrays:
// --------------------------------------------------------

void class_poisson_ibm3D::deallocate()
{
	hipfftDestroy(plan);
	hipFree(kx);
	hipFree(ky);
	hipFree(kz);
	hipFree(rhs);
	hipFree(G);
	hipFree(indicator);
	free(indicatorH);
	free(GH);
	free(GxH);
	free(GyH);
	free(GzH);
}



