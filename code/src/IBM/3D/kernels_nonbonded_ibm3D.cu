#include "hip/hip_runtime.h"

# include "kernels_nonbonded_ibm3D.cuh"
# include <stdio.h>



// --------------------------------------------------------
// IBM3D kernel to reset bin arrays:
// --------------------------------------------------------

__global__ void reset_bin_lists_IBM3D(
	int* binOccupancy,
	int* binMembers,
	int binMax,
	int nBins)
{
	// define bin:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBins) {
		
		// -------------------------------
		// reset binOccupancy[] to zero,
		// and binMembers[] array to -1:
		// -------------------------------
		
		binOccupancy[i] = 0;
		int offst = i*binMax;
		for (int k=offst; k<offst+binMax; k++) {
			binMembers[k] = -1;
		}
		
	}	
}



// --------------------------------------------------------
// IBM3D kernel to assign nodes to bins:
// --------------------------------------------------------

__global__ void build_bin_lists_IBM3D(
	float3* vertR,
	int* binOccupancy,
	int* binMembers,	
	int3 numBins,	
	float sizeBins,
	int nNodes,
	int binMax)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nNodes) {		
		
		// -------------------------------
		// calculate bin ID:
		// -------------------------------
		
		int binID = int(floor(vertR[i].x/sizeBins))*numBins.z*numBins.y +  
			        int(floor(vertR[i].y/sizeBins))*numBins.z +
		            int(floor(vertR[i].z/sizeBins));		
						
		// -------------------------------
		// update the lists:
		// -------------------------------
		
		if (binID >= 0 && binID < numBins.x*numBins.y*numBins.z) {
			atomicAdd(&binOccupancy[binID],1);
			int offst = binID*binMax;
			for (int k=offst; k<offst+binMax; k++) {
				int flag = atomicCAS(&binMembers[k],-1,i); 
				if (flag == -1) break;  
			}
		}
		
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate nonbonded node interactions
// using the bin lists:
// --------------------------------------------------------

__global__ void nonbonded_node_interactions_IBM3D(
	float3* vertR,
	float3* vertF,
	int* binOccupancy,
	int* binMembers,
	int* binMap,
	int* cellIDs,
	int3 numBins,	
	float sizeBins,
	float repA,
	float repD,
	float repFmax,
	int nNodes,
	int binMax,
	int nnbins,
	float3 Box,
	int3 pbcFlag)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nNodes) {		
		
		// -------------------------------
		// calculate bin ID:
		// -------------------------------
		
		int binID = int(floor(vertR[i].x/sizeBins))*numBins.z*numBins.y +  
			        int(floor(vertR[i].y/sizeBins))*numBins.z +
		            int(floor(vertR[i].z/sizeBins));		
		
		// -------------------------------
		// loop over nodes in the same bin:
		// -------------------------------
				
		int offst = binID*binMax;
		int occup = binOccupancy[binID];
		
		/*
		if (occup > binMax) {
			printf("Warning: linked-list bin has exceeded max capacity.  Occup. # = %i \n",occup);
		}
		*/
						
		for (int k=offst; k<offst+occup; k++) {
			int j = binMembers[k];
			if (i==j) continue;
			if (cellIDs[i]==cellIDs[j]) continue;
			pairwise_interaction_forces(i,j,repA,repD,repFmax,vertR,vertF,Box,pbcFlag);			
		}
		
		// -------------------------------
		// loop over neighboring bins:
		// -------------------------------
		
        for (int b=0; b<nnbins; b++) {
            // get neighboring bin ID
			int naborbinID = binMap[binID*nnbins + b];
			offst = naborbinID*binMax;
			occup = binOccupancy[naborbinID];
			// loop over nodes in this bin:
			for (int k=offst; k<offst+occup; k++) {
				int j = binMembers[k];
				if (cellIDs[i]==cellIDs[j]) continue;				
				pairwise_interaction_forces(i,j,repA,repD,repFmax,vertR,vertF,Box,pbcFlag);			
			}
		}
				
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate i-j force:
// --------------------------------------------------------

__device__ inline void pairwise_interaction_forces(
	const int i, 
	const int j,
	const float repA,
	const float repD,
	const float repFmax,
	float3* R,
	float3* F,
	float3 Box,
	int3 pbcFlag)
{
	float3 rij = R[i] - R[j];
	rij -= roundf(rij/Box)*Box*pbcFlag;  // PBC's	
	const float r = length(rij);
	if (r < repD) {
		// if separation is too small, we must adjust positions:
		/*
		if (r < 1.732) {
			float dr = 1.732 - r;
			// avoid adjusting positions twice:
			if (i < j) {
				R[i] += 0.5*dr*(rij/r);
				atomicAdd(&R[j].x,-0.5*dr*(rij.x/r));
				atomicAdd(&R[j].y,-0.5*dr*(rij.y/r));
				atomicAdd(&R[j].z,-0.5*dr*(rij.z/r));
			}
		}
		// otherwise, calculate soft repulsion as usual:
		else {
			float force = repA/pow(r,2) - repA/pow(repD,2);
			if (force > repFmax) force = repFmax;
			F[i] += force*(rij/r);
		}
		*/
		
		
		float force = repA/pow(r,2) - repA/pow(repD,2);
		//if (force > repFmax) force = repFmax;
		F[i] += force*(rij/r);
		
		if (r < 0.5) printf("separation = %f \n",r);
	} 	
}



// --------------------------------------------------------
// IBM3D kernel to calculate wall forces:
// --------------------------------------------------------

__global__ void wall_forces_ydir_IBM3D(
	float3* R,
	float3* F,
	float3 Box,
	int nNodes)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nNodes) {
		const float d = 2.0;
		const float A = 2.0;
		const float yi = R[i].y;
		// bottom wall
		if (yi < d) {
			const float force = A/pow(yi,2) - A/pow(d,2);
			F[i].y += force;
		}
		// top wall
		else if (yi > Box.y-d) {
			const float bmyi = Box.y - yi;
			const float force = A/pow(bmyi,2) - A/pow(d,2);
			F[i].y -= force;
		}
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate wall forces:
// --------------------------------------------------------

__global__ void wall_forces_zdir_IBM3D(
	float3* R,
	float3* F,
	float3 Box,
	float repA,
	float repD,
	int nNodes)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nNodes) {
		const float d = repD;
		const float A = repA;
		const float zi = R[i].z;
		// bottom wall
		if (zi < d) {
			const float force = A/pow(zi,2) - A/pow(d,2);
			F[i].z += force;
		}
		// top wall
		else if (zi > Box.z-d) {
			const float bmzi = Box.z - zi;
			const float force = A/pow(bmzi,2) - A/pow(d,2);
			F[i].z -= force;
		}
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate wall forces:
// --------------------------------------------------------

__global__ void wall_forces_ydir_zdir_IBM3D(
	float3* R,
	float3* F,
	float3 Box,
	float repA,
	float repD,
	int nNodes)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nNodes) {
		const float d = repD;
		const float A = repA;
		const float yi = R[i].y;
		const float zi = R[i].z;
		// bottom wall
		if (yi < d) {
			const float force = A/pow(yi,2) - A/pow(d,2);
			F[i].y += force;
		}
		// top wall
		else if (yi > Box.y-d) {
			const float bmyi = Box.y - yi;
			const float force = A/pow(bmyi,2) - A/pow(d,2);
			F[i].y -= force;
		}
		// back wall
		if (zi < d) {
			const float force = A/pow(zi,2) - A/pow(d,2);
			F[i].z += force;
		}
		// front wall
		else if (zi > Box.z-d) {
			const float bmzi = Box.z - zi;
			const float force = A/pow(bmzi,2) - A/pow(d,2);
			F[i].z -= force;
		}
	}
}



// --------------------------------------------------------
// IBM3D kernel to build the binMap array:
// --------------------------------------------------------

__global__ void build_binMap_IBM3D(
	int* binMap,
	int3 numBins,
	int nnbins,
	int nBins)
{
	// define bin:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBins) {
	
		// -------------------------------
		// calculate bin's x,y,z coordinates:
		// -------------------------------
				
		int binx = i/(numBins.y*numBins.z);
		int biny = (i/numBins.z)%numBins.y;
		int binz = i%numBins.z;
		
		// -------------------------------
		// determine neighboring bins:
		// -------------------------------
		
		int cnt = 0;
		int offst = i*nnbins;
		
		for (int bx = binx-1; bx < binx+2; bx++) {
			for (int by = biny-1; by < biny+2; by++) {
				for (int bz = binz-1; bz < binz+2; bz++) {
					// do not include current bin
					if (bx==binx && by==biny && bz==binz) continue;
					// bin index of neighbor
					binMap[offst+cnt] = bin_index(bx,by,bz,numBins);
					// update counter
					cnt++;
				}
			}
		}		
		
	}	
}
	
	

// --------------------------------------------------------
// IBM3D kernel to calculate i-j force:
// --------------------------------------------------------

__device__ inline int bin_index(
	int i, 
	int j,
	int k, 
	const int3 size)
{
    if (i < 0) i += size.x;
    if (i >= size.x) i -= size.x;
    if (j < 0) j += size.y;
    if (j >= size.y) j -= size.y;
    if (k < 0) k += size.z;
    if (k >= size.z) k -= size.z;
    return i*size.z*size.y + j*size.z + k;
}


