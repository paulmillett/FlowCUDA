#include "hip/hip_runtime.h"
# include "kernels_rods_ibm3D.cuh"
# include "kernels_filaments_ibm3D.cuh"
# include <stdio.h>



// --------------------------------------------------------
// IBM3D kernel to zero rod forces, torques, moment of
// inertia:
// --------------------------------------------------------

__global__ void zero_rod_forces_torques_moments_IBM3D(
	rod* rods,	
	int nRods)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nRods) {
		rods[i].f = make_float3(0.0f,0.0f,0.0f);
		rods[i].t = make_float3(0.0f,0.0f,0.0f);
		rods[i].Ixx = 0.0;
		rods[i].Iyy = 0.0;
		rods[i].Izz = 0.0;
		rods[i].Ixy = 0.0;
		rods[i].Ixz = 0.0;
		rods[i].Iyz = 0.0;
	}
}



// --------------------------------------------------------
// IBM3D kernel to zero bead forces:
// --------------------------------------------------------

__global__ void zero_bead_forces_IBM3D(
	beadrod* beads,	
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		beads[i].f = make_float3(0.0f,0.0f,0.0f);
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate rod orientation:
// --------------------------------------------------------

__global__ void set_rod_position_orientation_IBM3D(
	beadrod* beads,
	rod* rods,	
	int nRods)
{
	// define rod:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nRods) {
		int BH = rods[i].headBead;
		int BC = rods[i].centerBead;
		float3 orient = normalize(beads[BH].r - beads[BC].r);
		rods[i].r = beads[BC].r;
		rods[i].p = orient;
	}
}



// --------------------------------------------------------
// IBM3D enforce a maximum bead force:
// --------------------------------------------------------

__global__ void enforce_max_bead_force_IBM3D(
	beadrod* beads,
	float fmax,
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		float fi = length(beads[i].f);
		if (fi > fmax) {
			beads[i].f *= (fmax/fi);
		}
	}
}



// --------------------------------------------------------
// IBM3D enforce a maximum bead force:
// --------------------------------------------------------

__global__ void enforce_max_rod_force_torque_IBM3D(
	rod* rods,
	float fmax,
	float tmax,
	int nRods)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nRods) {
		float fi = length(rods[i].f);
		float ti = length(rods[i].t);
		if (fi > fmax) rods[i].f *= (fmax/fi);
		if (ti > tmax) rods[i].t *= (tmax/ti);
	}
}



// --------------------------------------------------------
// IBM3D bead update kernel:
// --------------------------------------------------------

__global__ void update_bead_positions_rods_IBM3D(
	beadrod* beads,
	rod* rods,
	float L0,	
	int nBeads)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		int rodID = beads[i].rodID;
		float offset = float(rods[rodID].centerBead - i);
		beads[i].r = rods[rodID].r + L0*offset*rods[rodID].p;
	}
}



// --------------------------------------------------------
// IBM3D bead update kernel:
// --------------------------------------------------------

__global__ void update_rod_position_orientation_IBM3D(
	rod* rods,
	float dt,
	float gam,	
	int nRods)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nRods) {
		rods[i].r += dt*(rods[i].f/gam);
		rods[i].p += dt*(cross(rods[i].t,rods[i].p));
		rods[i].p = normalize(rods[i].p);
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate rod propulsion force:
// --------------------------------------------------------

__global__ void compute_propulsion_force_rods_IBM3D(
	rod* rods,	
	int nRods)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nRods) {
		rods[i].f += rods[i].fp*rods[i].p;
	}
}



// --------------------------------------------------------
// IBM3D kernel to compute thermal force 
// --------------------------------------------------------

__global__ void compute_thermal_force_IBM3D(
	beadrod* beads,
	hiprandState* state,
	float pref,
	int nBeads)
{		
	// define edge:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	
	if (i < nBeads) {
		float r1 = hiprand_uniform(&state[i]);
		float r2 = hiprand_uniform(&state[i]);
		float r3 = hiprand_uniform(&state[i]);		
		beads[i].f.x += pref*(r1-0.5);
		beads[i].f.y += pref*(r2-0.5);
		beads[i].f.z += pref*(r3-0.5);
	}
}



// --------------------------------------------------------
// IBM3D kernel to compute thermal force 
// --------------------------------------------------------

__global__ void compute_thermal_force_torque_rod_IBM3D(
	rod* rods,
	hiprandState* state,
	float prefT,
	float prefR,
	int nRods)
{		
	// define edge:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	
	if (i < nRods) {
		float r1 = hiprand_uniform(&state[i]);
		float r2 = hiprand_uniform(&state[i]);
		float r3 = hiprand_uniform(&state[i]);		
		rods[i].f.x += prefT*(r1-0.5);
		rods[i].f.y += prefT*(r2-0.5);
		rods[i].f.z += prefT*(r3-0.5);
		float r4 = hiprand_uniform(&state[i]);
		float r5 = hiprand_uniform(&state[i]);
		float r6 = hiprand_uniform(&state[i]);
		rods[i].t.x += prefR*(r4-0.5);
		rods[i].t.y += prefR*(r5-0.5);
		rods[i].t.z += prefR*(r6-0.5);
	}
}



// --------------------------------------------------------
// IBM3D kernel to sum the forces, torques, and moments of
// inertia for the rods:
// --------------------------------------------------------

__global__ void sum_rod_forces_torques_moments_IBM3D(
	beadrod* beads,
	rod* rods,
	float m,	
	int nBeads)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		int rodID = beads[i].rodID;
		float3 com = beads[rods[rodID].centerBead].r;
		float3 ricom = beads[i].r - com;
		float3 force = beads[i].f;
		float3 torque = cross(ricom,beads[i].f);
		// add up forces
		atomicAdd(&rods[rodID].f.x,force.x);
		atomicAdd(&rods[rodID].f.y,force.y);
		atomicAdd(&rods[rodID].f.z,force.z);
		// add up torques
		atomicAdd(&rods[rodID].t.x,torque.x);
		atomicAdd(&rods[rodID].t.y,torque.y);
		atomicAdd(&rods[rodID].t.z,torque.z);		
		// add up moments
		atomicAdd(&rods[rodID].Ixx,m*(ricom.y*ricom.y + ricom.z*ricom.z));
		atomicAdd(&rods[rodID].Iyy,m*(ricom.x*ricom.x + ricom.z*ricom.z));
		atomicAdd(&rods[rodID].Izz,m*(ricom.x*ricom.x + ricom.y*ricom.y));		
		atomicAdd(&rods[rodID].Ixy,-m*(ricom.x*ricom.y));
		atomicAdd(&rods[rodID].Ixz,-m*(ricom.x*ricom.z));
		atomicAdd(&rods[rodID].Iyz,-m*(ricom.y*ricom.z));		
	}
}



// --------------------------------------------------------
// IBM3D kernel to unwrap bead coordinates.  Here, the
// beads of a rod are brought back close to the rod's 
// centerBead.  This is done to avoid complications with
// PBCs:
// --------------------------------------------------------

__global__ void unwrap_bead_coordinates_rods_IBM3D(
	beadrod* beads,
	rod* rods,
	float3 Box,
	int3 pbcFlag,
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		int f = beads[i].rodID;
		int j = rods[f].centerBead;
		float3 rij = beads[j].r - beads[i].r;		
		beads[i].r = beads[i].r + roundf(rij/Box)*Box*pbcFlag; // PBC's
	}
}



// --------------------------------------------------------
// IBM3D kernel to wrap bead coordinates for PBCs:
// --------------------------------------------------------

__global__ void wrap_bead_coordinates_IBM3D(
	beadrod* beads,
	float3 Box,
	int3 pbcFlag,
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {	
		beads[i].r = beads[i].r - floorf(beads[i].r/Box)*Box*pbcFlag;		
	}
}



// --------------------------------------------------------
// IBM3D kernel to wrap bead coordinates for PBCs:
// --------------------------------------------------------

__global__ void wrap_rod_coordinates_IBM3D(
	rod* rods,
	float3 Box,
	int3 pbcFlag,
	int nRods)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nRods) {	
		rods[i].r = rods[i].r - floorf(rods[i].r/Box)*Box*pbcFlag;		
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate wall forces:
// --------------------------------------------------------

__global__ void bead_wall_forces_ydir_IBM3D(
	beadrod* beads,
	float3 Box,
	float repA,
	float repD,
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		const float d = repD;
		const float A = repA;
		const float yi = beads[i].r.y;
		// bottom wall
		if (yi < d) {
			const float force = A/pow(yi,2) - A/pow(d,2);
			beads[i].f.y += force;
			if (yi < 0.0001) beads[i].r.y = 0.0001;
		}
		// top wall
		else if (yi > (Box.y-1.0)-d) {
			const float bmyi = (Box.y-1.0) - yi;
			const float force = A/pow(bmyi,2) - A/pow(d,2);
			beads[i].f.y -= force;
			if (yi > Box.y-1.0001) beads[i].r.y = Box.y-1.0001;
		}
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate wall forces:
// --------------------------------------------------------

__global__ void bead_wall_forces_zdir_IBM3D(
	beadrod* beads,
	float3 Box,
	float repA,
	float repD,
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		const float d = repD;
		const float A = repA;
		const float zi = beads[i].r.z;
		// bottom wall
		if (zi < d) {
			const float force = A/pow(zi,2) - A/pow(d,2);
			beads[i].f.z += force;
			if (zi < 0.0001) beads[i].r.z = 0.0001;
		}
		// top wall
		else if (zi > (Box.z-1.0)-d) {
			const float bmzi = (Box.z-1.0) - zi;
			const float force = A/pow(bmzi,2) - A/pow(d,2);
			beads[i].f.z -= force;
			if (zi > Box.z-1.0001) beads[i].r.z = Box.z-1.0001;
		}
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate wall forces:
// --------------------------------------------------------

__global__ void bead_wall_forces_ydir_zdir_IBM3D(
	beadrod* beads,
	float3 Box,
	float repA,
	float repD,
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		const float d = repD;
		const float A = repA;
		const float yi = beads[i].r.y;
		const float zi = beads[i].r.z;
		// bottom wall
		if (yi < d) {
			const float force = A/pow(yi,2) - A/pow(d,2);
			beads[i].f.y += force;
			if (yi < 0.0001) beads[i].r.y = 0.0001;
		}
		// top wall
		else if (yi > (Box.y-1.0)-d) {
			const float bmyi = (Box.y-1.0) - yi;
			const float force = A/pow(bmyi,2) - A/pow(d,2);
			beads[i].f.y -= force;
			if (yi > Box.y-1.0001) beads[i].r.y = Box.y-1.0001;
		}
		// back wall
		if (zi < d) {
			const float force = A/pow(zi,2) - A/pow(d,2);
			beads[i].f.z += force;
			if (zi < 0.0001) beads[i].r.z = 0.0001;
		}
		// front wall
		else if (zi > (Box.z-1.0)-d) {
			const float bmzi = (Box.z-1.0) - zi;
			const float force = A/pow(bmzi,2) - A/pow(d,2);
			beads[i].f.z -= force;
			if (zi > Box.z-1.0001) beads[i].r.z = Box.z-1.0001;
		}
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate wall forces:
// --------------------------------------------------------

__global__ void push_beads_into_sphere_IBM3D(
	beadrod* beads,
	float xs,
	float ys,
	float zs,
	float rs,
	int nBeads)
{
	// define node:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {
		float3 sphere = make_float3(xs,ys,zs);
		float3 ris = beads[i].r - sphere;
		float r = length(ris);
		if (r > (rs-1.5)) {
			ris /= r;
			beads[i].f -= 0.0005*ris;
		}
	}
}



// --------------------------------------------------------
// IBM3D kernel to assign beads to bins:
// --------------------------------------------------------

__global__ void build_bin_lists_for_beads_IBM3D(
	beadrod* beads,
	bindata bins,
	int nBeads)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {		
		
		// -------------------------------
		// calculate bin ID:
		// -------------------------------
		
		int binID = int(floor(beads[i].r.x/bins.sizeBins))*bins.numBins.z*bins.numBins.y +  
			        int(floor(beads[i].r.y/bins.sizeBins))*bins.numBins.z +
		            int(floor(beads[i].r.z/bins.sizeBins));		
						
		// -------------------------------
		// update the lists:
		// -------------------------------
		
		if (binID >= 0 && binID < bins.nBins) {
			atomicAdd(&bins.binOccupancy[binID],1);
			int offst = binID*bins.binMax;
			for (int k=offst; k<offst+bins.binMax; k++) {
				int flag = atomicCAS(&bins.binMembers[k],-1,i); 
				if (flag == -1) break;  
			}
		}
		
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate nonbonded bead interactions
// using the bin lists:
// --------------------------------------------------------

__global__ void nonbonded_bead_interactions_IBM3D(
	beadrod* beads,
	bindata bins,
	float repA,
	float repD,
	int nBeads,
	float3 Box,	
	int3 pbcFlag)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {		
		
		// -------------------------------
		// calculate bin ID:
		// -------------------------------
		
		int binID = int(floor(beads[i].r.x/bins.sizeBins))*bins.numBins.z*bins.numBins.y +  
			        int(floor(beads[i].r.y/bins.sizeBins))*bins.numBins.z +
		            int(floor(beads[i].r.z/bins.sizeBins));		
		
		// -------------------------------
		// loop over beads in the same bin:
		// -------------------------------
				
		int offst = binID*bins.binMax;
		int occup = bins.binOccupancy[binID];
		if (occup > bins.binMax) {
			printf("occup = %i \n", occup);
			occup = bins.binMax;
		}
								
		for (int k=offst; k<offst+occup; k++) {
			int j = bins.binMembers[k];
			if (i==j) continue;
			if (beads[i].rodID == beads[j].rodID) continue;
			pairwise_bead_interaction_forces_WCA(i,j,repA,repD,beads,Box,pbcFlag);			
		}
		
		// -------------------------------
		// loop over neighboring bins:
		// -------------------------------
		
        for (int b=0; b<bins.nnbins; b++) {
            // get neighboring bin ID
			int naborbinID = bins.binMap[binID*bins.nnbins + b];
			offst = naborbinID*bins.binMax;
			occup = bins.binOccupancy[naborbinID];
			if (occup > bins.binMax) occup = bins.binMax;
			// loop over beads in this bin:
			for (int k=offst; k<offst+occup; k++) {
				int j = bins.binMembers[k];
				if (beads[i].rodID == beads[j].rodID) continue;				
				pairwise_bead_interaction_forces_WCA(i,j,repA,repD,beads,Box,pbcFlag);			
			}
		}
				
	}
}



// --------------------------------------------------------
// IBM3D kernel to calculate nonbonded bead-node interactions
// using the bin lists.  Here, the nodes are from the 
// 'class_capsules_ibm3D' class
// --------------------------------------------------------

__global__ void nonbonded_bead_node_interactions_rods_IBM3D(
	beadrod* beads,
	node* nodes,
	bindata bins,
	float repA,
	float repD,
	int nBeads,
	float3 Box,	
	int3 pbcFlag)
{
	// define bead:
	int i = blockIdx.x*blockDim.x + threadIdx.x;		
	if (i < nBeads) {		
		
		// -------------------------------
		// calculate bin ID:
		// -------------------------------
		
		int binID = int(floor(beads[i].r.x/bins.sizeBins))*bins.numBins.z*bins.numBins.y +  
			        int(floor(beads[i].r.y/bins.sizeBins))*bins.numBins.z +
		            int(floor(beads[i].r.z/bins.sizeBins));		
		
		// -------------------------------
		// loop over beads in the same bin:
		// -------------------------------
				
		int offst = binID*bins.binMax;
		int occup = bins.binOccupancy[binID];
		if (occup > bins.binMax) occup = bins.binMax;
								
		for (int k=offst; k<offst+occup; k++) {
			int j = bins.binMembers[k];
			pairwise_bead_node_interaction_forces_WCA(i,j,repA,repD,beads,nodes,Box,pbcFlag);			
		}
		
		// -------------------------------
		// loop over neighboring bins:
		// -------------------------------
		
        for (int b=0; b<bins.nnbins; b++) {
            // get neighboring bin ID
			int naborbinID = bins.binMap[binID*bins.nnbins + b];
			offst = naborbinID*bins.binMax;
			occup = bins.binOccupancy[naborbinID];
			if (occup > bins.binMax) occup = bins.binMax;
			// loop over nodes in this bin:
			for (int k=offst; k<offst+occup; k++) {
				int j = bins.binMembers[k];
				pairwise_bead_node_interaction_forces_WCA(i,j,repA,repD,beads,nodes,Box,pbcFlag);		
			}
		}
				
	}
}











// **********************************************************************************************
// Miscellaneous kernels and functions
// **********************************************************************************************













// --------------------------------------------------------
// IBM3D kernel to calculate i-j force:
// Weeks-Chandler-Anderson potential
// --------------------------------------------------------

__device__ inline void pairwise_bead_interaction_forces_WCA(
	const int i, 
	const int j,
	const float repA,
	const float repD,
	beadrod* beads,
	float3 Box,
	int3 pbcFlag)
{
	float3 rij = beads[i].r - beads[j].r;
	rij -= roundf(rij/Box)*Box*pbcFlag;  // PBC's	
	const float r = length(rij);
	if (r < repD) {
		float sig = 0.8909*repD;  // this ensures F=0 is at cutoff
		float eps = 0.001;
		float sigor = sig/r;
		float sigor6 = sigor*sigor*sigor*sigor*sigor*sigor;
		float sigor12 = sigor6*sigor6;
		float force = 24.0*eps*(2*sigor12 - sigor6)/r/r;
		beads[i].f += force*rij;
	} 	
}



// --------------------------------------------------------
// IBM3D kernel to calculate i-j force:
// NOTE: here 'i' is a bead and 'j' is a node
// --------------------------------------------------------

__device__ inline void pairwise_bead_node_interaction_forces_WCA(
	const int i, 
	const int j,
	const float repA,
	const float repD,
	beadrod* beads,
	node* nodes,
	float3 Box,
	int3 pbcFlag)
{
	float3 rij = beads[i].r - nodes[j].r;
	rij -= roundf(rij/Box)*Box*pbcFlag;  // PBC's	
	const float r = length(rij);
	if (r < repD) {
		float delta = 1.0;
		float sig = 0.8909*(repD - delta);  // this ensures F=0 is at cutoff
		float eps = 0.001;
		float rmd = r - delta;
		float sigor = sig/rmd;
		float sigor6 = sigor*sigor*sigor*sigor*sigor*sigor;
		float sigor12 = sigor6*sigor6;
		float force = 24.0*eps*(2*sigor12 - sigor6)/rmd/rmd;
		beads[i].f += force*rij;
	} 	
}



// --------------------------------------------------------
// IBM3D kernel to solve for angular acceleration by
// solving [I][a]=[T], which is a 3x3 matrix problem
// --------------------------------------------------------

__device__ inline float3 solve_angular_acceleration(
	const float Ixx,
	const float Iyy,
	const float Izz,
	const float Ixy,
	const float Ixz,
	const float Iyz,
	const float3 t)
{	
	// first, set up coefficient matrix, which includes moments of inertia and torques:
	float coeff[3][4] = { {Ixx,Ixy,Ixz,t.x}, {Ixy,Iyy,Iyz,t.y}, {Ixz,Iyz,Izz,t.z} };
	
    // matrix d using coeff as given in cramer's rule
    float d[3][3] = {
        { coeff[0][0], coeff[0][1], coeff[0][2] },
        { coeff[1][0], coeff[1][1], coeff[1][2] },
        { coeff[2][0], coeff[2][1], coeff[2][2] },
    };
    // matrix d1 using coeff as given in cramer's rule
    float d1[3][3] = {
        { coeff[0][3], coeff[0][1], coeff[0][2] },
        { coeff[1][3], coeff[1][1], coeff[1][2] },
        { coeff[2][3], coeff[2][1], coeff[2][2] },
    };
    // matrix d2 using coeff as given in cramer's rule
    float d2[3][3] = {
        { coeff[0][0], coeff[0][3], coeff[0][2] },
        { coeff[1][0], coeff[1][3], coeff[1][2] },
        { coeff[2][0], coeff[2][3], coeff[2][2] },
    };
    // matrix d3 using coeff as given in cramer's rule
    float d3[3][3] = {
        { coeff[0][0], coeff[0][1], coeff[0][3] },
        { coeff[1][0], coeff[1][1], coeff[1][3] },
        { coeff[2][0], coeff[2][1], coeff[2][3] },
    };
	
    // calculating determinant of matrices d, d1, d2, d3
    float D  = determinantOfMatrix(d);
    float D1 = determinantOfMatrix(d1);
    float D2 = determinantOfMatrix(d2);
    float D3 = determinantOfMatrix(d3);
	
	// angular accelerations:
	float aa_x = D1/D;
	float aa_y = D2/D;
	float aa_z = D3/D;
	
	// if rod is alligned along a main axis, 
	// correct the ang. accel. to zero:
	if (Ixx == 0.0f) {
		aa_x = 0.0;
		aa_y = t.y/Iyy;
		aa_z = t.z/Izz;
	}
	if (Iyy == 0.0f) {
		aa_x = t.x/Ixx;
		aa_y = 0.0;
		aa_z = t.z/Izz;
	}
	if (Izz == 0.0f) {
		aa_x = t.x/Ixx;
		aa_y = t.y/Iyy;
		aa_z = 0.0;
	}
	
	return make_float3(aa_x,aa_y,aa_z);
}



// --------------------------------------------------------
// IBM3D kernel to find determinant of 3x3 matrix:
// --------------------------------------------------------

__device__ inline float determinantOfMatrix(float mat[3][3])
{
    float ans = mat[0][0] * (mat[1][1] * mat[2][2] - mat[2][1] * mat[1][2]) -
		        mat[0][1] * (mat[1][0] * mat[2][2] - mat[1][2] * mat[2][0]) + 
                mat[0][2] * (mat[1][0] * mat[2][1] - mat[1][1] * mat[2][0]);
    return ans;
}
 

 
// --------------------------------------------------------
// IBM3D kernel to initialize hiprand random num. generator:
// --------------------------------------------------------

__global__ void init_curand_rods_IBM3D(
	hiprandState* state,
	unsigned long seed,
	int nRods)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < nRods) {
		hiprand_init(seed,i,0,&state[i]);
	}    
}





