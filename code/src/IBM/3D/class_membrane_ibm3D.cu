#include "hip/hip_runtime.h"
 
# include "class_membrane_ibm3D.cuh"
# include "../../IO/GetPot"
# include "../../Utils/eig3.cuh"
# include <math.h>
# include <iostream>
# include <iomanip>
# include <fstream>
# include <string>
# include <sstream>
# include <stdlib.h>
using namespace std;  








// **********************************************************************************************
// Constructor, destructor, and array allocations...
// **********************************************************************************************








// --------------------------------------------------------
// Constructor:
// --------------------------------------------------------

class_membrane_ibm3D::class_membrane_ibm3D()
{
	// get some parameters:
	GetPot inputParams("input.dat");	
	nNodesPerCell = inputParams("IBM/nNodesPerCell",0);
	nFacesPerCell = inputParams("IBM/nFacesPerCell",0);	
	nEdgesPerCell = inputParams("IBM/nEdgesPerCell",0);
	nCells = inputParams("IBM/nCells",1);
	dt = inputParams("Time/dt",1.0);
	ks = inputParams("IBM/ks",0.0);
	kb = inputParams("IBM/kb",0.0);
	ka = inputParams("IBM/ka",0.0);
	kag = inputParams("IBM/kag",0.0);
	kv = inputParams("IBM/kv",0.0);
	C  = inputParams("IBM/C",0.0);
	repA = inputParams("IBM/repA",0.0);
	repD = inputParams("IBM/repD",0.0);
	repFmax = inputParams("IBM/repFmax",0.0);
	N.x = inputParams("Lattice/Nx",1);
	N.y = inputParams("Lattice/Ny",1);
	N.z = inputParams("Lattice/Nz",1);	
	nNodes = nNodesPerCell*nCells;
	nFaces = nFacesPerCell*nCells;
	nEdges = nEdgesPerCell*nCells;
	Box.x = float(N.x);   // assume dx=1
	Box.y = float(N.y);
	Box.z = float(N.z);
	pbcFlag = make_int3(1,1,1);  
		
	// if we need bins, do some calculations:
	binsFlag = false;
	if (nCells > 1) binsFlag = true;
	if (binsFlag) {
		sizeBins = inputParams("IBM/sizeBins",2.0);
		binMax = inputParams("IBM/binMax",1);			
		numBins.x = int(floor(N.x/sizeBins));
	    numBins.y = int(floor(N.y/sizeBins));
	    numBins.z = int(floor(N.z/sizeBins));
		nBins = numBins.x*numBins.y*numBins.z;
		nnbins = 26;
	}	
}



// --------------------------------------------------------
// Destructor:
// --------------------------------------------------------

class_membrane_ibm3D::~class_membrane_ibm3D()
{
		
}



// --------------------------------------------------------
// Allocate arrays:
// --------------------------------------------------------

void class_membrane_ibm3D::allocate()
{
	// allocate array memory (host):
	rH = (float3*)malloc(nNodes*sizeof(float3));		
	facesH = (triangle*)malloc(nFaces*sizeof(triangle));
	edgesH = (edge*)malloc(nEdges*sizeof(edge));
	cellsH = (cell*)malloc(nCells*sizeof(cell));
	cellIDsH = (int*)malloc(nNodes*sizeof(int));
					
	// allocate array memory (device):
	hipMalloc((void **) &r, nNodes*sizeof(float3));	
	hipMalloc((void **) &v, nNodes*sizeof(float3));	
	hipMalloc((void **) &f, nNodes*sizeof(float3));
	hipMalloc((void **) &faces, nFaces*sizeof(triangle));
	hipMalloc((void **) &edges, nEdges*sizeof(edge));
	hipMalloc((void **) &cells, nCells*sizeof(cell));
	hipMalloc((void **) &cellIDs, nNodes*sizeof(int));
	if (binsFlag) {
		hipMalloc((void **) &binMembers, nBins*binMax*sizeof(int));
		hipMalloc((void **) &binOccupancy, nBins*sizeof(int));
		hipMalloc((void **) &binMap, nBins*26*sizeof(int));		
	}	
}



// --------------------------------------------------------
// Deallocate arrays:
// --------------------------------------------------------

void class_membrane_ibm3D::deallocate()
{
	// free array memory (host):
	free(rH);	
	free(facesH);
	free(edgesH);
	free(cellsH);
	free(cellIDsH);	
				
	// free array memory (device):
	hipFree(r);	
	hipFree(v);	
	hipFree(f);
	hipFree(faces);
	hipFree(edges);
	hipFree(cells);
	hipFree(cellIDs);
	if (binsFlag) {
		hipFree(binMembers);
		hipFree(binOccupancy);
		hipFree(binMap);		
	}		
}



// --------------------------------------------------------
// Copy arrays from host to device:
// --------------------------------------------------------

void class_membrane_ibm3D::memcopy_host_to_device()
{
	hipMemcpy(r, rH, sizeof(float3)*nNodes, hipMemcpyHostToDevice);	
	hipMemcpy(faces, facesH, sizeof(triangle)*nFaces, hipMemcpyHostToDevice);	
	hipMemcpy(edges, edgesH, sizeof(edge)*nEdges, hipMemcpyHostToDevice);
	hipMemcpy(cells, cellsH, sizeof(cell)*nCells, hipMemcpyHostToDevice);
	hipMemcpy(cellIDs, cellIDsH, sizeof(int)*nNodes, hipMemcpyHostToDevice);
}
	


// --------------------------------------------------------
// Copy arrays from device to host:
// --------------------------------------------------------

void class_membrane_ibm3D::memcopy_device_to_host()
{
	hipMemcpy(rH, r, sizeof(float3)*nNodes, hipMemcpyDeviceToHost);	
	hipMemcpy(facesH, faces, sizeof(triangle)*nFaces, hipMemcpyDeviceToHost);
	//hipMemcpy(edgesH, edges, sizeof(edge)*nEdges, hipMemcpyDeviceToHost);
	
	// unwrap coordinate positions:
	unwrap_node_coordinates(); 
}











// **********************************************************************************************
// Initialization Stuff...
// **********************************************************************************************











// --------------------------------------------------------
// Read IBM information from file:
// --------------------------------------------------------

void class_membrane_ibm3D::read_ibm_information(std::string tagname)
{
	read_ibm_information_long(tagname,nNodesPerCell,nFacesPerCell,nEdgesPerCell,rH,facesH,edgesH);
}



// --------------------------------------------------------
// Setters:
// --------------------------------------------------------

void class_membrane_ibm3D::set_pbcFlag(int x, int y, int z)
{
	pbcFlag.x = x; pbcFlag.y = y; pbcFlag.z = z;
}

void class_membrane_ibm3D::set_ks(float val)
{
	ks = val;
}

void class_membrane_ibm3D::set_ka(float val)
{
	ka = val;
}

void class_membrane_ibm3D::set_kb(float val)
{
	kb = val;
}

void class_membrane_ibm3D::set_kv(float val)
{
	kv = val;
}

void class_membrane_ibm3D::set_kag(float val)
{
	kag = val;
}

void class_membrane_ibm3D::set_C(float val)
{
	C = val;
}



// --------------------------------------------------------
// Assign the reference node to every cell.  The reference
// node is arbitrary (here we use the first node), but it
// is necessary for handling PBC's.
// --------------------------------------------------------

void class_membrane_ibm3D::assign_refNode_to_cells()
{
	for (int c=0; c<nCells; c++) {
		cellsH[c].refNode = c*nNodesPerCell;
	}
}	



// --------------------------------------------------------
// Assign the cell ID to every node:
// --------------------------------------------------------

void class_membrane_ibm3D::assign_cellIDs_to_nodes()
{
	for (int c=0; c<nCells; c++) {
		for (int i=0; i<nNodesPerCell; i++) {
			int ii = i + c*nNodesPerCell;
			cellIDsH[ii] = c;
		}
	}
}



// --------------------------------------------------------
// Duplicate the first cell mesh information to all cells:
// --------------------------------------------------------

void class_membrane_ibm3D::duplicate_cells()
{
	if (nCells > 1) {
		for (int c=1; c<nCells; c++) {
			// copy node positions:
			for (int i=0; i<nNodesPerCell; i++) {
				int ii = i + c*nNodesPerCell;
				rH[ii] = rH[i];
			}
			// copy edge info:
			for (int i=0; i<nEdgesPerCell; i++) {
				int ii = i + c*nEdgesPerCell;
				edgesH[ii].v0 = edgesH[i].v0 + c*nNodesPerCell;
				edgesH[ii].v1 = edgesH[i].v1 + c*nNodesPerCell;
				edgesH[ii].f0 = edgesH[i].f0 + c*nFacesPerCell;
				edgesH[ii].f1 = edgesH[i].f1 + c*nFacesPerCell;
			}
			// copy face info:
			for (int i=0; i<nFacesPerCell; i++) {
				int ii = i + c*nFacesPerCell;
				facesH[ii].v0 = facesH[i].v0 + c*nNodesPerCell;
				facesH[ii].v1 = facesH[i].v1 + c*nNodesPerCell;
				facesH[ii].v2 = facesH[i].v2 + c*nNodesPerCell;
				facesH[ii].cellID = c;								
			}
		}
	}
	
}



// --------------------------------------------------------
// With the Host, shrink cells and randomly shift them with
// the box:
// --------------------------------------------------------

void class_membrane_ibm3D::shrink_and_randomize_cells(float shrinkFactor, float sepMin, float sepWall)
{
	// copy node positions from device to host:
	hipMemcpy(rH, r, sizeof(float3)*nNodes, hipMemcpyDeviceToHost);
	
	// shrink cells by specified amount:
	for (int c=0; c<nCells; c++) {
		for (int i=0; i<nNodesPerCell; i++) {
			int indx = i + c*nNodesPerCell;
			rH[indx] *= shrinkFactor;
		}
	}
	
	// randomly shift cells, without overlapping previous cells:
	float3* cellCOM = (float3*)malloc(nCells*sizeof(float3));
	for (int c=0; c<nCells; c++) {
		cellCOM[c] = make_float3(0.0);
		float3 shift = make_float3(0.0);		
		bool tooClose = true;
		while (tooClose) {
			// reset tooClose to false
			tooClose = false;
			// get random position
			shift.x = (float)rand()/RAND_MAX*Box.x;
			shift.y = sepWall + (float)rand()/RAND_MAX*(Box.y-2.0*sepWall);
			shift.z = sepWall + (float)rand()/RAND_MAX*(Box.z-2.0*sepWall);
			// check with other cells
			for (int d=0; d<c; d++) {
				float sep = calc_separation_pbc(shift,cellCOM[d]);
                if (sep < sepMin) 
                {
                    tooClose = true;
                    break;
                }
			}
			
		}
		cellCOM[c] = shift;		
		rotate_and_shift_node_positions(c,shift.x,shift.y,shift.z);
	}
	
	// last, copy node positions from host to device:
	hipMemcpy(r, rH, sizeof(float3)*nNodes, hipMemcpyHostToDevice);
}



// --------------------------------------------------------
// calculate separation distance using PBCs:
// --------------------------------------------------------

float class_membrane_ibm3D::calc_separation_pbc(float3 r1, float3 r2)
{
	float3 dr = r1 - r2;
	dr -= roundf(dr/Box)*Box;
	return length(dr);
}



// --------------------------------------------------------
// Shift IBM start positions by specified amount:
// --------------------------------------------------------

void class_membrane_ibm3D::shift_node_positions(int cellID, float xsh, float ysh, float zsh)
{
	for (int i=0; i<nNodesPerCell; i++) {
		int indx = i + cellID*nNodesPerCell;		 
		rH[indx].x += xsh;
		rH[indx].y += ysh;
		rH[indx].z += zsh;		
	}
}



// --------------------------------------------------------
// Shift IBM start positions by specified amount:
// --------------------------------------------------------

void class_membrane_ibm3D::rotate_and_shift_node_positions(int cellID, float xsh, float ysh, float zsh)
{
	float a = M_PI*(float)rand()/RAND_MAX;  // alpha
	float b = M_PI*(float)rand()/RAND_MAX;  // beta
	float g = M_PI*(float)rand()/RAND_MAX;  // gamma
	for (int i=0; i<nNodesPerCell; i++) {
		int indx = i + cellID*nNodesPerCell;
		// rotate:
		float xrot = rH[indx].x*(cos(a)*cos(b)) + rH[indx].y*(cos(a)*sin(b)*sin(g)-sin(a)*cos(g)) + rH[indx].z*(cos(a)*sin(b)*cos(g)+sin(a)*sin(g));
		float yrot = rH[indx].x*(sin(a)*cos(b)) + rH[indx].y*(sin(a)*sin(b)*sin(g)+cos(a)*cos(g)) + rH[indx].z*(sin(a)*sin(b)*cos(g)-cos(a)*sin(g));
		float zrot = rH[indx].x*(-sin(b))       + rH[indx].y*(cos(b)*sin(g))                      + rH[indx].z*(cos(b)*cos(g));
		// shift:		 
		rH[indx].x = xrot + xsh;
		rH[indx].y = yrot + ysh;
		rH[indx].z = zrot + zsh;		
	}
}



// --------------------------------------------------------
// Write IBM output to file:
// --------------------------------------------------------

void class_membrane_ibm3D::write_output(std::string tagname, int tagnum)
{
	write_vtk_immersed_boundary_3D(tagname,tagnum,
	nNodes,nFaces,rH,facesH);
}



// --------------------------------------------------------
// Write IBM output to file, including more information
// (edge angles):
// --------------------------------------------------------

void class_membrane_ibm3D::write_output_long(std::string tagname, int tagnum)
{
	write_vtk_immersed_boundary_normals_3D(tagname,tagnum,
	nNodes,nFaces,nEdges,rH,facesH,edgesH);
}



// --------------------------------------------------------
// Calculate rest geometries (Spring model):
// --------------------------------------------------------

void class_membrane_ibm3D::rest_geometries(int nBlocks, int nThreads)
{
	// rest edge lengths:
	rest_edge_lengths_IBM3D
	<<<nBlocks,nThreads>>> (r,edges,nEdges);
	
	// rest edge angles:
	rest_edge_angles_IBM3D
	<<<nBlocks,nThreads>>> (r,edges,faces,nEdges);
	
	// rest triangle area:
	rest_triangle_areas_IBM3D
	<<<nBlocks,nThreads>>> (r,faces,cells,nFaces);
}



// --------------------------------------------------------
// Calculate rest geometries (Skalak model):
// --------------------------------------------------------

void class_membrane_ibm3D::rest_geometries_skalak(int nBlocks, int nThreads)
{
	// rest triangle properties:
	rest_triangle_skalak_IBM3D
	<<<nBlocks,nThreads>>> (r,faces,cells,nFaces);
		
	// rest edge angles for bending:
	rest_edge_angles_IBM3D
	<<<nBlocks,nThreads>>> (r,edges,faces,nEdges);
}



// --------------------------------------------------------
// For a certain number of iterations, relax the 
// the node positions (for example, after cells are shrunk 
// to allow them to readjust to their regular volume):
// --------------------------------------------------------

void class_membrane_ibm3D::relax_node_positions(int nIts, float scale, float M, int nBlocks, int nThreads) 
{
	// per iteraction scale factor:
	float power = 1.0/float(nIts);
	float scalePerIter = powf(scale,power);
	
	// make sure node coordinates are wrapped for 
	// PBC's prior to building bin-lists the first time:
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);	
	
	// iterate to relax node positions while scaling equilibirum
	// cell size:
	for (int i=0; i<nIts; i++) {
		if (i%10000 == 0) cout << "relax step " << i << endl;		
		scale_equilibrium_cell_size(scalePerIter,nBlocks,nThreads);		
		reset_bin_lists(nBlocks,nThreads);		
		build_bin_lists(nBlocks,nThreads);		
		compute_node_forces(nBlocks,nThreads);		
		nonbonded_node_interactions(nBlocks,nThreads);		
		//wall_forces_ydir(nBlocks,nThreads);		
		wall_forces_ydir_zdir(nBlocks,nThreads);
		update_node_positions_vacuum(M,nBlocks,nThreads);		
		hipDeviceSynchronize();
	}	
}



// --------------------------------------------------------
// For a certain number of iterations, relax the 
// the node positions (for example, after cells are shrunk 
// to allow them to readjust to their regular volume):
// --------------------------------------------------------

void class_membrane_ibm3D::relax_node_positions_skalak(int nIts, float scale, float M, int nBlocks, int nThreads) 
{
	// per iteraction scale factor:
	float power = 1.0/float(nIts);
	float scalePerIter = powf(scale,power);
	
	// make sure node coordinates are wrapped for 
	// PBC's prior to building bin-lists the first time:
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);	
	
	// iterate to relax node positions while scaling equilibirum
	// cell size:
	for (int i=0; i<nIts; i++) {
		if (i%10000 == 0) cout << "relax step " << i << endl;		
		scale_equilibrium_cell_size(scalePerIter,nBlocks,nThreads);		
		reset_bin_lists(nBlocks,nThreads);		
		build_bin_lists(nBlocks,nThreads);		
		compute_node_forces_skalak(nBlocks,nThreads);		
		nonbonded_node_interactions(nBlocks,nThreads);		
		//wall_forces_ydir(nBlocks,nThreads);
		wall_forces_ydir_zdir(nBlocks,nThreads);
		update_node_positions_vacuum(M,nBlocks,nThreads);		
		hipDeviceSynchronize();
	}	
}









// **********************************************************************************************
// Calls to CUDA kernels for main calculations
// **********************************************************************************************











// --------------------------------------------------------
// Call to "update_node_position_vacuum_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::update_node_positions_vacuum(float M, int nBlocks, int nThreads)
{
	update_node_position_vacuum_IBM3D
	<<<nBlocks,nThreads>>> (r,f,M,nNodes);
	
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);	
}



// --------------------------------------------------------
// Call to "update_node_position_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::update_node_positions(int nBlocks, int nThreads)
{
	update_node_position_IBM3D
	<<<nBlocks,nThreads>>> (r,v,nNodes);
	
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);	
}



// --------------------------------------------------------
// Call to "update_node_position_dt_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::update_node_positions_dt(int nBlocks, int nThreads)
{
	update_node_position_dt_IBM3D
	<<<nBlocks,nThreads>>> (r,v,dt,nNodes);
	
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);	
}



// --------------------------------------------------------
// Call to "update_node_position_verlet_1_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::update_node_positions_verlet_1(int nBlocks, int nThreads)
{
	update_node_position_verlet_1_IBM3D
	<<<nBlocks,nThreads>>> (r,v,f,dt,1.0,nNodes);
	
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);	
}



// --------------------------------------------------------
// Call to "update_node_position_verlet_2_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::update_node_positions_verlet_2(int nBlocks, int nThreads)
{
	update_node_position_verlet_2_IBM3D
	<<<nBlocks,nThreads>>> (v,f,dt,1.0,nNodes);
}



// --------------------------------------------------------
// Call to "zero_velocities_forces_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::zero_velocities_forces(int nBlocks, int nThreads)
{
	zero_velocities_forces_IBM3D
	<<<nBlocks,nThreads>>> (v,f,nNodes);
}



// --------------------------------------------------------
// Call to "interpolate_velocity_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::interpolate_velocity(float* uLBM, float* vLBM, 
	float* wLBM, int nBlocks, int nThreads)
{
	interpolate_velocity_IBM3D
	<<<nBlocks,nThreads>>> (r,v,uLBM,vLBM,wLBM,N.x,N.y,N.z,nNodes);	
}



// --------------------------------------------------------
// Call to "extrapolate_force_IBM3D" kernel:
// --------------------------------------------------------

void class_membrane_ibm3D::extrapolate_force(float* fxLBM, float* fyLBM, 
	float* fzLBM, int nBlocks, int nThreads)
{
	extrapolate_force_IBM3D
	<<<nBlocks,nThreads>>> (r,v,fxLBM,fyLBM,fzLBM,N.x,N.y,N.z,nNodes);	
}



// --------------------------------------------------------
// Call to kernel that builds the binMap array:
// --------------------------------------------------------

void class_membrane_ibm3D::build_binMap(int nBlocks, int nThreads)
{
	if (!binsFlag) cout << "Warning: IBM bin arrays have not been initialized" << endl;
	
	cout << "nnbins = " << nnbins << endl;
	
	build_binMap_IBM3D
	<<<nBlocks,nThreads>>> (binMap,numBins,nnbins,nBins);
}



// --------------------------------------------------------
// Call to kernel that resets bin lists:
// --------------------------------------------------------

void class_membrane_ibm3D::reset_bin_lists(int nBlocks, int nThreads)
{
	if (!binsFlag) cout << "Warning: IBM bin arrays have not been initialized" << endl;
	reset_bin_lists_IBM3D
	<<<nBlocks,nThreads>>> (binOccupancy,binMembers,binMax,nBins);
}



// --------------------------------------------------------
// Call to kernel that builds bin lists:
// --------------------------------------------------------

void class_membrane_ibm3D::build_bin_lists(int nBlocks, int nThreads)
{
	if (!binsFlag) cout << "Warning: IBM bin arrays have not been initialized" << endl;
	build_bin_lists_IBM3D
	<<<nBlocks,nThreads>>> (r,binOccupancy,binMembers,numBins,sizeBins,nNodes,binMax);
}



// --------------------------------------------------------
// Call to kernel that calculates nonbonded forces:
// --------------------------------------------------------

void class_membrane_ibm3D::nonbonded_node_interactions(int nBlocks, int nThreads)
{
	if (!binsFlag) cout << "Warning: IBM bin arrays have not been initialized" << endl;
	nonbonded_node_interactions_IBM3D
	<<<nBlocks,nThreads>>> (r,f,binOccupancy,binMembers,binMap,cellIDs,numBins,sizeBins,
	                        repA,repD,repFmax,nNodes,binMax,nnbins,Box,pbcFlag);
}



// --------------------------------------------------------
// Calls to kernels that compute forces on nodes based 
// on the membrane mechanics model (Spring model):
// --------------------------------------------------------

void class_membrane_ibm3D::compute_node_forces(int nBlocks, int nThreads)
{
	// First, zero the node forces and the cell volumes:
	zero_node_forces_IBM3D
	<<<nBlocks,nThreads>>> (f,nNodes);
			
	zero_cell_volumes_IBM3D
	<<<nBlocks,nThreads>>> (cells,nCells);
	
	// Second, unwrap node coordinates:
	unwrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,cells,cellIDs,Box,pbcFlag,nNodes);	
					
	// Third, compute the area dilation force for each face:
	compute_node_force_membrane_area_IBM3D
	<<<nBlocks,nThreads>>> (faces,r,f,cells,ka,nFaces);	
		
	// Forth, compute the edge extension and bending force for each edge:
	compute_node_force_membrane_edge_IBM3D
	<<<nBlocks,nThreads>>> (faces,r,f,edges,ks,nEdges);
	
	compute_node_force_membrane_bending_IBM3D
	<<<nBlocks,nThreads>>> (faces,r,f,edges,kb,nEdges);
		
	// Fifth, compute the volume conservation force for each face:
	compute_node_force_membrane_volume_IBM3D
	<<<nBlocks,nThreads>>> (faces,f,cells,kv,nFaces);
	
	// Sixth, compute the global area conservation force for each face:
	compute_node_force_membrane_globalarea_IBM3D
	<<<nBlocks,nThreads>>> (faces,r,f,cells,kag,nFaces);
		
	// Seventh, re-wrap node coordinates:
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);
			
}



// --------------------------------------------------------
// Calls to kernels that compute forces on nodes based 
// on the membrane mechanics model (Skalak model):
// --------------------------------------------------------

void class_membrane_ibm3D::compute_node_forces_skalak(int nBlocks, int nThreads)
{
	// First, zero the node forces and the cell volumes:
	zero_node_forces_IBM3D
	<<<nBlocks,nThreads>>> (f,nNodes);
			
	zero_cell_volumes_IBM3D
	<<<nBlocks,nThreads>>> (cells,nCells);
	
	// Second, unwrap node coordinates:
	unwrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,cells,cellIDs,Box,pbcFlag,nNodes);	
					
	// Third, compute the Skalak forces for each face:
	compute_node_force_membrane_skalak_IBM3D
	<<<nBlocks,nThreads>>> (faces,r,f,cells,ks,C,nFaces);
	
	// Fourth, compute the bending force for each edge:		
	compute_node_force_membrane_bending_IBM3D
	<<<nBlocks,nThreads>>> (faces,r,f,edges,kb,nEdges);
		
	// Fifth, compute the volume conservation force for each face:
	compute_node_force_membrane_volume_IBM3D
	<<<nBlocks,nThreads>>> (faces,f,cells,kv,nFaces);
			
	// Sixth, re-wrap node coordinates:
	wrap_node_coordinates_IBM3D
	<<<nBlocks,nThreads>>> (r,Box,pbcFlag,nNodes);
			
}



// --------------------------------------------------------
// Call to kernel that calculates wall forces in y-dir:
// --------------------------------------------------------

void class_membrane_ibm3D::wall_forces_ydir(int nBlocks, int nThreads)
{
	wall_forces_ydir_IBM3D
	<<<nBlocks,nThreads>>> (r,f,Box,nNodes);
}



// --------------------------------------------------------
// Call to kernel that calculates wall forces in y-dir
// and z-dir:
// --------------------------------------------------------

void class_membrane_ibm3D::wall_forces_ydir_zdir(int nBlocks, int nThreads)
{
	wall_forces_ydir_zdir_IBM3D
	<<<nBlocks,nThreads>>> (r,f,Box,repA,repD,nNodes);
}



// --------------------------------------------------------
// Call to kernel that changes the default cell volume:
// --------------------------------------------------------

void class_membrane_ibm3D::change_cell_volume(float change, int nBlocks, int nThreads)
{
	change_cell_volumes_IBM3D
	<<<nBlocks,nThreads>>> (cells,change,nCells);
}



// --------------------------------------------------------
// Call to kernel that scales the default cell geometry:
// --------------------------------------------------------

void class_membrane_ibm3D::scale_equilibrium_cell_size(float scale, int nBlocks, int nThreads)
{
	// scale the equilibrium edge length:
	scale_edge_lengths_IBM3D
	<<<nBlocks,nThreads>>> (edges,scale,nEdges);		
	// scale the equilibrium local area:
	scale_face_areas_IBM3D
	<<<nBlocks,nThreads>>> (faces,scale,nFaces);
	// scale the equilibrium global area and volume:
	scale_cell_areas_volumes_IBM3D
	<<<nBlocks,nThreads>>> (cells,scale,nCells);		
}



// --------------------------------------------------------
// Call to kernel that scales the default edge lengths:
// --------------------------------------------------------

void class_membrane_ibm3D::scale_edge_lengths(float scale, int nBlocks, int nThreads)
{
	scale_edge_lengths_IBM3D
	<<<nBlocks,nThreads>>> (edges,scale,nEdges);
}











// **********************************************************************************************
// Analysis and Geometry calculations done by the host (CPU)
// **********************************************************************************************











// --------------------------------------------------------
// Unwrap node coordinates based on difference between node
// position and the cell's reference node position:
// --------------------------------------------------------

void class_membrane_ibm3D::unwrap_node_coordinates()
{
	for (int i=0; i<nNodes; i++) {
		int c = cellIDsH[i];
		int j = cellsH[c].refNode;
		float3 rij = rH[j] - rH[i];
		rH[i] = rH[i] + roundf(rij/Box)*Box*pbcFlag; // PBC's		
	}	
}



// --------------------------------------------------------
// Calculate various geometry properties of capsules,
// including center-of-mass, Taylor deformation index, etc.
// --------------------------------------------------------

void class_membrane_ibm3D::membrane_geometry_analysis(std::string tagname, int tagnum)
{
	
	// Define the file location and name:
	ofstream outfile;
	std::stringstream filenamecombine;
	filenamecombine << "vtkoutput/" << tagname << "_" << tagnum << ".dat";
	string filename = filenamecombine.str();
	outfile.open(filename.c_str(), ios::out | ios::app);
	outfile << nCells << endl;
		
	float yCFL = float(N.y);
	float zCFL = float(N.z);
	// Loop over the capsules, calculate center-of-mass
	// and Taylor deformation parameter.  Here, I'm using
	// the method described in: Eberly D, Polyhedral Mass
	// Properties (Revisited), Geometric Tools, Redmond WA	
	for (int c=0; c<nCells; c++) {
		
		float D = 0.0;
		float3 com = make_float3(0.0,0.0,0.0);
		float mult[10] = {1.0/6.0,1.0/24.0,1.0/24.0,1.0/24.0,1.0/60.0,1.0/60.0,1.0/60.0,1.0/120.0,1.0/120.0,1.0/120.0};
		float intg[10] = {0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0};
				
		for (int f=0; f<nFacesPerCell; f++) {
			// get vertices of triangle i:
			int fID = f + c*nFacesPerCell;
			int v0 = facesH[fID].v0;
			int v1 = facesH[fID].v1;
			int v2 = facesH[fID].v2;
			float x0 = rH[v0].x;
			float y0 = rH[v0].y;
			float z0 = rH[v0].z;
			float x1 = rH[v1].x;
			float y1 = rH[v1].y;
			float z1 = rH[v1].z;
			float x2 = rH[v2].x;
			float y2 = rH[v2].y;
			float z2 = rH[v2].z;
			// get edges and cross product of edges:
			float a1 = x1-x0;
			float b1 = y1-y0;
			float c1 = z1-z0;
			float a2 = x2-x0;
			float b2 = y2-y0;
			float c2 = z2-z0;
			float d0 = b1*c2-b2*c1;
			float d1 = a2*c1-a1*c2;
			float d2 = a1*b2-a2*b1;
			// compute integral terms:
			float f1x,f2x,f3x,g0x,g1x,g2x;
			float f1y,f2y,f3y,g0y,g1y,g2y;
			float f1z,f2z,f3z,g0z,g1z,g2z;
			subexpressions(x0,x1,x2,f1x,f2x,f3x,g0x,g1x,g2x);
			subexpressions(y0,y1,y2,f1y,f2y,f3y,g0y,g1y,g2y);
			subexpressions(z0,z1,z2,f1z,f2z,f3z,g0z,g1z,g2z);
			// update integrals:
			intg[0] += d0*f1x;
			intg[1] += d0*f2x;
			intg[2] += d1*f2y;
			intg[3] += d2*f2z;
			intg[4] += d0*f3x;
			intg[5] += d1*f3y;
			intg[6] += d2*f3z;
			intg[7] += d0*(y0*g0x + y1*g1x + y2*g2x);
			intg[8] += d1*(z0*g0y + z1*g1y + z2*g2y);
			intg[9] += d2*(x0*g0z + x1*g1z + x2*g2z);
			// check cell-free layer value:
			float ypos = (y0+y1+y2)/3.0;
			float zpos = (z0+z1+z2)/3.0;
			float ywallsep = std::fmin(ypos-0.0,float(N.y-1)-ypos);
			float zwallsep = std::fmin(zpos-0.0,float(N.z-1)-zpos);
			if (ywallsep < yCFL) yCFL = ywallsep;
			if (zwallsep < zCFL) zCFL = zwallsep;
		}
		
		for (int i=0; i<10; i++) intg[i] *= mult[i];
		
		// center of mass:
		float mass = intg[0];
		float vol = mass;   // assume density = 1
		com.x = intg[1]/mass;
		com.y = intg[2]/mass;
		com.z = intg[3]/mass;
		
		// inertia tensor relative to center of mass:
		float Ixx = intg[5] + intg[6] - mass*(com.y*com.y + com.z*com.z);
		float Iyy = intg[4] + intg[6] - mass*(com.z*com.z + com.x*com.x);
		float Izz = intg[4] + intg[5] - mass*(com.x*com.x + com.y*com.y);
		float Ixy = -(intg[7] - mass*com.x*com.y);
		float Iyz = -(intg[8] - mass*com.y*com.z);
		float Ixz = -(intg[9] - mass*com.x*com.z);
		float I[3][3] = {{Ixx,Ixy,Ixz}, {Ixy,Iyy,Iyz}, {Ixz,Iyz,Izz}};
		
		// calculate longest and shortest axes of capsule:
		// S = sqrt((5/2/vol)*(Ixx + Iyy - sqrt((Ixx-Iyy)^2 + 4*Ixy^2))/2);
		// L = sqrt((5/2/vol)*(Ixx + Iyy + sqrt((Ixx-Iyy)^2 + 4*Ixy^2))/2);
		// Dsl = (L-S)/(L+S)

		// calculate eigenvalues of inertia tensor:
		float eigvals[3] = {0.0,0.0,0.0}; 
		float eigvecs[3][3] = {{0.0,0.0,0.0}, {0.0,0.0,0.0}, {0.0,0.0,0.0}};
		eigen_decomposition(I,eigvecs,eigvals);
		float L1 = sqrt(5/2/vol*(eigvals[1] + eigvals[2] - eigvals[0]));
		float L2 = sqrt(5/2/vol*(eigvals[0] + eigvals[2] - eigvals[1]));
		float L3 = sqrt(5/2/vol*(eigvals[0] + eigvals[1] - eigvals[2]));

		// calculate Taylor deformation parameters:
		float Lmax = std::max({L1,L2,L3});
		float Lmin = std::min({L1,L2,L3});
		D = (Lmax-Lmin)/(Lmax+Lmin);
		//D12 = (L1-L2)/(L1+L2);
		//D13 = (L1-L3)/(L1+L3);
		//D23 = (L2-L3)/(L2+L3);
		
		// calculate the inclination angle:
		//phi = 0.5*atan(2*Ixy/(Ixx-Iyy));
		//phi = phi/pi;
		
		// print data:
		outfile << fixed << setprecision(4) << vol << "  " << com.x << "  " << com.y << "  " << com.z << "  "
		        << D << "  " << endl;
						
	}
	
	// print the cell-free layer thickness in the y-dir and z-dir:
	outfile << fixed << setprecision(4) << yCFL << "  " << zCFL << endl;
		
	// close file
	outfile.close();
	
}



void class_membrane_ibm3D::subexpressions(
	const float w0,
	const float w1,
	const float w2,
	float& f1,
	float& f2,
	float& f3,
	float& g0,
	float& g1,
	float& g2)
{
    float temp0 = w0 + w1;
    float temp1 = w0*w0;
    float temp2 = temp1 + w1*temp0;
    f1 = temp0 + w2;
	f2 = temp2 + w2*f1;
    f3 = w0*temp1 + w1*temp2 + w2*f2;
    g0 = f2 + w0*(f1 + w0); 
    g1 = f2 + w1*(f1 + w1);
    g2 = f2 + w2*(f1 + w2);	
}


